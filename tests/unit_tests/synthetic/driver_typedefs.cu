// RUN: %run_test hipify "%s" "%t" %hipify_args 2 --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
  printf("03. CUDA Driver API Typedefs synthetic test\n");

  // CHECK: hipDevice_t device;
  hipDevice_t device;
#if CUDA_VERSION > 11020
  // CHECK: hipDevice_t device_v1;
  hipDevice_t device_v1;
#endif

  // CHECK: hipDeviceptr_t deviceptr;
  // CHECK-NEXT: hipDeviceptr_t deviceptr_v1;
  hipDeviceptr_t deviceptr;
  hipDeviceptr_t deviceptr_v1;
#if CUDA_VERSION > 11020
  // CHECK: hipDeviceptr_t deviceptr_v2;
  hipDeviceptr_t deviceptr_v2;
#endif

#if CUDA_VERSION > 9020
  // CHECK: hipHostFn_t hostFn;
  hipHostFn_t hostFn;
#endif

  // CHECK: hipStreamCallback_t streamCallback;
  hipStreamCallback_t streamCallback;

  // CHECK: hipSurfaceObject_t surfObject;
  hipSurfaceObject_t surfObject;
#if CUDA_VERSION > 11020
  // CHECK: hipSurfaceObject_t surfObject_v1;
  hipSurfaceObject_t surfObject_v1;
#endif

  // CHECK: hipTextureObject_t texObject;
  hipTextureObject_t texObject;
#if CUDA_VERSION > 11020
  // CHECK: hipTextureObject_t texObject_v1;
  hipTextureObject_t texObject_v1;
#endif

  // CHECK: hipUUID uuid;
  hipUUID uuid;

#if CUDA_VERSION > 10020
  // CHECK: hipMemGenericAllocationHandle_t memGenericAllocationHandle_t;
  hipMemGenericAllocationHandle_t memGenericAllocationHandle_t;
#endif

#if CUDA_VERSION > 11030
  // CHECK: hipMemGenericAllocationHandle_t memGenericAllocationHandle_v1;
  hipMemGenericAllocationHandle_t memGenericAllocationHandle_v1;
#endif

  return 0;
}
