// RUN: %run_test hipify "%s" "%t" %hipify_args 2 --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <stdio.h>
#if defined(_WIN32)
  #include "windows.h"
  #include <GL/glew.h>
#endif
#include "cudaGL.h"

int main() {
  printf("12. CUDA Runtime API Functions synthetic test\n");

  size_t bytes = 0;
  int device = 0;
  void* deviceptr = nullptr;
  void* image = nullptr;

  // CHECK: hipError_t result = hipSuccess;
  // CHECK-NEXT: hipStream_t stream;
  hipError_t result = hipSuccess;
  hipStream_t stream;

#if CUDA_VERSION >= 11020
  // CHECK: hipMemPool_t memPool_t;
  hipMemPool_t memPool_t;
#endif

#if CUDA_VERSION >= 11020
  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceGetDefaultMemPool(hipMemPool_t *memPool, int device);
  // HIP: hipError_t hipDeviceGetDefaultMemPool(hipMemPool_t* mem_pool, int device);
  // CHECK: result = hipDeviceGetDefaultMemPool(&memPool_t, device);
  result = hipDeviceGetDefaultMemPool(&memPool_t, device);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceSetMemPool(int device, hipMemPool_t memPool);
  // HIP: hipError_t hipDeviceSetMemPool(int device, hipMemPool_t mem_pool);
  // CHECK: result = hipDeviceSetMemPool(device, memPool_t);
  result = hipDeviceSetMemPool(device, memPool_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceGetMemPool(hipMemPool_t *memPool, int device);
  // HIP: hipError_t hipDeviceGetMemPool(hipMemPool_t* mem_pool, int device);
  // CHECK: result = hipDeviceGetMemPool(&memPool_t, device);
  result = hipDeviceGetMemPool(&memPool_t, device);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMallocAsync(void **devPtr, size_t size, hipStream_t hStream);
  // HIP: hipError_t hipMallocAsync(void** dev_ptr, size_t size, hipStream_t stream);
  // CHECK: result = hipMallocAsync(&deviceptr, bytes, stream);
  result = hipMallocAsync(&deviceptr, bytes, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipFreeAsync(void *devPtr, hipStream_t hStream);
  // HIP: hipError_t hipFreeAsync(void* dev_ptr, hipStream_t stream);
  // CHECK: result = hipFreeAsync(deviceptr, stream);
  result = hipFreeAsync(deviceptr, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolTrimTo(hipMemPool_t memPool, size_t minBytesToKeep);
  // HIP: hipError_t hipMemPoolTrimTo(hipMemPool_t mem_pool, size_t min_bytes_to_hold);
  // CHECK: result = hipMemPoolTrimTo(memPool_t, bytes);
  result = hipMemPoolTrimTo(memPool_t, bytes);

  // CHECK: hipMemPoolAttr memPoolAttr;
  hipMemPoolAttr memPoolAttr;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolSetAttribute(hipMemPool_t memPool, enum hipMemPoolAttr attr, void *value );
  // HIP: hipError_t hipMemPoolSetAttribute(hipMemPool_t mem_pool, hipMemPoolAttr attr, void* value);
  // CHECK: result = hipMemPoolSetAttribute(memPool_t, memPoolAttr, image);
  result = hipMemPoolSetAttribute(memPool_t, memPoolAttr, image);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolGetAttribute(hipMemPool_t memPool, enum hipMemPoolAttr attr, void *value );
  // HIP: hipError_t hipMemPoolGetAttribute(hipMemPool_t mem_pool, hipMemPoolAttr attr, void* value);
  // CHECK: result = hipMemPoolGetAttribute(memPool_t, memPoolAttr, image);
  result = hipMemPoolGetAttribute(memPool_t, memPoolAttr, image);

  // CHECK: hipMemAccessDesc memAccessDesc;
  hipMemAccessDesc memAccessDesc;
  // CHECK: hipMemAccessFlags memAccessFlags;
  hipMemAccessFlags memAccessFlags;
  // CHECK: hipMemLocation memLocation;
  hipMemLocation memLocation;
  // CHECK: hipMemPoolProps memPoolProps;
  hipMemPoolProps memPoolProps;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolSetAccess(hipMemPool_t memPool, const struct hipMemAccessDesc *descList, size_t count);
  // HIP: hipError_t hipMemPoolSetAccess(hipMemPool_t mem_pool, const hipMemAccessDesc* desc_list, size_t count);
  // CHECK: result = hipMemPoolSetAccess(memPool_t, &memAccessDesc, bytes);
  result = hipMemPoolSetAccess(memPool_t, &memAccessDesc, bytes);

  // CUDA: hipError_t extern __host__ hipError_t CUDARTAPI hipMemPoolGetAccess(enum hipMemAccessFlags *flags, hipMemPool_t memPool, struct hipMemLocation *location);
  // HIP: hipError_t hipMemPoolGetAccess(hipMemAccessFlags* flags, hipMemPool_t mem_pool, hipMemLocation* location);
  // CHECK: result = hipMemPoolGetAccess(&memAccessFlags, memPool_t, &memLocation);
  result = hipMemPoolGetAccess(&memAccessFlags, memPool_t, &memLocation);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolCreate(hipMemPool_t *memPool, const struct hipMemPoolProps *poolProps);
  // HIP: hipError_t hipMemPoolCreate(hipMemPool_t* mem_pool, const hipMemPoolProps* pool_props);
  // CHECK: result = hipMemPoolCreate(&memPool_t, &memPoolProps);
  result = hipMemPoolCreate(&memPool_t, &memPoolProps);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolDestroy(hipMemPool_t memPool);
  // HIP: hipError_t hipMemPoolDestroy(hipMemPool_t mem_pool);
  // CHECK: result = hipMemPoolDestroy(memPool_t);
  result = hipMemPoolDestroy(memPool_t);
#endif

  return 0;
}
