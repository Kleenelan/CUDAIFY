// RUN: %run_test hipify "%s" "%t" %hipify_args 2 --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <stdio.h>
#if defined(_WIN32)
  #include "windows.h"
  #include <GL/glew.h>
#endif
#include "cudaGL.h"

int main() {
  printf("12. CUDA Runtime API Functions synthetic test\n");

  size_t bytes = 0;
  int device = 0;
  int deviceId = 0;
  int intVal = 0;
  unsigned int flags = 0;
  void* deviceptr = nullptr;
  void* image = nullptr;
  char* ch = nullptr;

#if defined(_WIN32)
  unsigned long long ull = 0;
#else
  unsigned long ull = 0;
#endif

  // CHECK: hipError_t result = hipSuccess;
  // CHECK-NEXT: hipStream_t stream;
  hipError_t result = hipSuccess;
  hipStream_t stream;

#if CUDA_VERSION >= 10000
  // CHECK: hipHostFn_t hostFn;
  hipHostFn_t hostFn;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipLaunchHostFunc(hipStream_t stream, hipHostFn_t fn, void *userData);
  // HIP: hipError_t hipLaunchHostFunc(hipStream_t stream, hipHostFn_t fn, void* userData);
  // CHECK: result = hipLaunchHostFunc(stream, hostFn, image);
  result = hipLaunchHostFunc(stream, hostFn, image);
#endif

#if CUDA_VERSION >= 10010
  // CHECK: hipStreamCaptureMode streamCaptureMode;
  hipStreamCaptureMode streamCaptureMode;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipThreadExchangeStreamCaptureMode(enum hipStreamCaptureMode *mode);
  // HIP: hipError_t hipThreadExchangeStreamCaptureMode(hipStreamCaptureMode* mode);
  // CHECK: result = hipThreadExchangeStreamCaptureMode(&streamCaptureMode);
  result = hipThreadExchangeStreamCaptureMode(&streamCaptureMode);
#endif

#if CUDA_VERSION >= 11000
  // CHECK: hipKernelNodeAttrID kernelNodeAttrID;
  hipKernelNodeAttrID kernelNodeAttrID;
  // CHECK: hipKernelNodeAttrValue kernelNodeAttrValue;
  hipKernelNodeAttrValue kernelNodeAttrValue;
  // CHECK: hipGraphNode_t graphNode;
  hipGraphNode_t graphNode;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphKernelNodeSetAttribute(hipGraphNode_t hNode, enum hipKernelNodeAttrID attr, const union hipKernelNodeAttrValue* value);
  // HIP: hipError_t hipGraphKernelNodeSetAttribute(hipGraphNode_t hNode, hipKernelNodeAttrID attr, const hipKernelNodeAttrValue* value);
  // CHECK: result = hipGraphKernelNodeSetAttribute(graphNode, kernelNodeAttrID, &kernelNodeAttrValue);
  result = hipGraphKernelNodeSetAttribute(graphNode, kernelNodeAttrID, &kernelNodeAttrValue);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphKernelNodeGetAttribute(hipGraphNode_t hNode, enum hipKernelNodeAttrID attr, union hipKernelNodeAttrValue* value_out);
  // HIP: hipError_t hipGraphKernelNodeGetAttribute(hipGraphNode_t hNode, hipKernelNodeAttrID attr, hipKernelNodeAttrValue* value);
  // CHECK: result = hipGraphKernelNodeGetAttribute(graphNode, kernelNodeAttrID, &kernelNodeAttrValue);
  result = hipGraphKernelNodeGetAttribute(graphNode, kernelNodeAttrID, &kernelNodeAttrValue);
#endif

#if CUDA_VERSION >= 11020
  // CHECK: hipMemPoolAttr memPoolAttr;
  hipMemPoolAttr memPoolAttr;
  // CHECK: hipMemAccessDesc memAccessDesc;
  hipMemAccessDesc memAccessDesc;
  // CHECK: hipMemAccessFlags memAccessFlags;
  hipMemAccessFlags memAccessFlags;
  // CHECK: hipMemLocation memLocation;
  hipMemLocation memLocation;
  // CHECK: hipMemPoolProps memPoolProps;
  hipMemPoolProps memPoolProps;
  // CHECK: hipMemPool_t memPool_t;
  hipMemPool_t memPool_t;
  // CHECK: hipMemAllocationHandleType memAllocationHandleType;
  hipMemAllocationHandleType memAllocationHandleType;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceGetDefaultMemPool(hipMemPool_t *memPool, int device);
  // HIP: hipError_t hipDeviceGetDefaultMemPool(hipMemPool_t* mem_pool, int device);
  // CHECK: result = hipDeviceGetDefaultMemPool(&memPool_t, device);
  result = hipDeviceGetDefaultMemPool(&memPool_t, device);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceSetMemPool(int device, hipMemPool_t memPool);
  // HIP: hipError_t hipDeviceSetMemPool(int device, hipMemPool_t mem_pool);
  // CHECK: result = hipDeviceSetMemPool(device, memPool_t);
  result = hipDeviceSetMemPool(device, memPool_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceGetMemPool(hipMemPool_t *memPool, int device);
  // HIP: hipError_t hipDeviceGetMemPool(hipMemPool_t* mem_pool, int device);
  // CHECK: result = hipDeviceGetMemPool(&memPool_t, device);
  result = hipDeviceGetMemPool(&memPool_t, device);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMallocAsync(void **devPtr, size_t size, hipStream_t hStream);
  // HIP: hipError_t hipMallocAsync(void** dev_ptr, size_t size, hipStream_t stream);
  // CHECK: result = hipMallocAsync(&deviceptr, bytes, stream);
  result = hipMallocAsync(&deviceptr, bytes, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipFreeAsync(void *devPtr, hipStream_t hStream);
  // HIP: hipError_t hipFreeAsync(void* dev_ptr, hipStream_t stream);
  // CHECK: result = hipFreeAsync(deviceptr, stream);
  result = hipFreeAsync(deviceptr, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolTrimTo(hipMemPool_t memPool, size_t minBytesToKeep);
  // HIP: hipError_t hipMemPoolTrimTo(hipMemPool_t mem_pool, size_t min_bytes_to_hold);
  // CHECK: result = hipMemPoolTrimTo(memPool_t, bytes);
  result = hipMemPoolTrimTo(memPool_t, bytes);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolSetAttribute(hipMemPool_t memPool, enum hipMemPoolAttr attr, void *value );
  // HIP: hipError_t hipMemPoolSetAttribute(hipMemPool_t mem_pool, hipMemPoolAttr attr, void* value);
  // CHECK: result = hipMemPoolSetAttribute(memPool_t, memPoolAttr, image);
  result = hipMemPoolSetAttribute(memPool_t, memPoolAttr, image);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolGetAttribute(hipMemPool_t memPool, enum hipMemPoolAttr attr, void *value );
  // HIP: hipError_t hipMemPoolGetAttribute(hipMemPool_t mem_pool, hipMemPoolAttr attr, void* value);
  // CHECK: result = hipMemPoolGetAttribute(memPool_t, memPoolAttr, image);
  result = hipMemPoolGetAttribute(memPool_t, memPoolAttr, image);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolSetAccess(hipMemPool_t memPool, const struct hipMemAccessDesc *descList, size_t count);
  // HIP: hipError_t hipMemPoolSetAccess(hipMemPool_t mem_pool, const hipMemAccessDesc* desc_list, size_t count);
  // CHECK: result = hipMemPoolSetAccess(memPool_t, &memAccessDesc, bytes);
  result = hipMemPoolSetAccess(memPool_t, &memAccessDesc, bytes);

  // CUDA: hipError_t extern __host__ hipError_t CUDARTAPI hipMemPoolGetAccess(enum hipMemAccessFlags *flags, hipMemPool_t memPool, struct hipMemLocation *location);
  // HIP: hipError_t hipMemPoolGetAccess(hipMemAccessFlags* flags, hipMemPool_t mem_pool, hipMemLocation* location);
  // CHECK: result = hipMemPoolGetAccess(&memAccessFlags, memPool_t, &memLocation);
  result = hipMemPoolGetAccess(&memAccessFlags, memPool_t, &memLocation);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolCreate(hipMemPool_t *memPool, const struct hipMemPoolProps *poolProps);
  // HIP: hipError_t hipMemPoolCreate(hipMemPool_t* mem_pool, const hipMemPoolProps* pool_props);
  // CHECK: result = hipMemPoolCreate(&memPool_t, &memPoolProps);
  result = hipMemPoolCreate(&memPool_t, &memPoolProps);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolDestroy(hipMemPool_t memPool);
  // HIP: hipError_t hipMemPoolDestroy(hipMemPool_t mem_pool);
  // CHECK: result = hipMemPoolDestroy(memPool_t);
  result = hipMemPoolDestroy(memPool_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMallocFromPoolAsync(void **ptr, size_t size, hipMemPool_t memPool, hipStream_t stream);
  // HIP: hipError_t hipMallocFromPoolAsync(void** dev_ptr, size_t size, hipMemPool_t mem_pool, hipStream_t stream);
  // CHECK: result = hipMallocFromPoolAsync(&deviceptr, bytes, memPool_t, stream);
  result = hipMallocFromPoolAsync(&deviceptr, bytes, memPool_t, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolExportToShareableHandle(void* shareableHandle, hipMemPool_t memPool, enum hipMemAllocationHandleType handleType, unsigned int flags);
  // HIP: hipError_t hipMemPoolExportToShareableHandle(void* shared_handle, hipMemPool_t mem_pool, hipMemAllocationHandleType handle_type, unsigned int flags);
  // CHECK: result = hipMemPoolExportToShareableHandle(image, memPool_t, memAllocationHandleType, ull);
  result = hipMemPoolExportToShareableHandle(image, memPool_t, memAllocationHandleType, ull);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolImportFromShareableHandle(hipMemPool_t* memPool, void* shareableHandle, enum hipMemAllocationHandleType handleType, unsigned int flags);
  // HIP: hipError_t hipMemPoolImportFromShareableHandle(hipMemPool_t* mem_pool, void* shared_handle, hipMemAllocationHandleType handle_type, unsigned int flags);
  // CHECK: result = hipMemPoolImportFromShareableHandle(&memPool_t, image, memAllocationHandleType, ull);
  result = hipMemPoolImportFromShareableHandle(&memPool_t, image, memAllocationHandleType, ull);

  // CHECK: hipMemPoolPtrExportData memPoolPtrExportData;
  hipMemPoolPtrExportData memPoolPtrExportData;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolExportPointer(struct hipMemPoolPtrExportData *exportData, void *ptr);
  // HIP: hipError_t hipMemPoolExportPointer(hipMemPoolPtrExportData* export_data, void* dev_ptr);
  // CHECK: result = hipMemPoolExportPointer(&memPoolPtrExportData, deviceptr);
  result = hipMemPoolExportPointer(&memPoolPtrExportData, deviceptr);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolImportPointer(void **ptr, hipMemPool_t memPool, struct hipMemPoolPtrExportData *exportData);
  // HIP: hipError_t hipMemPoolImportPointer(void** dev_ptr, hipMemPool_t mem_pool, hipMemPoolPtrExportData* export_data);
  // CHECK: result = hipMemPoolImportPointer(&deviceptr, memPool_t, &memPoolPtrExportData);
  result = hipMemPoolImportPointer(&deviceptr, memPool_t, &memPoolPtrExportData);
#endif

  // CHECK: hipDeviceProp_t DeviceProp;
  hipDeviceProp_t DeviceProp;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipChooseDevice(int *device, const struct hipDeviceProp_t *prop);
  // HIP: hipError_t hipChooseDevice(int* device, const hipDeviceProp_t* prop);
  // CHECK: result = hipChooseDevice(&device, &DeviceProp);
  result = hipChooseDevice(&device, &DeviceProp);

  // CHECK: hipDeviceAttribute_t DeviceAttr;
  hipDeviceAttribute_t DeviceAttr;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceGetAttribute(int *value, enum hipDeviceAttribute_t attr, int device);
  // HIP: hipError_t hipDeviceGetAttribute(int* pi, hipDeviceAttribute_t attr, int deviceId);
  // CHECK: result = hipDeviceGetAttribute(&device, DeviceAttr, deviceId);
  result = hipDeviceGetAttribute(&device, DeviceAttr, deviceId);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceGetByPCIBusId(int *device, const char *pciBusId);
  // HIP: hipError_t hipDeviceGetByPCIBusId(int* device, const char* pciBusId);
  // CHECK: result = hipDeviceGetByPCIBusId(&device, ch);
  result = hipDeviceGetByPCIBusId(&device, ch);

  // CHECK: hipFuncCache_t FuncCache;
  hipFuncCache_t FuncCache;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceGetCacheConfig(enum hipFuncCache_t *pCacheConfig);
  // HIP: hipError_t hipDeviceGetCacheConfig(hipFuncCache_t* cacheConfig);
  // CHECK: result = hipDeviceGetCacheConfig(&FuncCache);
  result = hipDeviceGetCacheConfig(&FuncCache);

  // CHECK: hipLimit_t Limit;
  hipLimit_t Limit;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceGetLimit(size_t *pValue, enum hipLimit_t limit);
  // HIP: hipError_t hipDeviceGetLimit(size_t* pValue, enum hipLimit_t limit);
  // CHECK: result = hipDeviceGetLimit(&bytes, Limit);
  result = hipDeviceGetLimit(&bytes, Limit);

#if CUDA_VERSION >= 8000
  // CHECK: hipDeviceP2PAttr DeviceP2PAttr;
  hipDeviceP2PAttr DeviceP2PAttr;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceGetP2PAttribute(int *value, enum hipDeviceP2PAttr attr, int srcDevice, int dstDevice);
  // HIP: hipError_t hipDeviceGetP2PAttribute(int* value, hipDeviceP2PAttr attr, int srcDevice, int dstDevice);
  // CHECK: result = hipDeviceGetP2PAttribute(&intVal, DeviceP2PAttr, device, deviceId);
  result = hipDeviceGetP2PAttribute(&intVal, DeviceP2PAttr, device, deviceId);
#endif

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceGetPCIBusId(char *pciBusId, int len, int device);
  // HIP: hipError_t hipError_t hipDeviceGetPCIBusId(char* pciBusId, int len, int device);
  // CHECK: result = hipDeviceGetPCIBusId(ch, intVal, device);
  result = hipDeviceGetPCIBusId(ch, intVal, device);

  // CHECK: hipSharedMemConfig SharedMemConfig;
  hipSharedMemConfig SharedMemConfig;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceGetSharedMemConfig(enum hipSharedMemConfig *pConfig);
  // HIP: hipError_t hipDeviceGetSharedMemConfig(hipSharedMemConfig* pConfig);
  // CHECK: result = hipDeviceGetSharedMemConfig(&SharedMemConfig);
  result = hipDeviceGetSharedMemConfig(&SharedMemConfig);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceGetStreamPriorityRange(int *leastPriority, int *greatestPriority);
  // HIP: hipError_t hipDeviceGetStreamPriorityRange(int* leastPriority, int* greatestPriority);
  // CHECK: result = hipDeviceGetStreamPriorityRange(&deviceId, &intVal);
  result = hipDeviceGetStreamPriorityRange(&deviceId, &intVal);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceReset(void);
  // HIP: hipError_t hipError_t hipDeviceReset(void);
  // CHECK: result = hipDeviceReset();
  result = hipDeviceReset();

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceSetCacheConfig(enum hipFuncCache_t cacheConfig);
  // HIP: hipError_t hipDeviceSetCacheConfig(hipFuncCache_t cacheConfig);
  // CHECK: result = hipDeviceSetCacheConfig(FuncCache);
  result = hipDeviceSetCacheConfig(FuncCache);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceSetSharedMemConfig(enum hipSharedMemConfig config);
  // HIP: hipError_t hipDeviceSetSharedMemConfig(hipSharedMemConfig config);
  // CHECK: result = hipDeviceSetSharedMemConfig(SharedMemConfig);
  result = hipDeviceSetSharedMemConfig(SharedMemConfig);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceSynchronize(void);
  // HIP: hipError_t hipDeviceSynchronize(void);
  // CHECK: result = hipDeviceSynchronize();
  result = hipDeviceSynchronize();

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipGetDevice(int *device);
  // HIP: hipError_t hipGetDevice(int* deviceId);
  // CHECK: result = hipGetDevice(&deviceId);
  result = hipGetDevice(&deviceId);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipGetDeviceCount(int *count);
  // HIP: hipError_t hipGetDeviceCount(int* count);
  // CHECK: result = hipGetDeviceCount(&deviceId);
  result = hipGetDeviceCount(&deviceId);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGetDeviceFlags( unsigned int *flags );
  // HIP: hipError_t hipGetDeviceFlags(unsigned int* flags);
  // CHECK: result = hipGetDeviceFlags(&flags);
  result = hipGetDeviceFlags(&flags);

  return 0;
}
