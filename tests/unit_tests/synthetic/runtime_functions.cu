// RUN: %run_test hipify "%s" "%t" %hipify_args 2 --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <stdio.h>
#if defined(_WIN32)
  #include "windows.h"
  #include <GL/glew.h>
#endif
#include "cudaGL.h"

int main() {
  printf("12. CUDA Runtime API Functions synthetic test\n");

  size_t bytes = 0;
  int device = 0;
  int deviceId = 0;
  int intVal = 0;
  unsigned int flags = 0;
  float ms = 0;
  void* deviceptr = nullptr;
  void* image = nullptr;
  void* func = nullptr;
  char* ch = nullptr;
  const char* const_ch = nullptr;
  dim3 gridDim;
  dim3 blockDim;

#if defined(_WIN32)
  unsigned long long ull = 0;
#else
  unsigned long ull = 0;
#endif
  unsigned long long ull_2 = 0;

  // CHECK: hipError_t result = hipSuccess;
  // CHECK-NEXT: hipError_t Error_t;
  // CHECK-NEXT: hipStream_t stream;
  hipError_t result = hipSuccess;
  hipError_t Error_t;
  hipStream_t stream;

#if CUDA_VERSION >= 8000
  // CHECK: hipDeviceP2PAttr DeviceP2PAttr;
  hipDeviceP2PAttr DeviceP2PAttr;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceGetP2PAttribute(int *value, enum hipDeviceP2PAttr attr, int srcDevice, int dstDevice);
  // HIP: hipError_t hipDeviceGetP2PAttribute(int* value, hipDeviceP2PAttr attr, int srcDevice, int dstDevice);
  // CHECK: result = hipDeviceGetP2PAttribute(&intVal, DeviceP2PAttr, device, deviceId);
  result = hipDeviceGetP2PAttribute(&intVal, DeviceP2PAttr, device, deviceId);
#endif

#if CUDA_VERSION >= 10000
  // CHECK: hipHostFn_t hostFn;
  hipHostFn_t hostFn;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipLaunchHostFunc(hipStream_t stream, hipHostFn_t fn, void *userData);
  // HIP: hipError_t hipLaunchHostFunc(hipStream_t stream, hipHostFn_t fn, void* userData);
  // CHECK: result = hipLaunchHostFunc(stream, hostFn, image);
  result = hipLaunchHostFunc(stream, hostFn, image);

  // CHECK: hipStreamCaptureMode StreamCaptureMode;
  hipStreamCaptureMode StreamCaptureMode;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipStreamBeginCapture(hipStream_t stream, enum hipStreamCaptureMode mode);
  // HIP: hipError_t hipStreamBeginCapture(hipStream_t stream, hipStreamCaptureMode mode);
  // CHECK: result = hipStreamBeginCapture(stream, StreamCaptureMode);
  result = hipStreamBeginCapture(stream, StreamCaptureMode);

  // CHECK: hipGraph_t Graph_t;
  hipGraph_t Graph_t;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipStreamEndCapture(hipStream_t stream, hipGraph_t *pGraph);
  // HIP: hipError_t hipStreamEndCapture(hipStream_t stream, hipGraph_t* pGraph);
  // CHECK: result = hipStreamEndCapture(stream, &Graph_t);
  result = hipStreamEndCapture(stream, &Graph_t);

  // CHECK: hipStreamCaptureStatus StreamCaptureStatus;
  hipStreamCaptureStatus StreamCaptureStatus;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipStreamIsCapturing(hipStream_t stream, enum hipStreamCaptureStatus *pCaptureStatus);
  // HIP: hipError_t hipStreamIsCapturing(hipStream_t stream, hipStreamCaptureStatus* pCaptureStatus);
  // CHECK: result = hipStreamIsCapturing(stream, &StreamCaptureStatus);
  result = hipStreamIsCapturing(stream, &StreamCaptureStatus);

  // CHECK: hipExternalMemory_t ExternalMemory_t;
  hipExternalMemory_t ExternalMemory_t;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDestroyExternalMemory(hipExternalMemory_t extMem);
  // HIP: hipError_t hipDestroyExternalMemory(hipExternalMemory_t extMem);
  // CHECK: result = hipDestroyExternalMemory(ExternalMemory_t);
  result = hipDestroyExternalMemory(ExternalMemory_t);

  // CHECK: hipExternalSemaphore_t ExternalSemaphore_t;
  hipExternalSemaphore_t ExternalSemaphore_t;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDestroyExternalSemaphore(hipExternalSemaphore_t extSem);
  // HIP: hipError_t hipDestroyExternalSemaphore(hipExternalSemaphore_t extSem);
  // CHECK: result = hipDestroyExternalSemaphore(ExternalSemaphore_t);
  result = hipDestroyExternalSemaphore(ExternalSemaphore_t);

  // CHECK: hipExternalMemoryBufferDesc ExternalMemoryBufferDesc;
  hipExternalMemoryBufferDesc ExternalMemoryBufferDesc;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipExternalMemoryGetMappedBuffer(void **devPtr, hipExternalMemory_t extMem, const struct hipExternalMemoryBufferDesc *bufferDesc);
  // HIP: hipError_t hipExternalMemoryGetMappedBuffer(void **devPtr, hipExternalMemory_t extMem, const hipExternalMemoryBufferDesc *bufferDesc);
  // CHECK: result = hipExternalMemoryGetMappedBuffer(&deviceptr, ExternalMemory_t, &ExternalMemoryBufferDesc);
  result = hipExternalMemoryGetMappedBuffer(&deviceptr, ExternalMemory_t, &ExternalMemoryBufferDesc);

  // CHECK: hipExternalMemoryHandleDesc ExternalMemoryHandleDesc;
  hipExternalMemoryHandleDesc ExternalMemoryHandleDesc;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipImportExternalMemory(hipExternalMemory_t *extMem_out, const struct hipExternalMemoryHandleDesc *memHandleDesc);
  // HIP: hipError_t hipImportExternalMemory(hipExternalMemory_t* extMem_out, const hipExternalMemoryHandleDesc* memHandleDesc);
  // CHECK: result = hipImportExternalMemory(&ExternalMemory_t, &ExternalMemoryHandleDesc);
  result = hipImportExternalMemory(&ExternalMemory_t, &ExternalMemoryHandleDesc);

  // CHECK: hipExternalSemaphoreHandleDesc ExternalSemaphoreHandleDesc;
  hipExternalSemaphoreHandleDesc ExternalSemaphoreHandleDesc;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipImportExternalSemaphore(hipExternalSemaphore_t *extSem_out, const struct hipExternalSemaphoreHandleDesc *semHandleDesc);
  // HIP: hipError_t hipImportExternalSemaphore(hipExternalSemaphore_t* extSem_out, const hipExternalSemaphoreHandleDesc* semHandleDesc);
  // CHECK: result = hipImportExternalSemaphore(&ExternalSemaphore_t, &ExternalSemaphoreHandleDesc);
  result = hipImportExternalSemaphore(&ExternalSemaphore_t, &ExternalSemaphoreHandleDesc);

  // CHECK: hipExternalSemaphoreSignalParams ExternalSemaphoreSignalParams;
  hipExternalSemaphoreSignalParams ExternalSemaphoreSignalParams;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipSignalExternalSemaphoresAsync(const hipExternalSemaphore_t *extSemArray, const struct hipExternalSemaphoreSignalParams *paramsArray, unsigned int numExtSems, hipStream_t stream __dv(0));
  // HIP: hipError_t hipSignalExternalSemaphoresAsync(const hipExternalSemaphore_t* extSemArray, const hipExternalSemaphoreSignalParams* paramsArray, unsigned int numExtSems, hipStream_t stream);
  // CHECK: result = hipSignalExternalSemaphoresAsync(&ExternalSemaphore_t, &ExternalSemaphoreSignalParams, flags, stream);
  result = hipSignalExternalSemaphoresAsync(&ExternalSemaphore_t, &ExternalSemaphoreSignalParams, flags, stream);

  // CHECK: hipExternalSemaphoreWaitParams ExternalSemaphoreWaitParams;
  hipExternalSemaphoreWaitParams ExternalSemaphoreWaitParams;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipWaitExternalSemaphoresAsync(const hipExternalSemaphore_t *extSemArray, const struct hipExternalSemaphoreWaitParams *paramsArray, unsigned int numExtSems, hipStream_t stream __dv(0));
  // HIP: hipError_t hipWaitExternalSemaphoresAsync(const hipExternalSemaphore_t* extSemArray, const hipExternalSemaphoreWaitParams* paramsArray, unsigned int numExtSems, hipStream_t stream);
  // CHECK: result = hipWaitExternalSemaphoresAsync(&ExternalSemaphore_t, &ExternalSemaphoreWaitParams, flags, stream);
  result = hipWaitExternalSemaphoresAsync(&ExternalSemaphore_t, &ExternalSemaphoreWaitParams, flags, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipLaunchHostFunc(hipStream_t stream, hipHostFn_t fn, void *userData);
  // HIP: hipError_t hipLaunchHostFunc(hipStream_t stream, hipHostFn_t fn, void* userData);
  // CHECK: result = hipLaunchHostFunc(stream, hostFn, image);
  result = hipLaunchHostFunc(stream, hostFn, image);
#endif

#if CUDA_VERSION >= 10010
  // CHECK: hipStreamCaptureMode streamCaptureMode;
  hipStreamCaptureMode streamCaptureMode;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipThreadExchangeStreamCaptureMode(enum hipStreamCaptureMode *mode);
  // HIP: hipError_t hipThreadExchangeStreamCaptureMode(hipStreamCaptureMode* mode);
  // CHECK: result = hipThreadExchangeStreamCaptureMode(&streamCaptureMode);
  result = hipThreadExchangeStreamCaptureMode(&streamCaptureMode);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipStreamGetCaptureInfo(hipStream_t stream, enum hipStreamCaptureStatus *pCaptureStatus, unsigned long long *pId);
  // HIP: hipError_t hipStreamGetCaptureInfo(hipStream_t stream, hipStreamCaptureStatus* pCaptureStatus, unsigned long long* pId);
  // CHECK: result = hipStreamGetCaptureInfo(stream, &StreamCaptureStatus, &ull_2);
  result = hipStreamGetCaptureInfo(stream, &StreamCaptureStatus, &ull_2);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipThreadExchangeStreamCaptureMode(enum hipStreamCaptureMode *mode);
  // HIP: hipError_t hipThreadExchangeStreamCaptureMode(hipStreamCaptureMode* mode);
  // CHECK: result = hipThreadExchangeStreamCaptureMode(&streamCaptureMode);
  result = hipThreadExchangeStreamCaptureMode(&streamCaptureMode);
#endif

#if CUDA_VERSION >= 11000
  // CHECK: hipKernelNodeAttrID kernelNodeAttrID;
  hipKernelNodeAttrID kernelNodeAttrID;
  // CHECK: hipKernelNodeAttrValue kernelNodeAttrValue;
  hipKernelNodeAttrValue kernelNodeAttrValue;
  // CHECK: hipGraphNode_t graphNode;
  hipGraphNode_t graphNode;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphKernelNodeSetAttribute(hipGraphNode_t hNode, enum hipKernelNodeAttrID attr, const union hipKernelNodeAttrValue* value);
  // HIP: hipError_t hipGraphKernelNodeSetAttribute(hipGraphNode_t hNode, hipKernelNodeAttrID attr, const hipKernelNodeAttrValue* value);
  // CHECK: result = hipGraphKernelNodeSetAttribute(graphNode, kernelNodeAttrID, &kernelNodeAttrValue);
  result = hipGraphKernelNodeSetAttribute(graphNode, kernelNodeAttrID, &kernelNodeAttrValue);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGraphKernelNodeGetAttribute(hipGraphNode_t hNode, enum hipKernelNodeAttrID attr, union hipKernelNodeAttrValue* value_out);
  // HIP: hipError_t hipGraphKernelNodeGetAttribute(hipGraphNode_t hNode, hipKernelNodeAttrID attr, hipKernelNodeAttrValue* value);
  // CHECK: result = hipGraphKernelNodeGetAttribute(graphNode, kernelNodeAttrID, &kernelNodeAttrValue);
  result = hipGraphKernelNodeGetAttribute(graphNode, kernelNodeAttrID, &kernelNodeAttrValue);
#endif

#if CUDA_VERSION >= 11020
  // CHECK: hipMemPoolAttr memPoolAttr;
  hipMemPoolAttr memPoolAttr;
  // CHECK: hipMemAccessDesc memAccessDesc;
  hipMemAccessDesc memAccessDesc;
  // CHECK: hipMemAccessFlags memAccessFlags;
  hipMemAccessFlags memAccessFlags;
  // CHECK: hipMemLocation memLocation;
  hipMemLocation memLocation;
  // CHECK: hipMemPoolProps memPoolProps;
  hipMemPoolProps memPoolProps;
  // CHECK: hipMemPool_t memPool_t;
  hipMemPool_t memPool_t;
  // CHECK: hipMemAllocationHandleType memAllocationHandleType;
  hipMemAllocationHandleType memAllocationHandleType;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceGetDefaultMemPool(hipMemPool_t *memPool, int device);
  // HIP: hipError_t hipDeviceGetDefaultMemPool(hipMemPool_t* mem_pool, int device);
  // CHECK: result = hipDeviceGetDefaultMemPool(&memPool_t, device);
  result = hipDeviceGetDefaultMemPool(&memPool_t, device);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceSetMemPool(int device, hipMemPool_t memPool);
  // HIP: hipError_t hipDeviceSetMemPool(int device, hipMemPool_t mem_pool);
  // CHECK: result = hipDeviceSetMemPool(device, memPool_t);
  result = hipDeviceSetMemPool(device, memPool_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceGetMemPool(hipMemPool_t *memPool, int device);
  // HIP: hipError_t hipDeviceGetMemPool(hipMemPool_t* mem_pool, int device);
  // CHECK: result = hipDeviceGetMemPool(&memPool_t, device);
  result = hipDeviceGetMemPool(&memPool_t, device);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMallocAsync(void **devPtr, size_t size, hipStream_t hStream);
  // HIP: hipError_t hipMallocAsync(void** dev_ptr, size_t size, hipStream_t stream);
  // CHECK: result = hipMallocAsync(&deviceptr, bytes, stream);
  result = hipMallocAsync(&deviceptr, bytes, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipFreeAsync(void *devPtr, hipStream_t hStream);
  // HIP: hipError_t hipFreeAsync(void* dev_ptr, hipStream_t stream);
  // CHECK: result = hipFreeAsync(deviceptr, stream);
  result = hipFreeAsync(deviceptr, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolTrimTo(hipMemPool_t memPool, size_t minBytesToKeep);
  // HIP: hipError_t hipMemPoolTrimTo(hipMemPool_t mem_pool, size_t min_bytes_to_hold);
  // CHECK: result = hipMemPoolTrimTo(memPool_t, bytes);
  result = hipMemPoolTrimTo(memPool_t, bytes);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolSetAttribute(hipMemPool_t memPool, enum hipMemPoolAttr attr, void *value );
  // HIP: hipError_t hipMemPoolSetAttribute(hipMemPool_t mem_pool, hipMemPoolAttr attr, void* value);
  // CHECK: result = hipMemPoolSetAttribute(memPool_t, memPoolAttr, image);
  result = hipMemPoolSetAttribute(memPool_t, memPoolAttr, image);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolGetAttribute(hipMemPool_t memPool, enum hipMemPoolAttr attr, void *value );
  // HIP: hipError_t hipMemPoolGetAttribute(hipMemPool_t mem_pool, hipMemPoolAttr attr, void* value);
  // CHECK: result = hipMemPoolGetAttribute(memPool_t, memPoolAttr, image);
  result = hipMemPoolGetAttribute(memPool_t, memPoolAttr, image);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolSetAccess(hipMemPool_t memPool, const struct hipMemAccessDesc *descList, size_t count);
  // HIP: hipError_t hipMemPoolSetAccess(hipMemPool_t mem_pool, const hipMemAccessDesc* desc_list, size_t count);
  // CHECK: result = hipMemPoolSetAccess(memPool_t, &memAccessDesc, bytes);
  result = hipMemPoolSetAccess(memPool_t, &memAccessDesc, bytes);

  // CUDA: hipError_t extern __host__ hipError_t CUDARTAPI hipMemPoolGetAccess(enum hipMemAccessFlags *flags, hipMemPool_t memPool, struct hipMemLocation *location);
  // HIP: hipError_t hipMemPoolGetAccess(hipMemAccessFlags* flags, hipMemPool_t mem_pool, hipMemLocation* location);
  // CHECK: result = hipMemPoolGetAccess(&memAccessFlags, memPool_t, &memLocation);
  result = hipMemPoolGetAccess(&memAccessFlags, memPool_t, &memLocation);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolCreate(hipMemPool_t *memPool, const struct hipMemPoolProps *poolProps);
  // HIP: hipError_t hipMemPoolCreate(hipMemPool_t* mem_pool, const hipMemPoolProps* pool_props);
  // CHECK: result = hipMemPoolCreate(&memPool_t, &memPoolProps);
  result = hipMemPoolCreate(&memPool_t, &memPoolProps);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolDestroy(hipMemPool_t memPool);
  // HIP: hipError_t hipMemPoolDestroy(hipMemPool_t mem_pool);
  // CHECK: result = hipMemPoolDestroy(memPool_t);
  result = hipMemPoolDestroy(memPool_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMallocFromPoolAsync(void **ptr, size_t size, hipMemPool_t memPool, hipStream_t stream);
  // HIP: hipError_t hipMallocFromPoolAsync(void** dev_ptr, size_t size, hipMemPool_t mem_pool, hipStream_t stream);
  // CHECK: result = hipMallocFromPoolAsync(&deviceptr, bytes, memPool_t, stream);
  result = hipMallocFromPoolAsync(&deviceptr, bytes, memPool_t, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolExportToShareableHandle(void* shareableHandle, hipMemPool_t memPool, enum hipMemAllocationHandleType handleType, unsigned int flags);
  // HIP: hipError_t hipMemPoolExportToShareableHandle(void* shared_handle, hipMemPool_t mem_pool, hipMemAllocationHandleType handle_type, unsigned int flags);
  // CHECK: result = hipMemPoolExportToShareableHandle(image, memPool_t, memAllocationHandleType, ull);
  result = hipMemPoolExportToShareableHandle(image, memPool_t, memAllocationHandleType, ull);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolImportFromShareableHandle(hipMemPool_t* memPool, void* shareableHandle, enum hipMemAllocationHandleType handleType, unsigned int flags);
  // HIP: hipError_t hipMemPoolImportFromShareableHandle(hipMemPool_t* mem_pool, void* shared_handle, hipMemAllocationHandleType handle_type, unsigned int flags);
  // CHECK: result = hipMemPoolImportFromShareableHandle(&memPool_t, image, memAllocationHandleType, ull);
  result = hipMemPoolImportFromShareableHandle(&memPool_t, image, memAllocationHandleType, ull);

  // CHECK: hipMemPoolPtrExportData memPoolPtrExportData;
  hipMemPoolPtrExportData memPoolPtrExportData;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolExportPointer(struct hipMemPoolPtrExportData *exportData, void *ptr);
  // HIP: hipError_t hipMemPoolExportPointer(hipMemPoolPtrExportData* export_data, void* dev_ptr);
  // CHECK: result = hipMemPoolExportPointer(&memPoolPtrExportData, deviceptr);
  result = hipMemPoolExportPointer(&memPoolPtrExportData, deviceptr);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolImportPointer(void **ptr, hipMemPool_t memPool, struct hipMemPoolPtrExportData *exportData);
  // HIP: hipError_t hipMemPoolImportPointer(void** dev_ptr, hipMemPool_t mem_pool, hipMemPoolPtrExportData* export_data);
  // CHECK: result = hipMemPoolImportPointer(&deviceptr, memPool_t, &memPoolPtrExportData);
  result = hipMemPoolImportPointer(&deviceptr, memPool_t, &memPoolPtrExportData);
#endif

  // CHECK: hipDeviceProp_t DeviceProp;
  hipDeviceProp_t DeviceProp;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipChooseDevice(int *device, const struct hipDeviceProp_t *prop);
  // HIP: hipError_t hipChooseDevice(int* device, const hipDeviceProp_t* prop);
  // CHECK: result = hipChooseDevice(&device, &DeviceProp);
  result = hipChooseDevice(&device, &DeviceProp);

  // CHECK: hipDeviceAttribute_t DeviceAttr;
  hipDeviceAttribute_t DeviceAttr;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceGetAttribute(int *value, enum hipDeviceAttribute_t attr, int device);
  // HIP: hipError_t hipDeviceGetAttribute(int* pi, hipDeviceAttribute_t attr, int deviceId);
  // CHECK: result = hipDeviceGetAttribute(&device, DeviceAttr, deviceId);
  result = hipDeviceGetAttribute(&device, DeviceAttr, deviceId);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceGetByPCIBusId(int *device, const char *pciBusId);
  // HIP: hipError_t hipDeviceGetByPCIBusId(int* device, const char* pciBusId);
  // CHECK: result = hipDeviceGetByPCIBusId(&device, ch);
  result = hipDeviceGetByPCIBusId(&device, ch);

  // CHECK: hipFuncCache_t FuncCache;
  hipFuncCache_t FuncCache;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceGetCacheConfig(enum hipFuncCache_t *pCacheConfig);
  // HIP: hipError_t hipDeviceGetCacheConfig(hipFuncCache_t* cacheConfig);
  // CHECK: result = hipDeviceGetCacheConfig(&FuncCache);
  result = hipDeviceGetCacheConfig(&FuncCache);

  // CHECK: hipLimit_t Limit;
  hipLimit_t Limit;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceGetLimit(size_t *pValue, enum hipLimit_t limit);
  // HIP: hipError_t hipDeviceGetLimit(size_t* pValue, enum hipLimit_t limit);
  // CHECK: result = hipDeviceGetLimit(&bytes, Limit);
  result = hipDeviceGetLimit(&bytes, Limit);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceGetPCIBusId(char *pciBusId, int len, int device);
  // HIP: hipError_t hipError_t hipDeviceGetPCIBusId(char* pciBusId, int len, int device);
  // CHECK: result = hipDeviceGetPCIBusId(ch, intVal, device);
  result = hipDeviceGetPCIBusId(ch, intVal, device);

  // CHECK: hipSharedMemConfig SharedMemConfig;
  hipSharedMemConfig SharedMemConfig;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceGetSharedMemConfig(enum hipSharedMemConfig *pConfig);
  // HIP: hipError_t hipDeviceGetSharedMemConfig(hipSharedMemConfig* pConfig);
  // CHECK: result = hipDeviceGetSharedMemConfig(&SharedMemConfig);
  result = hipDeviceGetSharedMemConfig(&SharedMemConfig);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceGetStreamPriorityRange(int *leastPriority, int *greatestPriority);
  // HIP: hipError_t hipDeviceGetStreamPriorityRange(int* leastPriority, int* greatestPriority);
  // CHECK: result = hipDeviceGetStreamPriorityRange(&deviceId, &intVal);
  result = hipDeviceGetStreamPriorityRange(&deviceId, &intVal);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceReset(void);
  // HIP: hipError_t hipError_t hipDeviceReset(void);
  // CHECK: result = hipDeviceReset();
  result = hipDeviceReset();

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceSetCacheConfig(enum hipFuncCache_t cacheConfig);
  // HIP: hipError_t hipDeviceSetCacheConfig(hipFuncCache_t cacheConfig);
  // CHECK: result = hipDeviceSetCacheConfig(FuncCache);
  result = hipDeviceSetCacheConfig(FuncCache);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceSetSharedMemConfig(enum hipSharedMemConfig config);
  // HIP: hipError_t hipDeviceSetSharedMemConfig(hipSharedMemConfig config);
  // CHECK: result = hipDeviceSetSharedMemConfig(SharedMemConfig);
  result = hipDeviceSetSharedMemConfig(SharedMemConfig);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipDeviceSynchronize(void);
  // HIP: hipError_t hipDeviceSynchronize(void);
  // CHECK: result = hipDeviceSynchronize();
  result = hipDeviceSynchronize();

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipGetDevice(int *device);
  // HIP: hipError_t hipGetDevice(int* deviceId);
  // CHECK: result = hipGetDevice(&deviceId);
  result = hipGetDevice(&deviceId);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipGetDeviceCount(int *count);
  // HIP: hipError_t hipGetDeviceCount(int* count);
  // CHECK: result = hipGetDeviceCount(&deviceId);
  result = hipGetDeviceCount(&deviceId);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipGetDeviceFlags( unsigned int *flags );
  // HIP: hipError_t hipGetDeviceFlags(unsigned int* flags);
  // CHECK: result = hipGetDeviceFlags(&flags);
  result = hipGetDeviceFlags(&flags);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipGetDeviceProperties(struct hipDeviceProp_t *prop, int device);
  // HIP: hipError_t hipGetDeviceProperties(hipDeviceProp_t* prop, int deviceId);
  // CHECK: result = hipGetDeviceProperties(&DeviceProp, deviceId);
  result = hipGetDeviceProperties(&DeviceProp, deviceId);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipIpcCloseMemHandle(void *devPtr);
  // HIP: hipError_t hipError_t hipIpcCloseMemHandle(void* devPtr);
  // CHECK: result = hipIpcCloseMemHandle(deviceptr);
  result = hipIpcCloseMemHandle(deviceptr);

  // CHECK: hipIpcEventHandle_t IpcEventHandle_t;
  hipIpcEventHandle_t IpcEventHandle_t;

  // CHECK: hipEvent_t Event_t;
  // CHECK-Next: hipEvent_t Event_2;
  hipEvent_t Event_t;
  hipEvent_t Event_2;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipIpcGetEventHandle(hipIpcEventHandle_t *handle, hipEvent_t event);
  // HIP: hipError_t hipIpcGetEventHandle(hipIpcEventHandle_t* handle, hipEvent_t event);
  // CHECK: result = hipIpcGetEventHandle(&IpcEventHandle_t, Event_t);
  result = hipIpcGetEventHandle(&IpcEventHandle_t, Event_t);

  // CHECK: hipIpcMemHandle_t IpcMemHandle_t;
  hipIpcMemHandle_t IpcMemHandle_t;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipIpcGetMemHandle(hipIpcMemHandle_t *handle, void *devPtr);
  // HIP: hipError_t hipIpcGetMemHandle(hipIpcMemHandle_t* handle, void* devPtr);
  // CHECK: result = hipIpcGetMemHandle(&IpcMemHandle_t, deviceptr);
  result = hipIpcGetMemHandle(&IpcMemHandle_t, deviceptr);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipIpcOpenEventHandle(hipEvent_t *event, hipIpcEventHandle_t handle);
  // HIP: hipError_t hipIpcOpenEventHandle(hipEvent_t* event, hipIpcEventHandle_t handle);
  // CHECK: result = hipIpcOpenEventHandle(&Event_t, IpcEventHandle_t);
  result = hipIpcOpenEventHandle(&Event_t, IpcEventHandle_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipIpcOpenMemHandle(void **devPtr, hipIpcMemHandle_t handle, unsigned int flags);
  // HIP: hipError_t hipIpcOpenMemHandle(void** devPtr, hipIpcMemHandle_t handle, unsigned int flags);
  // CHECK: result = hipIpcOpenMemHandle(&deviceptr, IpcMemHandle_t, flags);
  result = hipIpcOpenMemHandle(&deviceptr, IpcMemHandle_t, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipSetDevice(int device);
  // HIP: hipError_t hipSetDevice(int deviceId);
  // CHECK: result = hipSetDevice(deviceId);
  result = hipSetDevice(deviceId);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipSetDeviceFlags( unsigned int flags );
  // HIP: hipError_t hipSetDeviceFlags(unsigned flags);
  // CHECK: result = hipSetDeviceFlags(flags);
  result = hipSetDeviceFlags(flags);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipDeviceReset(void);
  // HIP: hipError_t hipDeviceReset(void);
  // CHECK: result = hipDeviceReset();
  result = hipDeviceReset();

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipDeviceGetCacheConfig(enum hipFuncCache_t *pCacheConfig);
  // HIP: hipError_t hipDeviceGetCacheConfig(hipFuncCache_t* cacheConfig);
  // CHECK: result = hipDeviceGetCacheConfig(&FuncCache);
  result = hipDeviceGetCacheConfig(&FuncCache);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipDeviceSetCacheConfig(enum hipFuncCache_t cacheConfig);
  // HIP: hipError_t hipError_t hipDeviceSetCacheConfig(hipFuncCache_t cacheConfig);
  // CHECK: result = hipDeviceSetCacheConfig(FuncCache);
  result = hipDeviceSetCacheConfig(FuncCache);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipDeviceSynchronize(void);
  // HIP: hipError_t hipError_t hipDeviceSynchronize(void);
  // CHECK: result = hipDeviceSynchronize();
  result = hipDeviceSynchronize();

  // CUDA: extern __host__ __cudart_builtin__ const char* CUDARTAPI hipGetErrorName(hipError_t error);
  // HIP: const char* hipGetErrorName(hipError_t hip_error);
  // CHECK: const_ch = hipGetErrorName(Error_t);
  const_ch = hipGetErrorName(Error_t);

  // CUDA: extern __host__ __cudart_builtin__ const char* CUDARTAPI hipGetErrorString(hipError_t error);
  // HIP: const char* hipGetErrorString(hipError_t hipError);
  // CHECK: const_ch = hipGetErrorString(Error_t);
  const_ch = hipGetErrorString(Error_t);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipGetLastError(void);
  // HIP: hipError_t hipGetLastError(void);
  // CHECK: result = hipGetLastError();
  result = hipGetLastError();

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipPeekAtLastError(void);
  // HIP: hipError_t hipPeekAtLastError(void);
  // CHECK: result = hipPeekAtLastError();
  result = hipPeekAtLastError();

  // CHECK: hipStreamCallback_t StreamCallback_t;
  hipStreamCallback_t StreamCallback_t;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipStreamAddCallback(hipStream_t stream, hipStreamCallback_t callback, void* userData, unsigned int flags);
  // HIP: hipError_t hipStreamAddCallback(hipStream_t stream, hipStreamCallback_t callback, void* userData, unsigned int flags);
  // CHECK: result = hipStreamAddCallback(stream, StreamCallback_t, image, flags);
  result = hipStreamAddCallback(stream, StreamCallback_t, image, flags);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipStreamAttachMemAsync(hipStream_t stream, void *devPtr, size_t length __dv(0), unsigned int flags = hipMemAttachSingle);
  // HIP: hipError_t hipStreamAttachMemAsync(hipStream_t stream, void* dev_ptr, size_t length __dparm(0), unsigned int flags __dparm(hipMemAttachSingle));
  // CHECK: result = hipStreamAttachMemAsync(stream, deviceptr, bytes, flags);
  result = hipStreamAttachMemAsync(stream, deviceptr, bytes, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipStreamCreate(hipStream_t *pStream);
  // HIP: hipError_t hipStreamCreate(hipStream_t* stream);
  // CHECK: result = hipStreamCreate(&stream);
  result = hipStreamCreate(&stream);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipStreamCreateWithFlags(hipStream_t *pStream, unsigned int flags);
  // HIP: hipError_t hipStreamCreateWithFlags(hipStream_t* stream, unsigned int flags);
  // CHECK: result = hipStreamCreateWithFlags(&stream, flags);
  result = hipStreamCreateWithFlags(&stream, flags);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipStreamCreateWithPriority(hipStream_t *pStream, unsigned int flags, int priority);
  // HIP: hipError_t hipStreamCreateWithPriority(hipStream_t* stream, unsigned int flags);
  // CHECK: result = hipStreamCreateWithPriority(&stream, flags, intVal);
  result = hipStreamCreateWithPriority(&stream, flags, intVal);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipStreamDestroy(hipStream_t stream);
  // HIP: hipError_t hipStreamDestroy(hipStream_t stream);
  // CHECK: result = hipStreamDestroy(stream);
  result = hipStreamDestroy(stream);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipStreamGetFlags(hipStream_t hStream, unsigned int *flags);
  // HIP: hipError_t hipStreamGetFlags(hipStream_t stream, unsigned int* flags);
  // CHECK: result = hipStreamGetFlags(stream, &flags);
  result = hipStreamGetFlags(stream, &flags);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipStreamGetPriority(hipStream_t hStream, int *priority);
  // HIP: hipError_t hipStreamGetPriority(hipStream_t stream, int* priority);
  // CHECK: result = hipStreamGetPriority(stream, &intVal);
  result = hipStreamGetPriority(stream, &intVal);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipStreamQuery(hipStream_t stream);
  // HIP: hipError_t hipStreamQuery(hipStream_t stream);
  // CHECK: result = hipStreamQuery(stream);
  result = hipStreamQuery(stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipStreamSynchronize(hipStream_t stream);
  // HIP: hipError_t hipStreamSynchronize(hipStream_t stream);
  // CHECK: result = hipStreamSynchronize(stream);
  result = hipStreamSynchronize(stream);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipStreamWaitEvent(hipStream_t stream, hipEvent_t event, unsigned int flags __dv(0));
  // HIP: hipError_t hipStreamWaitEvent(hipStream_t stream, hipEvent_t event, unsigned int flags);
  // CHECK: result = hipStreamWaitEvent(stream, Event_t, flags);
  result = hipStreamWaitEvent(stream, Event_t, flags);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipEventCreate(hipEvent_t *event);
  // HIP: hipError_t hipEventCreate(hipEvent_t* event);
  // CHECK: result = hipEventCreate(&Event_t);
  result = hipEventCreate(&Event_t);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipEventCreateWithFlags(hipEvent_t *event, unsigned int flags);
  // HIP: hipError_t hipEventCreateWithFlags(hipEvent_t* event, unsigned flags);
  // CHECK: result = hipEventCreateWithFlags(&Event_t, flags);
  result = hipEventCreateWithFlags(&Event_t, flags);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipEventDestroy(hipEvent_t event);
  // HIP: hipError_t hipEventDestroy(hipEvent_t event);
  // CHECK: result = hipEventDestroy(Event_t);
  result = hipEventDestroy(Event_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipEventElapsedTime(float *ms, hipEvent_t start, hipEvent_t end);
  // HIP: hipError_t hipEventElapsedTime(float* ms, hipEvent_t start, hipEvent_t stop);
  // CHECK: result = hipEventElapsedTime(&ms, Event_t, Event_2);
  result = hipEventElapsedTime(&ms, Event_t, Event_2);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipEventQuery(hipEvent_t event);
  // HIP: hipError_t hipEventQuery(hipEvent_t event);
  // CHECK: result = hipEventQuery(Event_t);
  result = hipEventQuery(Event_t);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipEventRecord(hipEvent_t event, hipStream_t stream __dv(0));
  // HIP: hipError_t hipEventRecord(hipEvent_t event, hipStream_t stream);
  // CHECK: result = hipEventRecord(Event_t, stream);
  result = hipEventRecord(Event_t, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipEventSynchronize(hipEvent_t event);
  // HIP: hipError_t hipEventSynchronize(hipEvent_t event);
  // CHECK: result = hipEventSynchronize(Event_t);
  result = hipEventSynchronize(Event_t);

  // CHECK: hipFuncAttributes FuncAttributes;
  hipFuncAttributes FuncAttributes;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipFuncGetAttributes(struct hipFuncAttributes *attr, reinterpret_cast<const void*>(const void *func));
  // HIP: hipError_t hipFuncGetAttributes(struct hipFuncAttributes* attr, reinterpret_cast<const void*>(const void* func));
  // CHECK: result = hipFuncGetAttributes(&FuncAttributes, reinterpret_cast<const void*>(func));
  result = hipFuncGetAttributes(&FuncAttributes, reinterpret_cast<const void*>(func));

#if CUDA_VERSION >= 9000
  // CHECK: hipFuncAttribute FuncAttribute;
  hipFuncAttribute FuncAttribute;

  // CHECK: hipLaunchParams LaunchParams;
  hipLaunchParams LaunchParams;

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipFuncSetAttribute(reinterpret_cast<const void*>(const void *func), enum hipFuncAttribute attr, int value);
  // HIP: hipError_t hipFuncSetAttribute(reinterpret_cast<const void*>(const void* func), hipFuncAttribute attr, int value);
  // CHECK: result = hipFuncSetAttribute(reinterpret_cast<const void*>(func), FuncAttribute, intVal);
  result = hipFuncSetAttribute(reinterpret_cast<const void*>(func), FuncAttribute, intVal);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipLaunchCooperativeKernel(reinterpret_cast<const void*>(const void *func), dim3 gridDim, dim3 blockDim, void **args, size_t sharedMem, hipStream_t stream);
  // HIP: hipError_t hipLaunchCooperativeKernel(reinterpret_cast<const void*>(const void* f), dim3 gridDim, dim3 blockDimX, void** kernelParams, unsigned int sharedMemBytes, hipStream_t stream);
  // CHECK: result = hipLaunchCooperativeKernel(reinterpret_cast<const void*>(func), gridDim, blockDim, &image, flags, stream);
  result = hipLaunchCooperativeKernel(reinterpret_cast<const void*>(func), gridDim, blockDim, &image, flags, stream);

  // CUDA: extern __CUDA_DEPRECATED __host__ hipError_t CUDARTAPI hipLaunchCooperativeKernelMultiDevice(struct hipLaunchParams *launchParamsList, unsigned int numDevices, unsigned int flags  __dv(0));
  // HIP: hipError_t hipLaunchCooperativeKernelMultiDevice(hipLaunchParams* launchParamsList, int numDevices, unsigned int flags);
  // CHECK: result = hipLaunchCooperativeKernelMultiDevice(&LaunchParams, intVal, flags);
  result = hipLaunchCooperativeKernelMultiDevice(&LaunchParams, intVal, flags);
#endif

  // CUDA: extern __host__ hipError_t CUDARTAPI hipFuncSetCacheConfig(reinterpret_cast<const void*>(const void *func), enum hipFuncCache_t cacheConfig);
  // HIP: hipError_t hipFuncSetCacheConfig(reinterpret_cast<const void*>(const void* func), hipFuncCache_t config);
  // CHECK: result = hipFuncSetCacheConfig(reinterpret_cast<const void*>(func), FuncCache);
  result = hipFuncSetCacheConfig(reinterpret_cast<const void*>(func), FuncCache);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(const void *func), enum hipSharedMemConfig config);
  // HIP: hipError_t hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(const void* func), hipSharedMemConfig config);
  // CHECK: result = hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(func), SharedMemConfig);
  result = hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(func), SharedMemConfig);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipLaunchKernel(reinterpret_cast<const void*>(const void *func), dim3 gridDim, dim3 blockDim, void **args, size_t sharedMem, hipStream_t stream);
  // HIP: hipError_t hipLaunchKernel(reinterpret_cast<const void*>(const void* function_address), dim3 numBlocks, dim3 dimBlocks, void** args, size_t sharedMemBytes __dparm(0), hipStream_t stream __dparm(0));
  // CHECK: result = hipLaunchKernel(reinterpret_cast<const void*>(func), gridDim, blockDim, &image, bytes, stream);
  result = hipLaunchKernel(reinterpret_cast<const void*>(func), gridDim, blockDim, &image, bytes, stream);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipOccupancyMaxActiveBlocksPerMultiprocessor(int *numBlocks, const void *func, int blockSize, size_t dynamicSMemSize);
  // HIP: hipError_t hipOccupancyMaxActiveBlocksPerMultiprocessor(int* numBlocks, const void* f, int blockSize, size_t dynSharedMemPerBlk);
  // CHECK: result = hipOccupancyMaxActiveBlocksPerMultiprocessor(&intVal, func, device, bytes);
  result = hipOccupancyMaxActiveBlocksPerMultiprocessor(&intVal, func, device, bytes);

  // CUDA: extern __host__ __cudart_builtin__ hipError_t CUDARTAPI hipOccupancyMaxActiveBlocksPerMultiprocessorWithFlags(int *numBlocks, const void *func, int blockSize, size_t dynamicSMemSize, unsigned int flags);
  // HIP: hipError_t hipOccupancyMaxActiveBlocksPerMultiprocessorWithFlags(int* numBlocks, const void* f, int blockSize, size_t dynSharedMemPerBlk, unsigned int flags __dparm(hipOccupancyDefault));
  // CHECK: result = hipOccupancyMaxActiveBlocksPerMultiprocessorWithFlags(&intVal, func, intVal, bytes, flags);
  result = hipOccupancyMaxActiveBlocksPerMultiprocessorWithFlags(&intVal, func, intVal, bytes, flags);

  // CUDA: template<class T> static __inline__ __host__ CUDART_DEVICE hipError_t hipOccupancyMaxPotentialBlockSize(int* minGridSize, int* blockSize, T func, size_t dynamicSMemSize = 0, int blockSizeLimit = 0);
  // HIP: template <typename T> static hipError_t __host__ inline hipOccupancyMaxPotentialBlockSize(int* gridSize, int* blockSize, T f, size_t dynSharedMemPerBlk = 0, int blockSizeLimit = 0);
  // CHECK: result = hipOccupancyMaxPotentialBlockSize(&intVal, &device, func, bytes, deviceId);
  result = hipOccupancyMaxPotentialBlockSize(&intVal, &device, func, bytes, deviceId);

  // CUDA: template<class T> static __inline__ __host__ CUDART_DEVICE hipError_t hipOccupancyMaxPotentialBlockSizeWithFlags(int* minGridSize, int* blockSize, T func, size_t dynamicSMemSize = 0, int blockSizeLimit = 0, unsigned int flags = 0);
  // HIP: template <typename T> static hipError_t __host__ inline hipOccupancyMaxPotentialBlockSizeWithFlags(int* gridSize, int* blockSize, T f, size_t dynSharedMemPerBlk = 0, int blockSizeLimit = 0, unsigned int  flags = 0);
  // CHECK: result = hipOccupancyMaxPotentialBlockSizeWithFlags(&intVal, &device, func, bytes, deviceId, flags);
  result = hipOccupancyMaxPotentialBlockSizeWithFlags(&intVal, &device, func, bytes, deviceId, flags);

  return 0;
}
