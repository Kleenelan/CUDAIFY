// RUN: %run_test hipify "%s" "%t" %hipify_args 2 --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <stdio.h>
#if defined(_WIN32)
  #include "windows.h"
  #include <GL/glew.h>
#endif
#include "cudaGL.h"

int main() {
  printf("12. CUDA Runtime API Functions synthetic test\n");

  size_t bytes = 0;
  int device = 0;
  void* deviceptr = nullptr;
  void* image = nullptr;

#if defined(_WIN32)
  unsigned long long ull = 0;
#else
  unsigned long ull = 0;
#endif

  // CHECK: hipError_t result = hipSuccess;
  // CHECK-NEXT: hipStream_t stream;
  hipError_t result = hipSuccess;
  hipStream_t stream;

#if CUDA_VERSION >= 10000
  // CHECK: hipHostFn_t hostFn;
  hipHostFn_t hostFn;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipLaunchHostFunc(hipStream_t stream, hipHostFn_t fn, void *userData);
  // HIP: hipError_t hipLaunchHostFunc(hipStream_t stream, hipHostFn_t fn, void* userData);
  // CHECK: result = hipLaunchHostFunc(stream, hostFn, image);
  result = hipLaunchHostFunc(stream, hostFn, image);
#endif

#if CUDA_VERSION >= 10010
  // CHECK: hipStreamCaptureMode streamCaptureMode;
  hipStreamCaptureMode streamCaptureMode;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipThreadExchangeStreamCaptureMode(enum hipStreamCaptureMode *mode);
  // HIP: hipError_t hipThreadExchangeStreamCaptureMode(hipStreamCaptureMode* mode);
  // CHECK: result = hipThreadExchangeStreamCaptureMode(&streamCaptureMode);
  result = hipThreadExchangeStreamCaptureMode(&streamCaptureMode);
#endif

#if CUDA_VERSION >= 11020
  // CHECK: hipMemPoolAttr memPoolAttr;
  hipMemPoolAttr memPoolAttr;
  // CHECK: hipMemAccessDesc memAccessDesc;
  hipMemAccessDesc memAccessDesc;
  // CHECK: hipMemAccessFlags memAccessFlags;
  hipMemAccessFlags memAccessFlags;
  // CHECK: hipMemLocation memLocation;
  hipMemLocation memLocation;
  // CHECK: hipMemPoolProps memPoolProps;
  hipMemPoolProps memPoolProps;
  // CHECK: hipMemPool_t memPool_t;
  hipMemPool_t memPool_t;
  // CHECK: hipMemAllocationHandleType memAllocationHandleType;
  hipMemAllocationHandleType memAllocationHandleType;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceGetDefaultMemPool(hipMemPool_t *memPool, int device);
  // HIP: hipError_t hipDeviceGetDefaultMemPool(hipMemPool_t* mem_pool, int device);
  // CHECK: result = hipDeviceGetDefaultMemPool(&memPool_t, device);
  result = hipDeviceGetDefaultMemPool(&memPool_t, device);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceSetMemPool(int device, hipMemPool_t memPool);
  // HIP: hipError_t hipDeviceSetMemPool(int device, hipMemPool_t mem_pool);
  // CHECK: result = hipDeviceSetMemPool(device, memPool_t);
  result = hipDeviceSetMemPool(device, memPool_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipDeviceGetMemPool(hipMemPool_t *memPool, int device);
  // HIP: hipError_t hipDeviceGetMemPool(hipMemPool_t* mem_pool, int device);
  // CHECK: result = hipDeviceGetMemPool(&memPool_t, device);
  result = hipDeviceGetMemPool(&memPool_t, device);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMallocAsync(void **devPtr, size_t size, hipStream_t hStream);
  // HIP: hipError_t hipMallocAsync(void** dev_ptr, size_t size, hipStream_t stream);
  // CHECK: result = hipMallocAsync(&deviceptr, bytes, stream);
  result = hipMallocAsync(&deviceptr, bytes, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipFreeAsync(void *devPtr, hipStream_t hStream);
  // HIP: hipError_t hipFreeAsync(void* dev_ptr, hipStream_t stream);
  // CHECK: result = hipFreeAsync(deviceptr, stream);
  result = hipFreeAsync(deviceptr, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolTrimTo(hipMemPool_t memPool, size_t minBytesToKeep);
  // HIP: hipError_t hipMemPoolTrimTo(hipMemPool_t mem_pool, size_t min_bytes_to_hold);
  // CHECK: result = hipMemPoolTrimTo(memPool_t, bytes);
  result = hipMemPoolTrimTo(memPool_t, bytes);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolSetAttribute(hipMemPool_t memPool, enum hipMemPoolAttr attr, void *value );
  // HIP: hipError_t hipMemPoolSetAttribute(hipMemPool_t mem_pool, hipMemPoolAttr attr, void* value);
  // CHECK: result = hipMemPoolSetAttribute(memPool_t, memPoolAttr, image);
  result = hipMemPoolSetAttribute(memPool_t, memPoolAttr, image);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolGetAttribute(hipMemPool_t memPool, enum hipMemPoolAttr attr, void *value );
  // HIP: hipError_t hipMemPoolGetAttribute(hipMemPool_t mem_pool, hipMemPoolAttr attr, void* value);
  // CHECK: result = hipMemPoolGetAttribute(memPool_t, memPoolAttr, image);
  result = hipMemPoolGetAttribute(memPool_t, memPoolAttr, image);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolSetAccess(hipMemPool_t memPool, const struct hipMemAccessDesc *descList, size_t count);
  // HIP: hipError_t hipMemPoolSetAccess(hipMemPool_t mem_pool, const hipMemAccessDesc* desc_list, size_t count);
  // CHECK: result = hipMemPoolSetAccess(memPool_t, &memAccessDesc, bytes);
  result = hipMemPoolSetAccess(memPool_t, &memAccessDesc, bytes);

  // CUDA: hipError_t extern __host__ hipError_t CUDARTAPI hipMemPoolGetAccess(enum hipMemAccessFlags *flags, hipMemPool_t memPool, struct hipMemLocation *location);
  // HIP: hipError_t hipMemPoolGetAccess(hipMemAccessFlags* flags, hipMemPool_t mem_pool, hipMemLocation* location);
  // CHECK: result = hipMemPoolGetAccess(&memAccessFlags, memPool_t, &memLocation);
  result = hipMemPoolGetAccess(&memAccessFlags, memPool_t, &memLocation);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolCreate(hipMemPool_t *memPool, const struct hipMemPoolProps *poolProps);
  // HIP: hipError_t hipMemPoolCreate(hipMemPool_t* mem_pool, const hipMemPoolProps* pool_props);
  // CHECK: result = hipMemPoolCreate(&memPool_t, &memPoolProps);
  result = hipMemPoolCreate(&memPool_t, &memPoolProps);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolDestroy(hipMemPool_t memPool);
  // HIP: hipError_t hipMemPoolDestroy(hipMemPool_t mem_pool);
  // CHECK: result = hipMemPoolDestroy(memPool_t);
  result = hipMemPoolDestroy(memPool_t);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMallocFromPoolAsync(void **ptr, size_t size, hipMemPool_t memPool, hipStream_t stream);
  // HIP: hipError_t hipMallocFromPoolAsync(void** dev_ptr, size_t size, hipMemPool_t mem_pool, hipStream_t stream);
  // CHECK: result = hipMallocFromPoolAsync(&deviceptr, bytes, memPool_t, stream);
  result = hipMallocFromPoolAsync(&deviceptr, bytes, memPool_t, stream);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolExportToShareableHandle(void* shareableHandle, hipMemPool_t memPool, enum hipMemAllocationHandleType handleType, unsigned int flags);
  // HIP: hipError_t hipMemPoolExportToShareableHandle(void* shared_handle, hipMemPool_t mem_pool, hipMemAllocationHandleType handle_type, unsigned int flags);
  // CHECK: result = hipMemPoolExportToShareableHandle(image, memPool_t, memAllocationHandleType, ull);
  result = hipMemPoolExportToShareableHandle(image, memPool_t, memAllocationHandleType, ull);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolImportFromShareableHandle(hipMemPool_t* memPool, void* shareableHandle, enum hipMemAllocationHandleType handleType, unsigned int flags);
  // HIP: hipError_t hipMemPoolImportFromShareableHandle(hipMemPool_t* mem_pool, void* shared_handle, hipMemAllocationHandleType handle_type, unsigned int flags);
  // CHECK: result = hipMemPoolImportFromShareableHandle(&memPool_t, image, memAllocationHandleType, ull);
  result = hipMemPoolImportFromShareableHandle(&memPool_t, image, memAllocationHandleType, ull);

  // CHECK: hipMemPoolPtrExportData memPoolPtrExportData;
  hipMemPoolPtrExportData memPoolPtrExportData;

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolExportPointer(struct hipMemPoolPtrExportData *exportData, void *ptr);
  // HIP: hipError_t hipMemPoolExportPointer(hipMemPoolPtrExportData* export_data, void* dev_ptr);
  // CHECK: result = hipMemPoolExportPointer(&memPoolPtrExportData, deviceptr);
  result = hipMemPoolExportPointer(&memPoolPtrExportData, deviceptr);

  // CUDA: extern __host__ hipError_t CUDARTAPI hipMemPoolImportPointer(void **ptr, hipMemPool_t memPool, struct hipMemPoolPtrExportData *exportData);
  // HIP: hipError_t hipMemPoolImportPointer(void** dev_ptr, hipMemPool_t mem_pool, hipMemPoolPtrExportData* export_data);
  // CHECK: result = hipMemPoolImportPointer(&deviceptr, memPool_t, &memPoolPtrExportData);
  result = hipMemPoolImportPointer(&deviceptr, memPool_t, &memPoolPtrExportData);
#endif

  return 0;
}
