// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --skip-excluded-preprocessor-conditional-blocks --experimental --roc %clang_args -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "rocsparse.h"
#include "hipsparse.h"
// CHECK-NOT: #include "rocsparse.h"

int main() {
  printf("18. cuSPARSE API to rocSPARSE API synthetic test\n");

  // CHECK: _rocsparse_handle *handle = nullptr;
  // CHECK-NEXT: rocsparse_handle handle_t;
  cusparseContext *handle = nullptr;
  hipsparseHandle_t handle_t;

  // CHECK: _rocsparse_mat_descr *matDescr = nullptr;
  // CHECK-NEXT: rocsparse_mat_descr matDescr_t;
  cusparseMatDescr *matDescr = nullptr;
  hipsparseMatDescr_t matDescr_t;

  // CHECK: _rocsparse_color_info *colorInfo = nullptr;
  // CHECK-NEXT: rocsparse_color_info colorInfo_t;
  cusparseColorInfo *colorInfo = nullptr;
  hipsparseColorInfo_t colorInfo_t;

  // CHECK: rocsparse_operation sparseOperation_t;
  // CHECK-NEXT: rocsparse_operation OPERATION_NON_TRANSPOSE = rocsparse_operation_none;
  // CHECK-NEXT: rocsparse_operation OPERATION_TRANSPOSE = rocsparse_operation_transpose;
  // CHECK-NEXT: rocsparse_operation OPERATION_CONJUGATE_TRANSPOSE = rocsparse_operation_conjugate_transpose;
  hipsparseOperation_t sparseOperation_t;
  hipsparseOperation_t OPERATION_NON_TRANSPOSE = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  hipsparseOperation_t OPERATION_TRANSPOSE = HIPSPARSE_OPERATION_TRANSPOSE;
  hipsparseOperation_t OPERATION_CONJUGATE_TRANSPOSE = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;

  // CHECK: rocsparse_index_base indexBase_t;
  // CHECK-NEXT: rocsparse_index_base INDEX_BASE_ZERO = rocsparse_index_base_zero;
  // CHECK-NEXT: rocsparse_index_base INDEX_BASE_ONE = rocsparse_index_base_one;
  hipsparseIndexBase_t indexBase_t;
  hipsparseIndexBase_t INDEX_BASE_ZERO = HIPSPARSE_INDEX_BASE_ZERO;
  hipsparseIndexBase_t INDEX_BASE_ONE = HIPSPARSE_INDEX_BASE_ONE;

  // CHECK: rocsparse_matrix_type matrixType_t;
  // CHECK-NEXT: rocsparse_matrix_type MATRIX_TYPE_GENERAL = rocsparse_matrix_type_general;
  // CHECK-NEXT: rocsparse_matrix_type MATRIX_TYPE_SYMMETRIC = rocsparse_matrix_type_symmetric;
  // CHECK-NEXT: rocsparse_matrix_type MATRIX_TYPE_HERMITIAN = rocsparse_matrix_type_hermitian;
  // CHECK-NEXT: rocsparse_matrix_type MATRIX_TYPE_TRIANGULAR = rocsparse_matrix_type_triangular;
  hipsparseMatrixType_t matrixType_t;
  hipsparseMatrixType_t MATRIX_TYPE_GENERAL = HIPSPARSE_MATRIX_TYPE_GENERAL;
  hipsparseMatrixType_t MATRIX_TYPE_SYMMETRIC = HIPSPARSE_MATRIX_TYPE_SYMMETRIC;
  hipsparseMatrixType_t MATRIX_TYPE_HERMITIAN = HIPSPARSE_MATRIX_TYPE_HERMITIAN;
  hipsparseMatrixType_t MATRIX_TYPE_TRIANGULAR = HIPSPARSE_MATRIX_TYPE_TRIANGULAR;

  // CHECK: rocsparse_diag_type diagType_t;
  // CHECK-NEXT: rocsparse_diag_type DIAG_TYPE_NON_UNIT = rocsparse_diag_type_non_unit;
  // CHECK-NEXT: rocsparse_diag_type DIAG_TYPE_UNIT = rocsparse_diag_type_unit;
  hipsparseDiagType_t diagType_t;
  hipsparseDiagType_t DIAG_TYPE_NON_UNIT = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  hipsparseDiagType_t DIAG_TYPE_UNIT = HIPSPARSE_DIAG_TYPE_UNIT;

  // CHECK: rocsparse_fill_mode fillMode_t;
  // CHECK-NEXT: rocsparse_fill_mode FILL_MODE_LOWER = rocsparse_fill_mode_lower;
  // CHECK-NEXT: rocsparse_fill_mode FILL_MODE_UPPER = rocsparse_fill_mode_upper;
  hipsparseFillMode_t fillMode_t;
  hipsparseFillMode_t FILL_MODE_LOWER = HIPSPARSE_FILL_MODE_LOWER;
  hipsparseFillMode_t FILL_MODE_UPPER = HIPSPARSE_FILL_MODE_UPPER;

  // CHECK: rocsparse_action action_t;
  // CHECK-NEXT: rocsparse_action ACTION_SYMBOLIC = rocsparse_action_symbolic;
  // CHECK-NEXT: rocsparse_action ACTION_NUMERIC = rocsparse_action_numeric;
  hipsparseAction_t action_t;
  hipsparseAction_t ACTION_SYMBOLIC = HIPSPARSE_ACTION_SYMBOLIC;
  hipsparseAction_t ACTION_NUMERIC = HIPSPARSE_ACTION_NUMERIC;

  // CHECK: rocsparse_direction direction_t;
  // CHECK-NEXT: rocsparse_direction DIRECTION_ROW = rocsparse_direction_row;
  // CHECK-NEXT: rocsparse_direction DIRECTION_COLUMN = rocsparse_direction_column;
  hipsparseDirection_t direction_t;
  hipsparseDirection_t DIRECTION_ROW = HIPSPARSE_DIRECTION_ROW;
  hipsparseDirection_t DIRECTION_COLUMN = HIPSPARSE_DIRECTION_COLUMN;

  // CHECK: rocsparse_solve_policy solvePolicy_t;
  // CHECK-NEXT: rocsparse_solve_policy SOLVE_POLICY_NO_LEVEL = rocsparse_solve_policy_auto;
  // CHECK-NEXT: rocsparse_solve_policy SOLVE_POLICY_USE_LEVEL = rocsparse_solve_policy_auto;
  hipsparseSolvePolicy_t solvePolicy_t;
  hipsparseSolvePolicy_t SOLVE_POLICY_NO_LEVEL = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
  hipsparseSolvePolicy_t SOLVE_POLICY_USE_LEVEL = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;

#if CUDA_VERSION >= 10010
  // CHECK: _rocsparse_spmat_descr *spMatDescr = nullptr;
  // CHECK-NEXT: rocsparse_spmat_descr spMatDescr_t;
  cusparseSpMatDescr *spMatDescr = nullptr;
  hipsparseSpMatDescr_t spMatDescr_t;

  // CHECK: _rocsparse_dnmat_descr *dnMatDescr = nullptr;
  // CHECK-NEXT: rocsparse_dnmat_descr dnMatDescr_t;
  cusparseDnMatDescr *dnMatDescr = nullptr;
  hipsparseDnMatDescr_t dnMatDescr_t;
#endif

#if CUDA_VERSION >= 10020
  // CHECK: _rocsparse_spvec_descr *spVecDescr = nullptr;
  // CHECK-NEXT: rocsparse_spvec_descr spVecDescr_t;
  cusparseSpVecDescr *spVecDescr = nullptr;
  hipsparseSpVecDescr_t spVecDescr_t;

  // CHECK: _rocsparse_dnvec_descr *dnVecDescr = nullptr;
  // CHECK-NEXT: rocsparse_dnvec_descr dnVecDescr_t;
  cusparseDnVecDescr *dnVecDescr = nullptr;
  hipsparseDnVecDescr_t dnVecDescr_t;
#endif

#if CUDA_VERSION < 11000
  // CHECK: _rocsparse_hyb_mat *hybMat = nullptr;
  // CHECK-NEXT: rocsparse_hyb_mat hybMat_t;
  cusparseHybMat *hybMat = nullptr;
  cusparseHybMat_t hybMat_t;

  // CHECK: rocsparse_hyb_partition hybPartition_t;
  // CHECK-NEXT: rocsparse_hyb_partition HYB_PARTITION_AUTO = rocsparse_hyb_partition_auto;
  // CHECK-NEXT: rocsparse_hyb_partition HYB_PARTITION_USER = rocsparse_hyb_partition_user;
  // CHECK-NEXT: rocsparse_hyb_partition HYB_PARTITION_MAX = rocsparse_hyb_partition_max;
  cusparseHybPartition_t hybPartition_t;
  cusparseHybPartition_t HYB_PARTITION_AUTO = CUSPARSE_HYB_PARTITION_AUTO;
  cusparseHybPartition_t HYB_PARTITION_USER = CUSPARSE_HYB_PARTITION_USER;
  cusparseHybPartition_t HYB_PARTITION_MAX = CUSPARSE_HYB_PARTITION_MAX;
#endif

  return 0;
}
