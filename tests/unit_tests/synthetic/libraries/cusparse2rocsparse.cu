// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --skip-excluded-preprocessor-conditional-blocks --experimental --roc %clang_args -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "rocsparse.h"
#include "hipsparse.h"
// CHECK-NOT: #include "rocsparse.h"

int main() {
  printf("18. cuSPARSE API to rocSPARSE API synthetic test\n");

  // CHECK: _rocsparse_handle *handle = nullptr;
  // CHECK-NEXT: rocsparse_handle handle_t;
  cusparseContext *handle = nullptr;
  hipsparseHandle_t handle_t;

  // CHECK: _rocsparse_mat_descr *matDescr = nullptr;
  // CHECK-NEXT: rocsparse_mat_descr matDescr_t;
  cusparseMatDescr *matDescr = nullptr;
  hipsparseMatDescr_t matDescr_t;

#if CUDA_VERSION >= 10010
  // CHECK: _rocsparse_spmat_descr *spMatDescr = nullptr;
  // CHECK-NEXT: rocsparse_spmat_descr spMatDescr_t;
  cusparseSpMatDescr *spMatDescr = nullptr;
  hipsparseSpMatDescr_t spMatDescr_t;

  // CHECK: _rocsparse_dnmat_descr *dnMatDescr = nullptr;
  // CHECK-NEXT: rocsparse_dnmat_descr dnMatDescr_t;
  cusparseDnMatDescr *dnMatDescr = nullptr;
  hipsparseDnMatDescr_t dnMatDescr_t;
#endif

#if CUDA_VERSION >= 10020
  // CHECK: _rocsparse_spvec_descr *spVecDescr = nullptr;
  // CHECK-NEXT: rocsparse_spvec_descr spVecDescr_t;
  cusparseSpVecDescr *spVecDescr = nullptr;
  hipsparseSpVecDescr_t spVecDescr_t;

  // CHECK: _rocsparse_dnvec_descr *dnVecDescr = nullptr;
  // CHECK-NEXT: rocsparse_dnvec_descr dnVecDescr_t;
  cusparseDnVecDescr *dnVecDescr = nullptr;
  hipsparseDnVecDescr_t dnVecDescr_t;
#endif

#if CUDA_VERSION < 11000
  // CHECK: _rocsparse_hyb_mat *hybMat = nullptr;
  // CHECK-NEXT: rocsparse_hyb_mat hybMat_t;
  cusparseHybMat *hybMat = nullptr;
  cusparseHybMat_t hybMat_t;
#endif

  return 0;
}
