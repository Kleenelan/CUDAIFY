// RUN: %run_test hipify "%s" "%t" %hipify_args 2 --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "hipsparse.h"
#include "hipsparse.h"
// CHECK-NOT: #include "hipsparse.h"

int main() {
  printf("17. cuSPARSE API to hipSPARSE API synthetic test\n");

  // CHECK: hipsparseHandle_t handle_t;
  hipsparseHandle_t handle_t;

  // CHECK: hipsparseMatDescr_t matDescr_t;
  hipsparseMatDescr_t matDescr_t;

  // CHECK: hipsparseColorInfo_t colorInfo_t;
  hipsparseColorInfo_t colorInfo_t;

  // CHECK: hipsparseOperation_t sparseOperation_t;
  // CHECK-NEXT: hipsparseOperation_t OPERATION_NON_TRANSPOSE = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  // CHECK-NEXT: hipsparseOperation_t OPERATION_TRANSPOSE = HIPSPARSE_OPERATION_TRANSPOSE;
  // CHECK-NEXT: hipsparseOperation_t OPERATION_CONJUGATE_TRANSPOSE = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;
  hipsparseOperation_t sparseOperation_t;
  hipsparseOperation_t OPERATION_NON_TRANSPOSE = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  hipsparseOperation_t OPERATION_TRANSPOSE = HIPSPARSE_OPERATION_TRANSPOSE;
  hipsparseOperation_t OPERATION_CONJUGATE_TRANSPOSE = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;

  // CHECK: hipsparseIndexBase_t indexBase_t;
  // CHECK-NEXT: hipsparseIndexBase_t INDEX_BASE_ZERO = HIPSPARSE_INDEX_BASE_ZERO;
  // CHECK-NEXT: hipsparseIndexBase_t INDEX_BASE_ONE = HIPSPARSE_INDEX_BASE_ONE;
  hipsparseIndexBase_t indexBase_t;
  hipsparseIndexBase_t INDEX_BASE_ZERO = HIPSPARSE_INDEX_BASE_ZERO;
  hipsparseIndexBase_t INDEX_BASE_ONE = HIPSPARSE_INDEX_BASE_ONE;

  // CHECK: hipsparseMatrixType_t matrixType_t;
  // CHECK-NEXT: hipsparseMatrixType_t MATRIX_TYPE_GENERAL = HIPSPARSE_MATRIX_TYPE_GENERAL;
  // CHECK-NEXT: hipsparseMatrixType_t MATRIX_TYPE_SYMMETRIC = HIPSPARSE_MATRIX_TYPE_SYMMETRIC;
  // CHECK-NEXT: hipsparseMatrixType_t MATRIX_TYPE_HERMITIAN = HIPSPARSE_MATRIX_TYPE_HERMITIAN;
  // CHECK-NEXT: hipsparseMatrixType_t MATRIX_TYPE_TRIANGULAR = HIPSPARSE_MATRIX_TYPE_TRIANGULAR;
  hipsparseMatrixType_t matrixType_t;
  hipsparseMatrixType_t MATRIX_TYPE_GENERAL = HIPSPARSE_MATRIX_TYPE_GENERAL;
  hipsparseMatrixType_t MATRIX_TYPE_SYMMETRIC = HIPSPARSE_MATRIX_TYPE_SYMMETRIC;
  hipsparseMatrixType_t MATRIX_TYPE_HERMITIAN = HIPSPARSE_MATRIX_TYPE_HERMITIAN;
  hipsparseMatrixType_t MATRIX_TYPE_TRIANGULAR = HIPSPARSE_MATRIX_TYPE_TRIANGULAR;

#if CUDA_VERSION >= 10010
  // CHECK: hipsparseSpMatDescr_t spMatDescr_t;
  hipsparseSpMatDescr_t spMatDescr_t;

  // CHECK: hipsparseDnMatDescr_t dnMatDescr_t;
  hipsparseDnMatDescr_t dnMatDescr_t;
#endif

#if CUDA_VERSION >= 10020
  // CHECK: hipsparseSpVecDescr_t spVecDescr_t;
  hipsparseSpVecDescr_t spVecDescr_t;

  // CHECK: hipsparseDnVecDescr_t dnVecDescr_t;
  hipsparseDnVecDescr_t dnVecDescr_t;
#endif

#if CUDA_VERSION < 11000
  // CHECK: hipsparseHybMat_t hybMat_t;
  cusparseHybMat_t hybMat_t;
#endif

  return 0;
}
