// RUN: %run_test hipify "%s" "%t" %hipify_args 2 --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "hipsolver.h"
#include "hipsolver.h"

int main() {
  printf("19. cuSOLVER API to hipSOLVER API synthetic test\n");

  int m = 0;
  int n = 0;
  int nrhs = 0;
  int lda = 0;
  int ldb = 0;
  int Lwork = 0;
  int devIpiv = 0;
  int devInfo = 0;
  float fA = 0.f;
  double dA = 0.f;
  float fB = 0.f;
  double dB = 0.f;
  float fWorkspace = 0.f;
  double dWorkspace = 0.f;

  // CHECK: hipsolverHandle_t handle;
  hipsolverHandle_t handle;

  // CHECK: hipsolverStatus_t status;
  hipsolverStatus_t status;

  // CHECK: hipblasOperation_t blasOperation;
  hipblasOperation_t blasOperation;

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCreate(cusolverDnHandle_t *handle);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCreate(hipsolverHandle_t* handle);
  // CHECK: status = hipsolverDnCreate(&handle);
  status = hipsolverDnCreate(&handle);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDestroy(cusolverDnHandle_t handle);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDestroy(hipsolverHandle_t handle);
  // CHECK: status = hipsolverDnDestroy(handle);
  status = hipsolverDnDestroy(handle);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDgetrf(cusolverDnHandle_t handle, int m, int n, double* A, int lda, double* Workspace, int* devIpiv, int* devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDgetrf(hipsolverHandle_t handle, int m, int n,double* A, int lda, double* work, int* devIpiv, int* devInfo);
  // CHECK: status = hipsolverDnDgetrf(handle, m, n, &dA, lda, &dWorkspace, &devIpiv, &devInfo);
  status = hipsolverDnDgetrf(handle, m, n, &dA, lda, &dWorkspace, &devIpiv, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDgetrf_bufferSize(cusolverDnHandle_t handle, int m, int n, double* A, int lda, int* Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDgetrf_bufferSize(hipsolverHandle_t handle, int m, int n, double* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnDgetrf_bufferSize(handle, m, n, &dA, lda, &Lwork);
  status = hipsolverDnDgetrf_bufferSize(handle, m, n, &dA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSgetrf(cusolverDnHandle_t handle, int m, int n, float* A, int lda, float* Workspace, int* devIpiv, int* devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSgetrf(hipsolverHandle_t handle, int m, int n, float* A, int lda, float* work, int* devIpiv, int* devInfo);
  // CHECK: status = hipsolverDnSgetrf(handle, m, n, &fA, lda, &fWorkspace, &devIpiv, &devInfo);
  status = hipsolverDnSgetrf(handle, m, n, &fA, lda, &fWorkspace, &devIpiv, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSgetrf_bufferSize(cusolverDnHandle_t handle, int m, int n, float* A, int lda, int* Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSgetrf_bufferSize(hipsolverHandle_t handle, int m, int n, float* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnSgetrf_bufferSize(handle, m, n, &fA, lda, &Lwork);
  status = hipsolverDnSgetrf_bufferSize(handle, m, n, &fA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDgetrs(cusolverDnHandle_t handle, cublasOperation_t trans, int n, int nrhs,const double* A, int lda, const int* devIpiv, double* B, int ldb, int* devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDgetrs(hipsolverHandle_t handle, hipsolverOperation_t trans, int n, int nrhs, const double* A, int lda, const int* devIpiv, double* B, int ldb, int* devInfo);
  // CHECK: status = hipsolverDnDgetrs(handle, blasOperation, n, nrhs , &dA, lda, &devIpiv, &dB, ldb, &devInfo);
  status = hipsolverDnDgetrs(handle, blasOperation, n, nrhs , &dA, lda, &devIpiv, &dB, ldb, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSgetrs(cusolverDnHandle_t handle, cublasOperation_t  trans, int n, int nrhs, const float* A, int lda, const int* devIpiv, float* B, int ldb, int* devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSgetrs(hipsolverHandle_t handle, hipsolverOperation_t trans, int n, int nrhs, const float* A, int lda, const int* devIpiv, float* B, int ldb, int* devInfo);
  // CHECK: status = hipsolverDnSgetrs(handle, blasOperation, n, nrhs , &fA, lda, &devIpiv, &fB, ldb, &devInfo);
  status = hipsolverDnSgetrs(handle, blasOperation, n, nrhs , &fA, lda, &devIpiv, &fB, ldb, &devInfo);

  return 0;
}
