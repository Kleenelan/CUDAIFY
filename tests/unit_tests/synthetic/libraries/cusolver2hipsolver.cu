// RUN: %run_test hipify "%s" "%t" %hipify_args 2 --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "hipsolver.h"
#include "hipsolver.h"

int main() {
  printf("19. cuSOLVER API to hipSOLVER API synthetic test\n");

  int m = 0;
  int n = 0;
  int nrhs = 0;
  int lda = 0;
  int ldb = 0;
  int Lwork = 0;
  int devIpiv = 0;
  int devInfo = 0;
  float fA = 0.f;
  float fB = 0.f;
  float fX = 0.f;
  double dA = 0.f;
  double dB = 0.f;
  double dX = 0.f;
  float fWorkspace = 0.f;
  double dWorkspace = 0.f;
  void *Workspace = nullptr;
  size_t lwork_bytes = 0;

  // CHECK: hipDoubleComplex dComplexA, dComplexB, dComplexX;
  hipDoubleComplex dComplexA, dComplexB, dComplexX;

  // CHECK: hipComplex complexA, complexB, complexX;
  hipComplex complexA, complexB, complexX;

  // CHECK: hipsolverHandle_t handle;
  hipsolverHandle_t handle;

  // CHECK: hipsolverStatus_t status;
  // CHECK-NEXT: hipsolverStatus_t STATUS_SUCCESS = HIPSOLVER_STATUS_SUCCESS;
  // CHECK-NEXT: hipsolverStatus_t STATUS_NOT_INITIALIZED = HIPSOLVER_STATUS_NOT_INITIALIZED;
  // CHECK-NEXT: hipsolverStatus_t STATUS_ALLOC_FAILED = HIPSOLVER_STATUS_ALLOC_FAILED;
  // CHECK-NEXT: hipsolverStatus_t STATUS_INVALID_VALUE = HIPSOLVER_STATUS_INVALID_VALUE;
  // CHECK-NEXT: hipsolverStatus_t STATUS_ARCH_MISMATCH = HIPSOLVER_STATUS_ARCH_MISMATCH;
  // CHECK-NEXT: hipsolverStatus_t STATUS_MAPPING_ERROR = HIPSOLVER_STATUS_MAPPING_ERROR;
  // CHECK-NEXT: hipsolverStatus_t STATUS_EXECUTION_FAILED = HIPSOLVER_STATUS_EXECUTION_FAILED;
  // CHECK-NEXT: hipsolverStatus_t STATUS_INTERNAL_ERROR = HIPSOLVER_STATUS_INTERNAL_ERROR;
  // CHECK-NEXT: hipsolverStatus_t STATUS_NOT_SUPPORTED = HIPSOLVER_STATUS_NOT_SUPPORTED;
  // CHECK-NEXT: hipsolverStatus_t STATUS_ZERO_PIVOT = HIPSOLVER_STATUS_ZERO_PIVOT;
  hipsolverStatus_t status;
  hipsolverStatus_t STATUS_SUCCESS = HIPSOLVER_STATUS_SUCCESS;
  hipsolverStatus_t STATUS_NOT_INITIALIZED = HIPSOLVER_STATUS_NOT_INITIALIZED;
  hipsolverStatus_t STATUS_ALLOC_FAILED = HIPSOLVER_STATUS_ALLOC_FAILED;
  hipsolverStatus_t STATUS_INVALID_VALUE = HIPSOLVER_STATUS_INVALID_VALUE;
  hipsolverStatus_t STATUS_ARCH_MISMATCH = HIPSOLVER_STATUS_ARCH_MISMATCH;
  hipsolverStatus_t STATUS_MAPPING_ERROR = HIPSOLVER_STATUS_MAPPING_ERROR;
  hipsolverStatus_t STATUS_EXECUTION_FAILED = HIPSOLVER_STATUS_EXECUTION_FAILED;
  hipsolverStatus_t STATUS_INTERNAL_ERROR = HIPSOLVER_STATUS_INTERNAL_ERROR;
  hipsolverStatus_t STATUS_NOT_SUPPORTED = HIPSOLVER_STATUS_NOT_SUPPORTED;
  hipsolverStatus_t STATUS_ZERO_PIVOT = HIPSOLVER_STATUS_ZERO_PIVOT;

  // CHECK: hipblasOperation_t blasOperation;
  hipblasOperation_t blasOperation;

  // CHECK: hipStream_t stream_t;
  hipStream_t stream_t;

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCreate(cusolverDnHandle_t *handle);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCreate(hipsolverHandle_t* handle);
  // CHECK: status = hipsolverDnCreate(&handle);
  status = hipsolverDnCreate(&handle);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDestroy(cusolverDnHandle_t handle);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDestroy(hipsolverHandle_t handle);
  // CHECK: status = hipsolverDnDestroy(handle);
  status = hipsolverDnDestroy(handle);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDgetrf(cusolverDnHandle_t handle, int m, int n, double* A, int lda, double* Workspace, int* devIpiv, int* devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDgetrf(hipsolverHandle_t handle, int m, int n,double* A, int lda, double* work, int* devIpiv, int* devInfo);
  // CHECK: status = hipsolverDnDgetrf(handle, m, n, &dA, lda, &dWorkspace, &devIpiv, &devInfo);
  status = hipsolverDnDgetrf(handle, m, n, &dA, lda, &dWorkspace, &devIpiv, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDgetrf_bufferSize(cusolverDnHandle_t handle, int m, int n, double* A, int lda, int* Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDgetrf_bufferSize(hipsolverHandle_t handle, int m, int n, double* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnDgetrf_bufferSize(handle, m, n, &dA, lda, &Lwork);
  status = hipsolverDnDgetrf_bufferSize(handle, m, n, &dA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSgetrf(cusolverDnHandle_t handle, int m, int n, float* A, int lda, float* Workspace, int* devIpiv, int* devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSgetrf(hipsolverHandle_t handle, int m, int n, float* A, int lda, float* work, int* devIpiv, int* devInfo);
  // CHECK: status = hipsolverDnSgetrf(handle, m, n, &fA, lda, &fWorkspace, &devIpiv, &devInfo);
  status = hipsolverDnSgetrf(handle, m, n, &fA, lda, &fWorkspace, &devIpiv, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSgetrf_bufferSize(cusolverDnHandle_t handle, int m, int n, float* A, int lda, int* Lwork);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSgetrf_bufferSize(hipsolverHandle_t handle, int m, int n, float* A, int lda, int* lwork);
  // CHECK: status = hipsolverDnSgetrf_bufferSize(handle, m, n, &fA, lda, &Lwork);
  status = hipsolverDnSgetrf_bufferSize(handle, m, n, &fA, lda, &Lwork);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDgetrs(cusolverDnHandle_t handle, cublasOperation_t trans, int n, int nrhs,const double* A, int lda, const int* devIpiv, double* B, int ldb, int* devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDgetrs(hipsolverHandle_t handle, hipsolverOperation_t trans, int n, int nrhs, const double* A, int lda, const int* devIpiv, double* B, int ldb, int* devInfo);
  // CHECK: status = hipsolverDnDgetrs(handle, blasOperation, n, nrhs , &dA, lda, &devIpiv, &dB, ldb, &devInfo);
  status = hipsolverDnDgetrs(handle, blasOperation, n, nrhs , &dA, lda, &devIpiv, &dB, ldb, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSgetrs(cusolverDnHandle_t handle, cublasOperation_t  trans, int n, int nrhs, const float* A, int lda, const int* devIpiv, float* B, int ldb, int* devInfo);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSgetrs(hipsolverHandle_t handle, hipsolverOperation_t trans, int n, int nrhs, const float* A, int lda, const int* devIpiv, float* B, int ldb, int* devInfo);
  // CHECK: status = hipsolverDnSgetrs(handle, blasOperation, n, nrhs , &fA, lda, &devIpiv, &fB, ldb, &devInfo);
  status = hipsolverDnSgetrs(handle, blasOperation, n, nrhs , &fA, lda, &devIpiv, &fB, ldb, &devInfo);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSetStream(cusolverDnHandle_t handle, cudaStream_t streamId);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverSetStream(hipsolverHandle_t handle, hipStream_t streamId);
  // CHECK: status = hipsolverSetStream(handle, stream_t);
  status = hipsolverSetStream(handle, stream_t);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnGetStream(cusolverDnHandle_t handle, cudaStream_t *streamId);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverGetStream(hipsolverHandle_t handle, hipStream_t* streamId);
  // CHECK: status = hipsolverGetStream(handle, &stream_t);
  status = hipsolverGetStream(handle, &stream_t);

#if CUDA_VERSION >= 8000
  // CHECK: hipsolverEigType_t eigType;
  // CHECK-NEXT: hipsolverEigType_t EIG_TYPE_1 = HIPSOLVER_EIG_TYPE_1;
  // CHECK-NEXT: hipsolverEigType_t EIG_TYPE_2 = HIPSOLVER_EIG_TYPE_2;
  // CHECK-NEXT: hipsolverEigType_t EIG_TYPE_3 = HIPSOLVER_EIG_TYPE_3;
  hipsolverEigType_t eigType;
  hipsolverEigType_t EIG_TYPE_1 = HIPSOLVER_EIG_TYPE_1;
  hipsolverEigType_t EIG_TYPE_2 = HIPSOLVER_EIG_TYPE_2;
  hipsolverEigType_t EIG_TYPE_3 = HIPSOLVER_EIG_TYPE_3;

  // CHECK: hipsolverEigMode_t eigMode;
  // CHECK-NEXT: hipsolverEigMode_t SOLVER_EIG_MODE_NOVECTOR = HIPSOLVER_EIG_MODE_NOVECTOR;
  // CHECK-NEXT: hipsolverEigMode_t SOLVER_EIG_MODE_VECTOR = HIPSOLVER_EIG_MODE_VECTOR;
  hipsolverEigMode_t eigMode;
  hipsolverEigMode_t SOLVER_EIG_MODE_NOVECTOR = HIPSOLVER_EIG_MODE_NOVECTOR;
  hipsolverEigMode_t SOLVER_EIG_MODE_VECTOR = HIPSOLVER_EIG_MODE_VECTOR;
#endif

#if CUDA_VERSION >= 9000
  // CHECK: hipsolverSyevjInfo_t syevj_info;
  hipsolverSyevjInfo_t syevj_info;

  // CHECK: hipsolverGesvdjInfo_t gesvdj_info;
  hipsolverGesvdjInfo_t gesvdj_info;
#endif

#if CUDA_VERSION >= 10010
  // CHECK: int solver_int = 0;
  // CHECK: int ln = 0;
  // CHECK: int lnrhs = 0;
  // CHECK: int ldda = 0;
  // CHECK: int lddb = 0;
  // CHECK: int lddx = 0;
  // CHECK: int dipiv = 0;
  // CHECK: int iter = 0;
  // CHECK: int d_info = 0;
  int solver_int = 0;
  int ln = 0;
  int lnrhs = 0;
  int ldda = 0;
  int lddb = 0;
  int lddx = 0;
  int dipiv = 0;
  int iter = 0;
  int d_info = 0;

  // CHECK: hipsolverEigRange_t eigRange;
  // CHECK-NEXT: hipsolverEigRange_t EIG_RANGE_ALL = HIPSOLVER_EIG_RANGE_ALL;
  // CHECK-NEXT: hipsolverEigRange_t EIG_RANGE_I = HIPSOLVER_EIG_RANGE_I;
  // CHECK-NEXT: hipsolverEigRange_t EIG_RANGE_V = HIPSOLVER_EIG_RANGE_V;
  hipsolverEigRange_t eigRange;
  hipsolverEigRange_t EIG_RANGE_ALL = HIPSOLVER_EIG_RANGE_ALL;
  hipsolverEigRange_t EIG_RANGE_I = HIPSOLVER_EIG_RANGE_I;
  hipsolverEigRange_t EIG_RANGE_V = HIPSOLVER_EIG_RANGE_V;
#endif

#if CUDA_VERSION >= 10020
  // CHECK: hipsolverStatus_t STATUS_IRS_PARAMS_INVALID = HIPSOLVER_STATUS_INVALID_VALUE;
  // CHECK-NEXT: hipsolverStatus_t STATUS_IRS_INTERNAL_ERROR = HIPSOLVER_STATUS_INTERNAL_ERROR;
  // CHECK-NEXT: hipsolverStatus_t STATUS_IRS_NOT_SUPPORTED = HIPSOLVER_STATUS_NOT_SUPPORTED;
  hipsolverStatus_t STATUS_IRS_PARAMS_INVALID = HIPSOLVER_STATUS_INVALID_VALUE;
  hipsolverStatus_t STATUS_IRS_INTERNAL_ERROR = HIPSOLVER_STATUS_INTERNAL_ERROR;
  hipsolverStatus_t STATUS_IRS_NOT_SUPPORTED = HIPSOLVER_STATUS_NOT_SUPPORTED;

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZZgesv(cusolverDnHandle_t handle, cusolver_int_t n, cusolver_int_t nrhs, cuDoubleComplex * dA, cusolver_int_t ldda, cusolver_int_t * dipiv, cuDoubleComplex * dB, cusolver_int_t lddb, cuDoubleComplex * dX, cusolver_int_t lddx, void * dWorkspace, size_t lwork_bytes, cusolver_int_t * iter, cusolver_int_t * d_info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZZgesv(hipsolverHandle_t handle, int n, int nrhs, hipDoubleComplex* A, int lda, int* devIpiv, hipDoubleComplex* B, int ldb, hipDoubleComplex* X, int ldx, void* work, size_t lwork, int* niters, int* devInfo);
  // CHECK: status = hipsolverDnZZgesv(handle, ln, lnrhs, &dComplexA, ldda, &dipiv, &dComplexB, lddb, &dComplexX, lddx, &Workspace, lwork_bytes, &iter, &d_info);
  status = hipsolverDnZZgesv(handle, ln, lnrhs, &dComplexA, ldda, &dipiv, &dComplexB, lddb, &dComplexX, lddx, &Workspace, lwork_bytes, &iter, &d_info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCCgesv(cusolverDnHandle_t handle, cusolver_int_t n, cusolver_int_t nrhs, cuComplex * dA, cusolver_int_t ldda, cusolver_int_t * dipiv, cuComplex * dB, cusolver_int_t lddb, cuComplex * dX, cusolver_int_t lddx, void * dWorkspace, size_t lwork_bytes, cusolver_int_t * iter, cusolver_int_t * d_info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCCgesv(hipsolverHandle_t handle, int n, int nrhs, hipFloatComplex* A, int lda, int* devIpiv, hipFloatComplex* B, int ldb, hipFloatComplex* X, int ldx, void* work, size_t lwork, int* niters, int* devInfo);
  // CHECK: status = hipsolverDnCCgesv(handle, ln, lnrhs, &complexA, ldda, &dipiv, &complexB, lddb, &complexX, lddx, &Workspace, lwork_bytes, &iter, &d_info);
  status = hipsolverDnCCgesv(handle, ln, lnrhs, &complexA, ldda, &dipiv, &complexB, lddb, &complexX, lddx, &Workspace, lwork_bytes, &iter, &d_info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDDgesv(cusolverDnHandle_t handle, cusolver_int_t n, cusolver_int_t nrhs, double * dA, cusolver_int_t ldda, cusolver_int_t * dipiv, double * dB, cusolver_int_t lddb, double * dX, cusolver_int_t lddx, void * dWorkspace, size_t lwork_bytes, cusolver_int_t * iter, cusolver_int_t * d_info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDDgesv(hipsolverHandle_t handle, int n, int nrhs, double* A, int lda, int* devIpiv, double* B, int ldb, double* X, int ldx, void* work, size_t lwork, int* niters, int* devInfo);
  // CHECK: status = hipsolverDnDDgesv(handle, ln, lnrhs, &dA, ldda, &dipiv, &dB, lddb, &dX, lddx, &Workspace, lwork_bytes, &iter, &d_info);
  status = hipsolverDnDDgesv(handle, ln, lnrhs, &dA, ldda, &dipiv, &dB, lddb, &dX, lddx, &Workspace, lwork_bytes, &iter, &d_info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSSgesv(cusolverDnHandle_t handle, cusolver_int_t n, cusolver_int_t nrhs, float * dA, cusolver_int_t ldda, cusolver_int_t * dipiv, float * dB, cusolver_int_t lddb, float * dX, cusolver_int_t lddx, void * dWorkspace, size_t lwork_bytes, cusolver_int_t * iter, cusolver_int_t * d_info);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSSgesv(hipsolverHandle_t handle, int n, int nrhs, float* A, int lda, int* devIpiv, float* B, int ldb, float* X, int ldx, void* work, size_t lwork, int* niters, int* devInfo);
  // CHECK: status = hipsolverDnSSgesv(handle, ln, lnrhs, &fA, ldda, &dipiv, &fB, lddb, &fX, lddx, &Workspace, lwork_bytes, &iter, &d_info);
  status = hipsolverDnSSgesv(handle, ln, lnrhs, &fA, ldda, &dipiv, &fB, lddb, &fX, lddx, &Workspace, lwork_bytes, &iter, &d_info);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnZZgesv_bufferSize(cusolverDnHandle_t handle, cusolver_int_t n, cusolver_int_t nrhs, cuDoubleComplex * dA, cusolver_int_t ldda, cusolver_int_t * dipiv, cuDoubleComplex * dB, cusolver_int_t lddb, cuDoubleComplex * dX, cusolver_int_t lddx, void * dWorkspace, size_t * lwork_bytes);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnZZgesv_bufferSize(hipsolverHandle_t handle, int n, int nrhs, hipDoubleComplex* A, int lda, int* devIpiv, hipDoubleComplex* B, int ldb, hipDoubleComplex* X, int ldx, void* work, size_t* lwork);
  // CHECK: status = hipsolverDnZZgesv_bufferSize(handle, ln, lnrhs, &dComplexA, ldda, &dipiv, &dComplexB, lddb, &dComplexX, lddx, &Workspace, &lwork_bytes);
  status = hipsolverDnZZgesv_bufferSize(handle, ln, lnrhs, &dComplexA, ldda, &dipiv, &dComplexB, lddb, &dComplexX, lddx, &Workspace, &lwork_bytes);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCCgesv_bufferSize(cusolverDnHandle_t handle, cusolver_int_t n, cusolver_int_t nrhs, cuComplex * dA, cusolver_int_t ldda, cusolver_int_t * dipiv, cuComplex * dB, cusolver_int_t lddb, cuComplex * dX, cusolver_int_t lddx, void * dWorkspace, size_t * lwork_bytes);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCCgesv_bufferSize(hipsolverHandle_t handle, int n, int nrhs, hipFloatComplex* A, int lda, int* devIpiv, hipFloatComplex* B, int ldb, hipFloatComplex* X, int ldx, void* work, size_t* lwork);
  // CHECK: status = hipsolverDnCCgesv_bufferSize(handle, ln, lnrhs, &complexA, ldda, &dipiv, &complexB, lddb, &complexX, lddx, &Workspace, &lwork_bytes);
  status = hipsolverDnCCgesv_bufferSize(handle, ln, lnrhs, &complexA, ldda, &dipiv, &complexB, lddb, &complexX, lddx, &Workspace, &lwork_bytes);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDDgesv_bufferSize(cusolverDnHandle_t handle, cusolver_int_t n, cusolver_int_t nrhs, double * dA, cusolver_int_t ldda, cusolver_int_t * dipiv, double * dB, cusolver_int_t lddb, double * dX, cusolver_int_t lddx, void * dWorkspace, size_t * lwork_bytes);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDDgesv_bufferSize(hipsolverHandle_t handle, int n, int nrhs, double* A, int lda, int* devIpiv, double* B, int ldb, double* X, int ldx, void* work, size_t* lwork);
  // CHECK: status = hipsolverDnDDgesv_bufferSize(handle, ln, lnrhs, &dA, ldda, &dipiv, &dB, lddb, &dX, lddx, &Workspace, &lwork_bytes);
  status = hipsolverDnDDgesv_bufferSize(handle, ln, lnrhs, &dA, ldda, &dipiv, &dB, lddb, &dX, lddx, &Workspace, &lwork_bytes);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnSSgesv_bufferSize(cusolverDnHandle_t handle, cusolver_int_t n, cusolver_int_t nrhs, float * dA, cusolver_int_t ldda, cusolver_int_t * dipiv, float * dB, cusolver_int_t lddb, float * dX, cusolver_int_t lddx, void * dWorkspace, size_t * lwork_bytes);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnSSgesv_bufferSize(hipsolverHandle_t handle, int n, int nrhs, float* A, int lda, int* devIpiv, float* B, int ldb, float* X, int ldx, void* work, size_t* lwork);
  // CHECK: status = hipsolverDnSSgesv_bufferSize(handle, ln, lnrhs, &fA, ldda, &dipiv, &fB, lddb, &fX, lddx, &Workspace, &lwork_bytes);
  status = hipsolverDnSSgesv_bufferSize(handle, ln, lnrhs, &fA, ldda, &dipiv, &fB, lddb, &fX, lddx, &Workspace, &lwork_bytes);
#endif

  return 0;
}
