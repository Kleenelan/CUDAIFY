// RUN: %run_test hipify "%s" "%t" %hipify_args 2 --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "hipsolver.h"
#include "hipsolver.h"

int main() {
  printf("19. cuSOLVER API to hipSOLVER API synthetic test\n");

  // CHECK: hipsolverHandle_t handle;
  hipsolverHandle_t handle;

  // CHECK: hipsolverStatus_t status;
  hipsolverStatus_t status;

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCreate(cusolverDnHandle_t *handle);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnCreate(hipsolverHandle_t* handle);
  // CHECK: status = hipsolverDnCreate(&handle);
  status = hipsolverDnCreate(&handle);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDestroy(cusolverDnHandle_t handle);
  // HIP: HIPSOLVER_EXPORT hipsolverStatus_t hipsolverDnDestroy(hipsolverHandle_t handle);
  // CHECK: status = hipsolverDnDestroy(handle);
  status = hipsolverDnDestroy(handle);

  return 0;
}
