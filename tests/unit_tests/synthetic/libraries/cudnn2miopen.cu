// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --skip-excluded-preprocessor-conditional-blocks --experimental -roc %clang_args -D__CUDA_API_VERSION_INTERNAL

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "miopen/miopen.h"
#include "cudnn.h"

int main() {
  printf("15. cuDNN API to MIOpen API synthetic test\n");

  // CHECK: miopenStatus_t dnnStatus_t;
  // CHECK-NEXT: miopenStatus_t STATUS_SUCCESS = miopenStatusSuccess;
  // CHECK-NEXT: miopenStatus_t STATUS_NOT_INITIALIZED = miopenStatusNotInitialized;
  // CHECK-NEXT: miopenStatus_t STATUS_ALLOC_FAILED = miopenStatusAllocFailed;
  // CHECK-NEXT: miopenStatus_t STATUS_BAD_PARAM = miopenStatusBadParm;
  // CHECK-NEXT: miopenStatus_t STATUS_INTERNAL_ERROR = miopenStatusInternalError;
  // CHECK-NEXT: miopenStatus_t STATUS_INVALID_VALUE = miopenStatusInvalidValue;
  // CHECK-NEXT: miopenStatus_t STATUS_NOT_SUPPORTED = miopenStatusUnsupportedOp;
  cudnnStatus_t dnnStatus_t;
  cudnnStatus_t STATUS_SUCCESS = CUDNN_STATUS_SUCCESS;
  cudnnStatus_t STATUS_NOT_INITIALIZED = CUDNN_STATUS_NOT_INITIALIZED;
  cudnnStatus_t STATUS_ALLOC_FAILED = CUDNN_STATUS_ALLOC_FAILED;
  cudnnStatus_t STATUS_BAD_PARAM = CUDNN_STATUS_BAD_PARAM;
  cudnnStatus_t STATUS_INTERNAL_ERROR = CUDNN_STATUS_INTERNAL_ERROR;
  cudnnStatus_t STATUS_INVALID_VALUE = CUDNN_STATUS_INVALID_VALUE;
  cudnnStatus_t STATUS_NOT_SUPPORTED = CUDNN_STATUS_NOT_SUPPORTED;

  // CHECK: miopenStatus_t status;
  cudnnStatus_t status;

  // CHECK: miopenHandle_t handle;
  cudnnHandle_t handle;

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnCreate(cudnnHandle_t *handle);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenCreate(miopenHandle_t* handle);
  // CHECK: status = miopenCreate(&handle);
  status = cudnnCreate(&handle);

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnDestroy(cudnnHandle_t handle);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenDestroy(miopenHandle_t handle);
  // CHECK: status = miopenDestroy(handle);
  status = cudnnDestroy(handle);

  const char* const_ch = nullptr;

  // CUDA: const char *CUDNNWINAPI cudnnGetErrorString(cudnnStatus_t status);
  // MIOPEN: MIOPEN_EXPORT const char* miopenGetErrorString(miopenStatus_t error);
  // CHECK: const_ch = miopenGetErrorString(status);
  const_ch = cudnnGetErrorString(status);

  // CHECK: miopenAcceleratorQueue_t streamId;
  hipStream_t streamId;

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnSetStream(cudnnHandle_t handle, cudaStream_t streamId);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenSetStream(miopenHandle_t handle, miopenAcceleratorQueue_t streamID);
  // CHECK: status = miopenSetStream(handle, streamId);
  status = cudnnSetStream(handle, streamId);

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnGetStream(cudnnHandle_t handle, cudaStream_t *streamId);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenGetStream(miopenHandle_t handle, miopenAcceleratorQueue_t* streamID);
  // CHECK: status = miopenGetStream(handle, &streamId);
  status = cudnnGetStream(handle, &streamId);

  return 0;
}
