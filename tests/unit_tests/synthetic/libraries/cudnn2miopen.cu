// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --skip-excluded-preprocessor-conditional-blocks --experimental -roc %clang_args -D__CUDA_API_VERSION_INTERNAL

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "miopen/miopen.h"
#include "cudnn.h"

int main() {
  printf("15. cuDNN API to MIOpen API synthetic test\n");

  // CHECK: miopenStatus_t dnnStatus_t;
  // CHECK-NEXT: miopenStatus_t STATUS_SUCCESS = miopenStatusSuccess;
  // CHECK-NEXT: miopenStatus_t STATUS_NOT_INITIALIZED = miopenStatusNotInitialized;
  // CHECK-NEXT: miopenStatus_t STATUS_ALLOC_FAILED = miopenStatusAllocFailed;
  // CHECK-NEXT: miopenStatus_t STATUS_BAD_PARAM = miopenStatusBadParm;
  // CHECK-NEXT: miopenStatus_t STATUS_INTERNAL_ERROR = miopenStatusInternalError;
  // CHECK-NEXT: miopenStatus_t STATUS_INVALID_VALUE = miopenStatusInvalidValue;
  // CHECK-NEXT: miopenStatus_t STATUS_NOT_SUPPORTED = miopenStatusUnsupportedOp;
  cudnnStatus_t dnnStatus_t;
  cudnnStatus_t STATUS_SUCCESS = CUDNN_STATUS_SUCCESS;
  cudnnStatus_t STATUS_NOT_INITIALIZED = CUDNN_STATUS_NOT_INITIALIZED;
  cudnnStatus_t STATUS_ALLOC_FAILED = CUDNN_STATUS_ALLOC_FAILED;
  cudnnStatus_t STATUS_BAD_PARAM = CUDNN_STATUS_BAD_PARAM;
  cudnnStatus_t STATUS_INTERNAL_ERROR = CUDNN_STATUS_INTERNAL_ERROR;
  cudnnStatus_t STATUS_INVALID_VALUE = CUDNN_STATUS_INVALID_VALUE;
  cudnnStatus_t STATUS_NOT_SUPPORTED = CUDNN_STATUS_NOT_SUPPORTED;

  // CHECK: miopenStatus_t status;
  cudnnStatus_t status;

  // CHECK: miopenHandle_t handle;
  cudnnHandle_t handle;

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnCreate(cudnnHandle_t *handle);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenCreate(miopenHandle_t* handle);
  // CHECK: status = miopenCreate(&handle);
  status = cudnnCreate(&handle);

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnDestroy(cudnnHandle_t handle);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenDestroy(miopenHandle_t handle);
  // CHECK: status = miopenDestroy(handle);
  status = cudnnDestroy(handle);

  const char* const_ch = nullptr;

  // CUDA: const char *CUDNNWINAPI cudnnGetErrorString(cudnnStatus_t status);
  // MIOPEN: MIOPEN_EXPORT const char* miopenGetErrorString(miopenStatus_t error);
  // CHECK: const_ch = miopenGetErrorString(status);
  const_ch = cudnnGetErrorString(status);

  // CHECK: miopenAcceleratorQueue_t streamId;
  hipStream_t streamId;

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnSetStream(cudnnHandle_t handle, cudaStream_t streamId);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenSetStream(miopenHandle_t handle, miopenAcceleratorQueue_t streamID);
  // CHECK: status = miopenSetStream(handle, streamId);
  status = cudnnSetStream(handle, streamId);

  // CUDA: cudnnStatus_t CUDNNWINAPI cudnnGetStream(cudnnHandle_t handle, cudaStream_t *streamId);
  // MIOPEN: MIOPEN_EXPORT miopenStatus_t miopenGetStream(miopenHandle_t handle, miopenAcceleratorQueue_t* streamID);
  // CHECK: status = miopenGetStream(handle, &streamId);
  status = cudnnGetStream(handle, &streamId);

  // CHECK: miopenTensorDescriptor_t tensorDescriptor;
  cudnnTensorDescriptor_t tensorDescriptor;

  // CHECK: miopenConvolutionDescriptor_t convolutionDescriptor;
  cudnnConvolutionDescriptor_t convolutionDescriptor;

  // CHECK: miopenPoolingDescriptor_t poolingDescriptor;
  cudnnPoolingDescriptor_t poolingDescriptor;

  // CHECK: miopenLRNDescriptor_t LRNDescriptor;
  cudnnLRNDescriptor_t LRNDescriptor;

  // CHECK: miopenActivationDescriptor_t activationDescriptor;
  cudnnActivationDescriptor_t activationDescriptor;

  // CHECK: miopenRNNDescriptor_t RNNDescriptor;
  cudnnRNNDescriptor_t RNNDescriptor;

  // CHECK: miopenCTCLossDescriptor_t CTCLossDescriptor;
  cudnnCTCLossDescriptor_t CTCLossDescriptor;

  // CHECK: miopenDropoutDescriptor_t DropoutDescriptor;
  cudnnDropoutDescriptor_t DropoutDescriptor;

  // CHECK: miopenReduceTensorDescriptor_t ReduceTensorDescriptor;
  cudnnReduceTensorDescriptor_t ReduceTensorDescriptor;

  // CHECK: miopenDataType_t dataType;
  // CHECK-NEXT: miopenDataType_t DATA_FLOAT = miopenFloat;
  // CHECK-NEXT: miopenDataType_t DATA_DOUBLE = miopenDouble;
  // CHECK-NEXT: miopenDataType_t DATA_HALF = miopenHalf;
  // CHECK-NEXT: miopenDataType_t DATA_INT8 = miopenInt8;
  // CHECK-NEXT: miopenDataType_t DATA_INT32 = miopenInt32;
  // CHECK-NEXT: miopenDataType_t DATA_INT8x4 = miopenInt8x4;
  // CHECK-NEXT: miopenDataType_t DATA_BFLOAT16 = miopenBFloat16;
  cudnnDataType_t dataType;
  cudnnDataType_t DATA_FLOAT = CUDNN_DATA_FLOAT;
  cudnnDataType_t DATA_DOUBLE = CUDNN_DATA_DOUBLE;
  cudnnDataType_t DATA_HALF = CUDNN_DATA_HALF;
  cudnnDataType_t DATA_INT8 = CUDNN_DATA_INT8;
  cudnnDataType_t DATA_INT32 = CUDNN_DATA_INT32;
  cudnnDataType_t DATA_INT8x4 = CUDNN_DATA_INT8x4;
  cudnnDataType_t DATA_BFLOAT16 = CUDNN_DATA_BFLOAT16;

  // CHECK: miopenTensorOp_t tensorOp;
  // CHECK-NEXT: miopenTensorOp_t OP_TENSOR_ADD = miopenTensorOpAdd;
  // CHECK-NEXT: miopenTensorOp_t OP_TENSOR_MUL = miopenTensorOpMul;
  // CHECK-NEXT: miopenTensorOp_t OP_TENSOR_MIN = miopenTensorOpMin;
  // CHECK-NEXT: miopenTensorOp_t OP_TENSOR_MAX = miopenTensorOpMax;
  cudnnOpTensorOp_t tensorOp;
  cudnnOpTensorOp_t OP_TENSOR_ADD = CUDNN_OP_TENSOR_ADD;
  cudnnOpTensorOp_t OP_TENSOR_MUL = CUDNN_OP_TENSOR_MUL;
  cudnnOpTensorOp_t OP_TENSOR_MIN = CUDNN_OP_TENSOR_MIN;
  cudnnOpTensorOp_t OP_TENSOR_MAX = CUDNN_OP_TENSOR_MAX;

  // CHECK: miopenConvolutionMode_t convolutionMode;
  cudnnConvolutionMode_t convolutionMode;

  // CHECK: miopenPoolingMode_t poolingMode;
  // CHECK-NEXT: miopenPoolingMode_t POOLING_MAX = miopenPoolingMax;
  cudnnPoolingMode_t poolingMode;
  cudnnPoolingMode_t POOLING_MAX = CUDNN_POOLING_MAX;

  // CHECK: miopenLRNMode_t LRNMode;
  // CHECK-NEXT: miopenLRNMode_t LRN_CROSS_CHANNEL_DIM1 = miopenLRNCrossChannel;
  cudnnLRNMode_t LRNMode;
  cudnnLRNMode_t LRN_CROSS_CHANNEL_DIM1 = CUDNN_LRN_CROSS_CHANNEL_DIM1;

  // CHECK: miopenBatchNormMode_t batchNormMode;
  // CHECK-NEXT: miopenBatchNormMode_t BATCHNORM_PER_ACTIVATION = miopenBNPerActivation;
  // CHECK-NEXT: miopenBatchNormMode_t BATCHNORM_SPATIAL = miopenBNSpatial;
  cudnnBatchNormMode_t batchNormMode;
  cudnnBatchNormMode_t BATCHNORM_PER_ACTIVATION = CUDNN_BATCHNORM_PER_ACTIVATION;
  cudnnBatchNormMode_t BATCHNORM_SPATIAL = CUDNN_BATCHNORM_SPATIAL;

  // CHECK: miopenActivationMode_t activationMode;
  // CHECK-NEXT: miopenActivationMode_t ACTIVATION_RELU = miopenActivationRELU;
  // CHECK-NEXT: miopenActivationMode_t ACTIVATION_TANH = miopenActivationTANH;
  // CHECK-NEXT: miopenActivationMode_t ACTIVATION_CLIPPED_RELU = miopenActivationCLIPPEDRELU;
  // CHECK-NEXT: miopenActivationMode_t ACTIVATION_ELU = miopenActivationELU;
  // CHECK-NEXT: miopenActivationMode_t ACTIVATION_IDENTITY = miopenActivationPASTHRU;
  cudnnActivationMode_t activationMode;
  cudnnActivationMode_t ACTIVATION_RELU = CUDNN_ACTIVATION_RELU;
  cudnnActivationMode_t ACTIVATION_TANH = CUDNN_ACTIVATION_TANH;
  cudnnActivationMode_t ACTIVATION_CLIPPED_RELU = CUDNN_ACTIVATION_CLIPPED_RELU;
  cudnnActivationMode_t ACTIVATION_ELU = CUDNN_ACTIVATION_ELU;
  cudnnActivationMode_t ACTIVATION_IDENTITY = CUDNN_ACTIVATION_IDENTITY;

  // CHECK: miopenSoftmaxAlgorithm_t softmaxAlgorithm;
  // CHECK-NEXT: miopenSoftmaxAlgorithm_t SOFTMAX_FAST = MIOPEN_SOFTMAX_FAST;
  // CHECK-NEXT: miopenSoftmaxAlgorithm_t SOFTMAX_ACCURATE = MIOPEN_SOFTMAX_ACCURATE;
  // CHECK-NEXT: miopenSoftmaxAlgorithm_t SOFTMAX_LOG = MIOPEN_SOFTMAX_LOG;
  cudnnSoftmaxAlgorithm_t softmaxAlgorithm;
  cudnnSoftmaxAlgorithm_t SOFTMAX_FAST = CUDNN_SOFTMAX_FAST;
  cudnnSoftmaxAlgorithm_t SOFTMAX_ACCURATE = CUDNN_SOFTMAX_ACCURATE;
  cudnnSoftmaxAlgorithm_t SOFTMAX_LOG = CUDNN_SOFTMAX_LOG;

  // CHECK: miopenReduceTensorOp_t reduceTensorOp;
  // CHECK-NEXT: miopenReduceTensorOp_t REDUCE_TENSOR_ADD = MIOPEN_REDUCE_TENSOR_ADD;
  // CHECK-NEXT: miopenReduceTensorOp_t REDUCE_TENSOR_MUL = MIOPEN_REDUCE_TENSOR_MUL;
  // CHECK-NEXT: miopenReduceTensorOp_t REDUCE_TENSOR_MIN = MIOPEN_REDUCE_TENSOR_MIN;
  // CHECK-NEXT: miopenReduceTensorOp_t REDUCE_TENSOR_MAX = MIOPEN_REDUCE_TENSOR_MAX;
  // CHECK-NEXT: miopenReduceTensorOp_t REDUCE_TENSOR_AMAX = MIOPEN_REDUCE_TENSOR_AMAX;
  // CHECK-NEXT: miopenReduceTensorOp_t REDUCE_TENSOR_AVG = MIOPEN_REDUCE_TENSOR_AVG;
  // CHECK-NEXT: miopenReduceTensorOp_t REDUCE_TENSOR_NORM1 = MIOPEN_REDUCE_TENSOR_NORM1;
  // CHECK-NEXT: miopenReduceTensorOp_t REDUCE_TENSOR_NORM2 = MIOPEN_REDUCE_TENSOR_NORM2;
  cudnnReduceTensorOp_t reduceTensorOp;
  cudnnReduceTensorOp_t REDUCE_TENSOR_ADD = CUDNN_REDUCE_TENSOR_ADD;
  cudnnReduceTensorOp_t REDUCE_TENSOR_MUL = CUDNN_REDUCE_TENSOR_MUL;
  cudnnReduceTensorOp_t REDUCE_TENSOR_MIN = CUDNN_REDUCE_TENSOR_MIN;
  cudnnReduceTensorOp_t REDUCE_TENSOR_MAX = CUDNN_REDUCE_TENSOR_MAX;
  cudnnReduceTensorOp_t REDUCE_TENSOR_AMAX = CUDNN_REDUCE_TENSOR_AMAX;
  cudnnReduceTensorOp_t REDUCE_TENSOR_AVG = CUDNN_REDUCE_TENSOR_AVG;
  cudnnReduceTensorOp_t REDUCE_TENSOR_NORM1 = CUDNN_REDUCE_TENSOR_NORM1;
  cudnnReduceTensorOp_t REDUCE_TENSOR_NORM2 = CUDNN_REDUCE_TENSOR_NORM2;

  return 0;
}
