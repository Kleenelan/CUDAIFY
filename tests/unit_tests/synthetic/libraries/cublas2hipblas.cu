#include "hip/hip_runtime.h"
// RUN: %run_test hipify "%s" "%t" %hipify_args 2 --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "hipblas.h"
// CHECK-NOT: #include "hipblas.h"
#include "hipblas.h"
#include "hipblas.h"
// CHECK-NOT: #include "hipblas.h"

int main() {
  printf("14. cuBLAS API to hipBLAS API synthetic test\n");

  // CHECK: hipblasOperation_t blasOperation;
  // CHECK-NEXT: hipblasOperation_t BLAS_OP_N = HIPBLAS_OP_N;
  // CHECK-NEXT: hipblasOperation_t BLAS_OP_T = HIPBLAS_OP_T;
  // CHECK-NEXT: hipblasOperation_t BLAS_OP_C = HIPBLAS_OP_C;
  hipblasOperation_t blasOperation;
  hipblasOperation_t BLAS_OP_N = HIPBLAS_OP_N;
  hipblasOperation_t BLAS_OP_T = HIPBLAS_OP_T;
  hipblasOperation_t BLAS_OP_C = HIPBLAS_OP_C;

  // CHECK: hipblasStatus_t blasStatus;
  // CHECK-NEXT: hipblasStatus_t blasStatus_t;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_SUCCESS = HIPBLAS_STATUS_SUCCESS;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_NOT_INITIALIZED = HIPBLAS_STATUS_NOT_INITIALIZED;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_ALLOC_FAILED = HIPBLAS_STATUS_ALLOC_FAILED;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_INVALID_VALUE = HIPBLAS_STATUS_INVALID_VALUE;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_MAPPING_ERROR = HIPBLAS_STATUS_MAPPING_ERROR;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_EXECUTION_FAILED = HIPBLAS_STATUS_EXECUTION_FAILED;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_INTERNAL_ERROR = HIPBLAS_STATUS_INTERNAL_ERROR;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_NOT_SUPPORTED = HIPBLAS_STATUS_NOT_SUPPORTED;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_ARCH_MISMATCH = HIPBLAS_STATUS_ARCH_MISMATCH;
  hipblasStatus_t blasStatus;
  hipblasStatus_t blasStatus_t;
  hipblasStatus_t BLAS_STATUS_SUCCESS = HIPBLAS_STATUS_SUCCESS;
  hipblasStatus_t BLAS_STATUS_NOT_INITIALIZED = HIPBLAS_STATUS_NOT_INITIALIZED;
  hipblasStatus_t BLAS_STATUS_ALLOC_FAILED = HIPBLAS_STATUS_ALLOC_FAILED;
  hipblasStatus_t BLAS_STATUS_INVALID_VALUE = HIPBLAS_STATUS_INVALID_VALUE;
  hipblasStatus_t BLAS_STATUS_MAPPING_ERROR = HIPBLAS_STATUS_MAPPING_ERROR;
  hipblasStatus_t BLAS_STATUS_EXECUTION_FAILED = HIPBLAS_STATUS_EXECUTION_FAILED;
  hipblasStatus_t BLAS_STATUS_INTERNAL_ERROR = HIPBLAS_STATUS_INTERNAL_ERROR;
  hipblasStatus_t BLAS_STATUS_NOT_SUPPORTED = HIPBLAS_STATUS_NOT_SUPPORTED;
  hipblasStatus_t BLAS_STATUS_ARCH_MISMATCH = HIPBLAS_STATUS_ARCH_MISMATCH;

  // CHECK: hipblasFillMode_t blasFillMode;
  // CHECK-NEXT: hipblasFillMode_t BLAS_FILL_MODE_LOWER = HIPBLAS_FILL_MODE_LOWER;
  // CHECK-NEXT: hipblasFillMode_t BLAS_FILL_MODE_UPPER = HIPBLAS_FILL_MODE_UPPER;
  hipblasFillMode_t blasFillMode;
  hipblasFillMode_t BLAS_FILL_MODE_LOWER = HIPBLAS_FILL_MODE_LOWER;
  hipblasFillMode_t BLAS_FILL_MODE_UPPER = HIPBLAS_FILL_MODE_UPPER;

  // CHECK: hipblasDiagType_t blasDiagType;
  // CHECK-NEXT: hipblasDiagType_t BLAS_DIAG_NON_UNIT = HIPBLAS_DIAG_NON_UNIT;
  // CHECK-NEXT: hipblasDiagType_t BLAS_DIAG_UNIT = HIPBLAS_DIAG_UNIT;
  hipblasDiagType_t blasDiagType;
  hipblasDiagType_t BLAS_DIAG_NON_UNIT = HIPBLAS_DIAG_NON_UNIT;
  hipblasDiagType_t BLAS_DIAG_UNIT = HIPBLAS_DIAG_UNIT;

  // CHECK: hipblasSideMode_t blasSideMode;
  // CHECK-NEXT: hipblasSideMode_t BLAS_SIDE_LEFT = HIPBLAS_SIDE_LEFT;
  // CHECK-NEXT: hipblasSideMode_t BLAS_SIDE_RIGHT = HIPBLAS_SIDE_RIGHT;
  hipblasSideMode_t blasSideMode;
  hipblasSideMode_t BLAS_SIDE_LEFT = HIPBLAS_SIDE_LEFT;
  hipblasSideMode_t BLAS_SIDE_RIGHT = HIPBLAS_SIDE_RIGHT;

  // CHECK: hipblasPointerMode_t blasPointerMode;
  // CHECK-NEXT: hipblasPointerMode_t BLAS_POINTER_MODE_HOST = HIPBLAS_POINTER_MODE_HOST;
  // CHECK-NEXT: hipblasPointerMode_t BLAS_POINTER_MODE_DEVICE = HIPBLAS_POINTER_MODE_DEVICE;
  hipblasPointerMode_t blasPointerMode;
  hipblasPointerMode_t BLAS_POINTER_MODE_HOST = HIPBLAS_POINTER_MODE_HOST;
  hipblasPointerMode_t BLAS_POINTER_MODE_DEVICE = HIPBLAS_POINTER_MODE_DEVICE;

  // CHECK: hipblasAtomicsMode_t blasAtomicsMode;
  // CHECK-NEXT: hipblasAtomicsMode_t BLAS_ATOMICS_NOT_ALLOWED = HIPBLAS_ATOMICS_NOT_ALLOWED;
  // CHECK-NEXT: hipblasAtomicsMode_t BLAS_ATOMICS_ALLOWED = HIPBLAS_ATOMICS_ALLOWED;
  hipblasAtomicsMode_t blasAtomicsMode;
  hipblasAtomicsMode_t BLAS_ATOMICS_NOT_ALLOWED = HIPBLAS_ATOMICS_NOT_ALLOWED;
  hipblasAtomicsMode_t BLAS_ATOMICS_ALLOWED = HIPBLAS_ATOMICS_ALLOWED;

  // CHECK: hipblasHandle_t blasHandle;
  hipblasHandle_t blasHandle;

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasGetAtomicsMode(hipblasHandle_t handle, hipblasAtomicsMode_t* mode);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetAtomicsMode(hipblasHandle_t handle, hipblasAtomicsMode_t* atomics_mode);
  // CHECK: blasStatus = hipblasGetAtomicsMode(blasHandle, &blasAtomicsMode);
  blasStatus = hipblasGetAtomicsMode(blasHandle, &blasAtomicsMode);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSetAtomicsMode(hipblasHandle_t handle, hipblasAtomicsMode_t mode);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetAtomicsMode(hipblasHandle_t handle, hipblasAtomicsMode_t atomics_mode);
  // CHECK: blasStatus = hipblasSetAtomicsMode(blasHandle, blasAtomicsMode);
  blasStatus = hipblasSetAtomicsMode(blasHandle, blasAtomicsMode);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCreate(hipblasHandle_t* handle);
  // CUDA: #define hipblasCreate hipblasCreate
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCreate(hipblasHandle_t* handle);
  // CHECK: blasStatus = hipblasCreate(&blasHandle);
  // CHECK-NEXT: blasStatus = hipblasCreate(&blasHandle);
  blasStatus = hipblasCreate(&blasHandle);
  blasStatus = hipblasCreate(&blasHandle);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDestroy(hipblasHandle_t handle);
  // CUDA: #define hipblasDestroy hipblasDestroy
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDestroy(hipblasHandle_t handle);
  // CHECK: blasStatus = hipblasDestroy(blasHandle);
  // CHECK-NEXT: blasStatus = hipblasDestroy(blasHandle);
  blasStatus = hipblasDestroy(blasHandle);
  blasStatus = hipblasDestroy(blasHandle);

  // CHECK: hipStream_t stream;
  hipStream_t stream;

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSetStream(hipblasHandle_t handle, hipStream_t streamId);
  // CUDA: #define hipblasSetStream hipblasSetStream
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetStream(hipblasHandle_t handle, hipStream_t streamId);
  // CHECK: blasStatus = hipblasSetStream(blasHandle, stream);
  // CHECK-NEXT: blasStatus = hipblasSetStream(blasHandle, stream);
  blasStatus = hipblasSetStream(blasHandle, stream);
  blasStatus = hipblasSetStream(blasHandle, stream);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasGetStream(hipblasHandle_t handle, hipStream_t* streamId);
  // CUDA: #define hipblasGetStream hipblasGetStream
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetStream(hipblasHandle_t handle, hipStream_t* streamId);
  // CHECK: blasStatus = hipblasGetStream(blasHandle, &stream);
  // CHECK-NEXT: blasStatus = hipblasGetStream(blasHandle, &stream);
  blasStatus = hipblasGetStream(blasHandle, &stream);
  blasStatus = hipblasGetStream(blasHandle, &stream);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSetPointerMode(hipblasHandle_t handle, hipblasPointerMode_t mode);
  // CUDA: #define hipblasSetPointerMode hipblasSetPointerMode
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetPointerMode(hipblasHandle_t handle, hipblasPointerMode_t mode);
  // CHECK: blasStatus = hipblasSetPointerMode(blasHandle, blasPointerMode);
  // CHECK-NEXT: blasStatus = hipblasSetPointerMode(blasHandle, blasPointerMode);
  blasStatus = hipblasSetPointerMode(blasHandle, blasPointerMode);
  blasStatus = hipblasSetPointerMode(blasHandle, blasPointerMode);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasGetPointerMode(hipblasHandle_t handle, hipblasPointerMode_t* mode);
  // CUDA: #define hipblasGetPointerMode hipblasGetPointerMode
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetPointerMode(hipblasHandle_t handle, hipblasPointerMode_t* mode);
  // CHECK: blasStatus = hipblasGetPointerMode(blasHandle, &blasPointerMode);
  // CHECK-NEXT: blasStatus = hipblasGetPointerMode(blasHandle, &blasPointerMode);
  blasStatus = hipblasGetPointerMode(blasHandle, &blasPointerMode);
  blasStatus = hipblasGetPointerMode(blasHandle, &blasPointerMode);

  int n = 0;
  int m = 0;
  int num = 0;
  int lda = 0;
  int res = 0;
  int incx = 0;
  int incy = 0;
  int k = 0;
  int kl = 0;
  int ku = 0;
  void* image = nullptr;
  void* image_2 = nullptr;
  void* deviceptr = nullptr;

  // CUDA: hipblasStatus_t CUBLASWINAPI hipblasSetVector(int n, int elemSize, const void* x, int incx, void* devicePtr, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetVector(int n, int elemSize, const void* x, int incx, void* y, int incy);
  // CHECK: blasStatus = hipblasSetVector(n, num, image, incx, image_2, incy);
  blasStatus = hipblasSetVector(n, num, image, incx, image_2, incy);

  // CUDA: hipblasStatus_t CUBLASWINAPI hipblasGetVector(int n, int elemSize, const void* x, int incx, void* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetVector(int n, int elemSize, const void* x, int incx, void* y, int incy);
  // CHECK: blasStatus = hipblasGetVector(n, num, image, incx, image_2, incy);
  blasStatus = hipblasGetVector(n, num, image, incx, image_2, incy);

  // CUDA: hipblasStatus_t CUBLASWINAPI hipblasSetVectorAsync(int n, int elemSize, const void* hostPtr, int incx, void* devicePtr, int incy, hipStream_t stream);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetVectorAsync(int n, int elemSize, const void* x, int incx, void* y, int incy, hipStream_t stream);
  // CHECK: blasStatus = hipblasSetVectorAsync(n, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasSetVectorAsync(n, num, image, incx, image_2, incy, stream);

  // CUDA: hipblasStatus_t CUBLASWINAPI hipblasGetVectorAsync(int n, int elemSize, const void* devicePtr, int incx, void* hostPtr, int incy, hipStream_t stream);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetVectorAsync(int n, int elemSize, const void* x, int incx, void* y, int incy, hipStream_t stream);
  // CHECK: blasStatus = hipblasGetVectorAsync(n, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasGetVectorAsync(n, num, image, incx, image_2, incy, stream);

  int rows = 0;
  int cols = 0;

  // CUDA: hipblasStatus_t CUBLASWINAPI hipblasSetMatrix(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetMatrix(int rows, int cols, int elemSize, const void* AP, int lda, void* BP, int ldb);
  // CHECK: blasStatus = hipblasSetMatrix(rows, cols, num, image, incx, image_2, incy);
  blasStatus = hipblasSetMatrix(rows, cols, num, image, incx, image_2, incy);

  // CUDA: hipblasStatus_t CUBLASWINAPI hipblasGetMatrix(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetMatrix(int rows, int cols, int elemSize, const void* AP, int lda, void* BP, int ldb);
  // CHECK: blasStatus = hipblasGetMatrix(rows, cols, num, image, incx, image_2, incy);
  blasStatus = hipblasGetMatrix(rows, cols, num, image, incx, image_2, incy);

  // CUDA: hipblasStatus_t CUBLASWINAPI hipblasSetMatrixAsync(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb, hipStream_t stream);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSetMatrixAsync(int rows, int cols, int elemSize, const void* AP, int lda, void* BP, int ldb, hipStream_t stream);
  // CHECK: blasStatus = hipblasSetMatrixAsync(rows, cols, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasSetMatrixAsync(rows, cols, num, image, incx, image_2, incy, stream);

  // CUDA: hipblasStatus_t CUBLASWINAPI hipblasGetMatrixAsync(int rows, int cols, int elemSize, const void* A, int lda, void* B, int ldb, hipStream_t stream);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasGetMatrixAsync(int rows, int cols, int elemSize, const void* AP, int lda, void* BP, int ldb, hipStream_t stream);
  // CHECK: blasStatus = hipblasGetMatrixAsync(rows, cols, num, image, incx, image_2, incy, stream);
  blasStatus = hipblasGetMatrixAsync(rows, cols, num, image, incx, image_2, incy, stream);

  float fa = 0;
  float fA = 0;
  float fb = 0;
  float fx = 0;
  float fx1 = 0;
  float fy = 0;
  float fy1 = 0;
  float fc = 0;
  float fs = 0;
  float fd1 = 0;
  float fd2 = 0;
  float fresult = 0;

  double da = 0;
  double dA = 0;
  double db = 0;
  double dx = 0;
  double dx1 = 0;
  double dy = 0;
  double dy1 = 0;
  double dc = 0;
  double ds = 0;
  double dd1 = 0;
  double dd2 = 0;
  double dresult = 0;

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSnrm2(hipblasHandle_t handle, int n, const float* x, int incx, float* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSnrm2(hipblasHandle_t handle, int n, const float* x, int incx, float* result);
  // CHECK: blasStatus = hipblasSnrm2(blasHandle, n, &fx, incx, &fresult);
  // CHECK-NEXT: blasStatus = hipblasSnrm2(blasHandle, n, &fx, incx, &fresult);
  blasStatus = hipblasSnrm2(blasHandle, n, &fx, incx, &fresult);
  blasStatus = hipblasSnrm2(blasHandle, n, &fx, incx, &fresult);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDnrm2(hipblasHandle_t handle, int n, const double* x, int incx, double* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDnrm2(hipblasHandle_t handle, int n, const double* x, int incx, double* result);
  // CHECK: blasStatus = hipblasDnrm2(blasHandle, n, &dx, incx, &dresult);
  // CHECK-NEXT: blasStatus = hipblasDnrm2(blasHandle, n, &dx, incx, &dresult);
  blasStatus = hipblasDnrm2(blasHandle, n, &dx, incx, &dresult);
  blasStatus = hipblasDnrm2(blasHandle, n, &dx, incx, &dresult);

  // CHECK: hipComplex complex, complexa, complexA, complexx, complexy, complexs, complexb;
  hipComplex complex, complexa, complexA, complexx, complexy, complexs, complexb;
  // CHECK: hipDoubleComplex dcomplex, dcomplexa, dcomplexA, dcomplexx, dcomplexy, dcomplexs, dcomplexb;
  hipDoubleComplex dcomplex, dcomplexa, dcomplexA, dcomplexx, dcomplexy, dcomplexs, dcomplexb;

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasScnrm2(hipblasHandle_t handle, int n, const hipComplex* x, int incx, float* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasScnrm2(hipblasHandle_t handle, int n, const hipblasComplex* x, int incx, float* result);
  // CHECK: blasStatus = hipblasScnrm2(blasHandle, n, &complex, incx, &fresult);
  // CHECK-NEXT: blasStatus = hipblasScnrm2(blasHandle, n, &complex, incx, &fresult);
  blasStatus = hipblasScnrm2(blasHandle, n, &complex, incx, &fresult);
  blasStatus = hipblasScnrm2(blasHandle, n, &complex, incx, &fresult);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDznrm2(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, double* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDznrm2(hipblasHandle_t handle, int n, const hipblasDoubleComplex* x, int incx, double* result);
  // CHECK: blasStatus = hipblasDznrm2(blasHandle, n, &dcomplex, incx, &dresult);
  // CHECK-NEXT: blasStatus = hipblasDznrm2(blasHandle, n, &dcomplex, incx, &dresult);
  blasStatus = hipblasDznrm2(blasHandle, n, &dcomplex, incx, &dresult);
  blasStatus = hipblasDznrm2(blasHandle, n, &dcomplex, incx, &dresult);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSdot(hipblasHandle_t handle, int n, const float* x, int incx, const float* y, int incy, float* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSdot(hipblasHandle_t handle, int n, const float* x, int incx, const float* y, int incy, float* result);
  // CHECK: blasStatus = hipblasSdot(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  // CHECK-NEXT: blasStatus = hipblasSdot(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  blasStatus = hipblasSdot(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  blasStatus = hipblasSdot(blasHandle, n, &fx, incx, &fy, incy, &fresult);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDdot(hipblasHandle_t handle, int n, const double* x, int incx, const double* y, int incy, double* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDdot(hipblasHandle_t handle, int n, const double* x, int incx, const double* y, int incy, double* result);
  // CHECK: blasStatus = hipblasDdot(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  // CHECK-NEXT: blasStatus = hipblasDdot(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  blasStatus = hipblasDdot(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  blasStatus = hipblasDdot(blasHandle, n, &dx, incx, &dy, incy, &dresult);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCdotu(hipblasHandle_t handle, int n, const hipComplex* x, int incx, const hipComplex* y, int incy, hipComplex* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCdotu(hipblasHandle_t handle, int n, const hipblasComplex* x, int incx, const hipblasComplex* y, int incy, hipblasComplex* result);
  // CHECK: blasStatus = hipblasCdotu(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  // CHECK-NEXT: blasStatus = hipblasCdotu(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  blasStatus = hipblasCdotu(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  blasStatus = hipblasCdotu(blasHandle, n, &complexx, incx, &complexy, incy, &complex);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCdotc(hipblasHandle_t handle, int n, const hipComplex* x, int incx, const hipComplex* y, int incy, hipComplex* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCdotc(hipblasHandle_t handle, int n, const hipblasComplex* x, int incx, const hipblasComplex* y, int incy, hipblasComplex* result);
  // CHECK: blasStatus = hipblasCdotc(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  // CHECK-NEXT: blasStatus = hipblasCdotc(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  blasStatus = hipblasCdotc(blasHandle, n, &complexx, incx, &complexy, incy, &complex);
  blasStatus = hipblasCdotc(blasHandle, n, &complexx, incx, &complexy, incy, &complex);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZdotu(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, const hipDoubleComplex* y, int incy, hipDoubleComplex* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZdotu(hipblasHandle_t handle, int n, const hipblasDoubleComplex* x, int incx, const hipblasDoubleComplex* y, int incy, hipblasDoubleComplex* result);
  // CHECK: blasStatus = hipblasZdotu(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  // CHECK-NEXT: blasStatus = hipblasZdotu(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  blasStatus = hipblasZdotu(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  blasStatus = hipblasZdotu(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZdotc(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, const hipDoubleComplex* y, int incy, hipDoubleComplex* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZdotc(hipblasHandle_t handle, int n, const hipblasDoubleComplex* x, int incx, const hipblasDoubleComplex* y, int incy, hipblasDoubleComplex* result);
  // CHECK: blasStatus = hipblasZdotc(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  // CHECK-NEXT: blasStatus = hipblasZdotc(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  blasStatus = hipblasZdotc(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);
  blasStatus = hipblasZdotc(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dcomplex);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSscal(hipblasHandle_t handle, int n, const float* alpha, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSscal(hipblasHandle_t handle, int n, const float* alpha, float* x, int incx);
  // CHECK: blasStatus = hipblasSscal(blasHandle, n, &fy, &fx, incx);
  // CHECK-NEXT: blasStatus = hipblasSscal(blasHandle, n, &fy, &fx, incx);
  blasStatus = hipblasSscal(blasHandle, n, &fy, &fx, incx);
  blasStatus = hipblasSscal(blasHandle, n, &fy, &fx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDscal(hipblasHandle_t handle, int n, const double* alpha, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDscal(hipblasHandle_t handle, int n, const double* alpha, double* x, int incx);
  // CHECK: blasStatus = hipblasDscal(blasHandle, n, &dx, &dy, incx);
  // CHECK-NEXT: blasStatus = hipblasDscal(blasHandle, n, &dx, &dy, incx);
  blasStatus = hipblasDscal(blasHandle, n, &dx, &dy, incx);
  blasStatus = hipblasDscal(blasHandle, n, &dx, &dy, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCscal(hipblasHandle_t handle, int n, const hipComplex* alpha, hipComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCscal(hipblasHandle_t handle, int n, const hipblasComplex* alpha, hipblasComplex* x, int incx);
  // CHECK: blasStatus = hipblasCscal(blasHandle, n, &complexa, &complexx, incx);
  // CHECK-NEXT: blasStatus = hipblasCscal(blasHandle, n, &complexa, &complexx, incx);
  blasStatus = hipblasCscal(blasHandle, n, &complexa, &complexx, incx);
  blasStatus = hipblasCscal(blasHandle, n, &complexa, &complexx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCsscal(hipblasHandle_t handle, int n, const float* alpha, hipComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsscal(hipblasHandle_t handle, int n, const float* alpha, hipblasComplex* x, int incx);
  // CHECK: blasStatus = hipblasCsscal(blasHandle, n, &fx, &complexx, incx);
  // CHECK-NEXT: blasStatus = hipblasCsscal(blasHandle, n, &fx, &complexx, incx);
  blasStatus = hipblasCsscal(blasHandle, n, &fx, &complexx, incx);
  blasStatus = hipblasCsscal(blasHandle, n, &fx, &complexx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZscal(hipblasHandle_t handle, int n, const hipDoubleComplex* alpha, hipDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZscal(hipblasHandle_t handle, int n, const hipblasDoubleComplex* alpha, hipblasDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZscal(blasHandle, n, &dcomplexa, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = hipblasZscal(blasHandle, n, &dcomplexa, &dcomplexx, incx);
  blasStatus = hipblasZscal(blasHandle, n, &dcomplexa, &dcomplexx, incx);
  blasStatus = hipblasZscal(blasHandle, n, &dcomplexa, &dcomplexx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZdscal(hipblasHandle_t handle, int n, const double* alpha, hipDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZdscal(hipblasHandle_t handle, int n, const double* alpha, hipblasDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZdscal(blasHandle, n, &dx, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = hipblasZdscal(blasHandle, n, &dx, &dcomplexx, incx);
  blasStatus = hipblasZdscal(blasHandle, n, &dx, &dcomplexx, incx);
  blasStatus = hipblasZdscal(blasHandle, n, &dx, &dcomplexx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSaxpy(hipblasHandle_t handle, int n, const float* alpha, const float* x, int incx, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSaxpy(hipblasHandle_t handle, int n, const float* alpha, const float* x, int incx, float* y, int incy);
  // CHECK: blasStatus = hipblasSaxpy(blasHandle, n, &fa, &fx, incx, &fy, incy);
  // CHECK-NEXT: blasStatus = hipblasSaxpy(blasHandle, n, &fa, &fx, incx, &fy, incy);
  blasStatus = hipblasSaxpy(blasHandle, n, &fa, &fx, incx, &fy, incy);
  blasStatus = hipblasSaxpy(blasHandle, n, &fa, &fx, incx, &fy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDaxpy(hipblasHandle_t handle, int n, const double* alpha, const double* x, int incx, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDaxpy(hipblasHandle_t handle, int n, const double* alpha, const double* x, int incx, double* y, int incy);
  // CHECK: blasStatus = hipblasDaxpy(blasHandle, n, &da, &dx, incx, &dy, incy);
  // CHECK-NEXT: blasStatus = hipblasDaxpy(blasHandle, n, &da, &dx, incx, &dy, incy);
  blasStatus = hipblasDaxpy(blasHandle, n, &da, &dx, incx, &dy, incy);
  blasStatus = hipblasDaxpy(blasHandle, n, &da, &dx, incx, &dy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCaxpy(hipblasHandle_t handle, int n, const hipComplex* alpha, const hipComplex* x, int incx, hipComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCaxpy(hipblasHandle_t handle, int n, const hipblasComplex* alpha, const hipblasComplex* x, int incx, hipblasComplex* y, int incy);
  // CHECK: blasStatus = hipblasCaxpy(blasHandle, n, &complexa, &complexx, incx, &complexy, incy);
  // CHECK-NEXT: blasStatus = hipblasCaxpy(blasHandle, n, &complexa, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCaxpy(blasHandle, n, &complexa, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCaxpy(blasHandle, n, &complexa, &complexx, incx, &complexy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZaxpy(hipblasHandle_t handle, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* x, int incx, hipDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZaxpy(hipblasHandle_t handle, int n, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* x, int incx, hipblasDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZaxpy(blasHandle, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = hipblasZaxpy(blasHandle, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZaxpy(blasHandle, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZaxpy(blasHandle, n, &dcomplexa, &dcomplexx, incx, &dcomplexy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasScopy(hipblasHandle_t handle, int n, const float* x, int incx, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasScopy(hipblasHandle_t handle, int n, const float* x, int incx, float* y, int incy);
  // CHECK: blasStatus = hipblasScopy(blasHandle, n, &fx, incx, &fy, incy);
  // CHECK-NEXT: blasStatus = hipblasScopy(blasHandle, n, &fx, incx, &fy, incy);
  blasStatus = hipblasScopy(blasHandle, n, &fx, incx, &fy, incy);
  blasStatus = hipblasScopy(blasHandle, n, &fx, incx, &fy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDcopy(hipblasHandle_t handle, int n, const double* x, int incx, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDcopy(hipblasHandle_t handle, int n, const double* x, int incx, double* y, int incy);
  // CHECK: blasStatus = hipblasDcopy(blasHandle, n, &dx, incx, &dy, incy);
  // CHECK-NEXT: blasStatus = hipblasDcopy(blasHandle, n, &dx, incx, &dy, incy);
  blasStatus = hipblasDcopy(blasHandle, n, &dx, incx, &dy, incy);
  blasStatus = hipblasDcopy(blasHandle, n, &dx, incx, &dy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCcopy(hipblasHandle_t handle, int n, const hipComplex* x, int incx, hipComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCcopy(hipblasHandle_t handle, int n, const hipblasComplex* x, int incx, hipblasComplex* y, int incy);
  // CHECK: blasStatus = hipblasCcopy(blasHandle, n, &complexx, incx, &complexy, incy);
  // CHECK-NEXT: blasStatus = hipblasCcopy(blasHandle, n, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCcopy(blasHandle, n, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCcopy(blasHandle, n, &complexx, incx, &complexy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZcopy(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, hipDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZcopy(hipblasHandle_t handle, int n, const hipblasDoubleComplex* x, int incx, hipblasDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZcopy(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = hipblasZcopy(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZcopy(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZcopy(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSswap(hipblasHandle_t handle, int n, float* x, int incx, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSswap(hipblasHandle_t handle, int n, float* x, int incx, float* y, int incy);
  // CHECK: blasStatus = hipblasSswap(blasHandle, n, &fx, incx, &fy, incy);
  // CHECK-NEXT: blasStatus = hipblasSswap(blasHandle, n, &fx, incx, &fy, incy);
  blasStatus = hipblasSswap(blasHandle, n, &fx, incx, &fy, incy);
  blasStatus = hipblasSswap(blasHandle, n, &fx, incx, &fy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDswap(hipblasHandle_t handle, int n, double* x, int incx, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDswap(hipblasHandle_t handle, int n, double* x, int incx, double* y, int incy);
  // CHECK: blasStatus = hipblasDswap(blasHandle, n, &dx, incx, &dy, incy);
  // CHECK-NEXT: blasStatus = hipblasDswap(blasHandle, n, &dx, incx, &dy, incy);
  blasStatus = hipblasDswap(blasHandle, n, &dx, incx, &dy, incy);
  blasStatus = hipblasDswap(blasHandle, n, &dx, incx, &dy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCswap(hipblasHandle_t handle, int n, hipComplex* x, int incx, hipComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCswap(hipblasHandle_t handle, int n, hipblasComplex* x, int incx, hipblasComplex* y, int incy);
  // CHECK: blasStatus = hipblasCswap(blasHandle, n, &complexx, incx, &complexy, incy);
  // CHECK-NEXT: blasStatus = hipblasCswap(blasHandle, n, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCswap(blasHandle, n, &complexx, incx, &complexy, incy);
  blasStatus = hipblasCswap(blasHandle, n, &complexx, incx, &complexy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZswap(hipblasHandle_t handle, int n, hipDoubleComplex* x, int incx, hipDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZswap(hipblasHandle_t handle, int n, hipblasDoubleComplex* x, int incx, hipblasDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZswap(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = hipblasZswap(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZswap(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);
  blasStatus = hipblasZswap(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasIsamax(hipblasHandle_t handle, int n, const float* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIsamax(hipblasHandle_t handle, int n, const float* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIsamax(blasHandle, n, &fx, incx, &res);
  // CHECK-NEXT: blasStatus = hipblasIsamax(blasHandle, n, &fx, incx, &res);
  blasStatus = hipblasIsamax(blasHandle, n, &fx, incx, &res);
  blasStatus = hipblasIsamax(blasHandle, n, &fx, incx, &res);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasIdamax(hipblasHandle_t handle, int n, const double* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIdamax(hipblasHandle_t handle, int n, const double* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIdamax(blasHandle, n, &dx, incx, &res);
  // CHECK-NEXT: blasStatus = hipblasIdamax(blasHandle, n, &dx, incx, &res);
  blasStatus = hipblasIdamax(blasHandle, n, &dx, incx, &res);
  blasStatus = hipblasIdamax(blasHandle, n, &dx, incx, &res);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasIcamax(hipblasHandle_t handle, int n, const hipComplex* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIcamax(hipblasHandle_t handle, int n, const hipblasComplex* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIcamax(blasHandle, n, &complexx, incx, &res);
  // CHECK-NEXT: blasStatus = hipblasIcamax(blasHandle, n, &complexx, incx, &res);
  blasStatus = hipblasIcamax(blasHandle, n, &complexx, incx, &res);
  blasStatus = hipblasIcamax(blasHandle, n, &complexx, incx, &res);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasIzamax(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIzamax(hipblasHandle_t handle, int n, const hipblasDoubleComplex* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIzamax(blasHandle, n, &dcomplexx, incx, &res);
  // CHECK-NEXT: blasStatus = hipblasIzamax(blasHandle, n, &dcomplexx, incx, &res);
  blasStatus = hipblasIzamax(blasHandle, n, &dcomplexx, incx, &res);
  blasStatus = hipblasIzamax(blasHandle, n, &dcomplexx, incx, &res);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasIsamin(hipblasHandle_t handle, int n, const float* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIsamin(hipblasHandle_t handle, int n, const float* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIsamin(blasHandle, n, &fx, incx, &res);
  // CHECK-NEXT: blasStatus = hipblasIsamin(blasHandle, n, &fx, incx, &res);
  blasStatus = hipblasIsamin(blasHandle, n, &fx, incx, &res);
  blasStatus = hipblasIsamin(blasHandle, n, &fx, incx, &res);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasIdamin(hipblasHandle_t handle, int n, const double* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIdamin(hipblasHandle_t handle, int n, const double* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIdamin(blasHandle, n, &dx, incx, &res);
  // CHECK-NEXT: blasStatus = hipblasIdamin(blasHandle, n, &dx, incx, &res);
  blasStatus = hipblasIdamin(blasHandle, n, &dx, incx, &res);
  blasStatus = hipblasIdamin(blasHandle, n, &dx, incx, &res);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasIcamin(hipblasHandle_t handle, int n, const hipComplex* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIcamin(hipblasHandle_t handle, int n, const hipblasComplex* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIcamin(blasHandle, n, &complexx, incx, &res);
  // CHECK-NEXT: blasStatus = hipblasIcamin(blasHandle, n, &complexx, incx, &res);
  blasStatus = hipblasIcamin(blasHandle, n, &complexx, incx, &res);
  blasStatus = hipblasIcamin(blasHandle, n, &complexx, incx, &res);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasIzamin(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, int* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasIzamin(hipblasHandle_t handle, int n, const hipblasDoubleComplex* x, int incx, int* result);
  // CHECK: blasStatus = hipblasIzamin(blasHandle, n, &dcomplexx, incx, &res);
  // CHECK-NEXT: blasStatus = hipblasIzamin(blasHandle, n, &dcomplexx, incx, &res);
  blasStatus = hipblasIzamin(blasHandle, n, &dcomplexx, incx, &res);
  blasStatus = hipblasIzamin(blasHandle, n, &dcomplexx, incx, &res);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSasum(hipblasHandle_t handle, int n, const float* x, int incx, float* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSasum(hipblasHandle_t handle, int n, const float* x, int incx, float* result);
  // CHECK: blasStatus = hipblasSasum(blasHandle, n, &fx, incx, &fresult);
  // CHECK-NEXT: blasStatus = hipblasSasum(blasHandle, n, &fx, incx, &fresult);
  blasStatus = hipblasSasum(blasHandle, n, &fx, incx, &fresult);
  blasStatus = hipblasSasum(blasHandle, n, &fx, incx, &fresult);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDasum(hipblasHandle_t handle, int n, const double* x, int incx, double* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDasum(hipblasHandle_t handle, int n, const double* x, int incx, double* result);
  // CHECK: blasStatus = hipblasDasum(blasHandle, n, &dx, incx, &dresult);
  // CHECK-NEXT: blasStatus = hipblasDasum(blasHandle, n, &dx, incx, &dresult);
  blasStatus = hipblasDasum(blasHandle, n, &dx, incx, &dresult);
  blasStatus = hipblasDasum(blasHandle, n, &dx, incx, &dresult);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasScasum(hipblasHandle_t handle, int n, const hipComplex* x, int incx, float* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasScasum(hipblasHandle_t handle, int n, const hipblasComplex* x, int incx, float* result);
  // CHECK: blasStatus = hipblasScasum(blasHandle, n, &complexx, incx, &fresult);
  // CHECK-NEXT: blasStatus = hipblasScasum(blasHandle, n, &complexx, incx, &fresult);
  blasStatus = hipblasScasum(blasHandle, n, &complexx, incx, &fresult);
  blasStatus = hipblasScasum(blasHandle, n, &complexx, incx, &fresult);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDzasum(hipblasHandle_t handle, int n, const hipDoubleComplex* x, int incx, double* result);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDzasum(hipblasHandle_t handle, int n, const hipblasDoubleComplex* x, int incx, double* result);
  // CHECK: blasStatus = hipblasDzasum(blasHandle, n, &dcomplexx, incx, &dresult);
  // CHECK-NEXT: blasStatus = hipblasDzasum(blasHandle, n, &dcomplexx, incx, &dresult);
  blasStatus = hipblasDzasum(blasHandle, n, &dcomplexx, incx, &dresult);
  blasStatus = hipblasDzasum(blasHandle, n, &dcomplexx, incx, &dresult);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSrot(hipblasHandle_t handle, int n, float* x, int incx, float* y, int incy, const float* c, const float* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSrot(hipblasHandle_t handle, int n, float* x, int incx, float* y, int incy, const float* c, const float* s);
  // CHECK: blasStatus = hipblasSrot(blasHandle, n, &fx, incx, &fy, incy, &fc, &fs);
  // CHECK-NEXT: blasStatus = hipblasSrot(blasHandle, n, &fx, incx, &fy, incy, &fc, &fs);
  blasStatus = hipblasSrot(blasHandle, n, &fx, incx, &fy, incy, &fc, &fs);
  blasStatus = hipblasSrot(blasHandle, n, &fx, incx, &fy, incy, &fc, &fs);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDrot(hipblasHandle_t handle, int n, double* x, int incx, double* y, int incy, const double* c, const double* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDrot(hipblasHandle_t handle, int n, double* x, int incx, double* y, int incy, const double* c, const double* s);
  // CHECK: blasStatus = hipblasDrot(blasHandle, n, &dx, incx, &dy, incy, &dc, &ds);
  // CHECK-NEXT: blasStatus = hipblasDrot(blasHandle, n, &dx, incx, &dy, incy, &dc, &ds);
  blasStatus = hipblasDrot(blasHandle, n, &dx, incx, &dy, incy, &dc, &ds);
  blasStatus = hipblasDrot(blasHandle, n, &dx, incx, &dy, incy, &dc, &ds);

  // CUDA: CUBLASAPI CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCrot(hipblasHandle_t handle, int n, hipComplex* x, int incx, hipComplex* y, int incy, const float* c, const hipComplex* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCrot(hipblasHandle_t handle, int n, hipblasComplex* x, int incx, hipblasComplex* y, int incy, const float* c, const hipblasComplex* s);
  // CHECK: blasStatus = hipblasCrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &complexs);
  // CHECK-NEXT: blasStatus = hipblasCrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &complexs);
  blasStatus = hipblasCrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &complexs);
  blasStatus = hipblasCrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &complexs);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCsrot(hipblasHandle_t handle, int n, hipComplex* x, int incx, hipComplex* y, int incy, const float* c, const float* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsrot(hipblasHandle_t handle, int n, hipblasComplex* x, int incx, hipblasComplex* y, int incy, const float* c, const float* s);
  // CHECK: blasStatus = hipblasCsrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &fs);
  // CHECK-NEXT: blasStatus = hipblasCsrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &fs);
  blasStatus = hipblasCsrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &fs);
  blasStatus = hipblasCsrot(blasHandle, n, &complexx, incx, &complexy, incy, &fc, &fs);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZrot(hipblasHandle_t handle, int n, hipDoubleComplex* x, int incx, hipDoubleComplex* y, int incy, const double* c, const hipDoubleComplex* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZrot(hipblasHandle_t handle, int n, hipblasDoubleComplex* x, int incx, hipblasDoubleComplex* y, int incy, const double* c, const hipblasDoubleComplex* s);
  // CHECK: blasStatus = hipblasZrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &dcomplexs);
  // CHECK-NEXT: blasStatus = hipblasZrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &dcomplexs);
  blasStatus = hipblasZrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &dcomplexs);
  blasStatus = hipblasZrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &dcomplexs);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZdrot(hipblasHandle_t handle, int n, hipDoubleComplex* x, int incx, hipDoubleComplex* y, int incy, const double* c, const double* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZdrot(hipblasHandle_t handle, int n, hipblasDoubleComplex* x, int incx, hipblasDoubleComplex* y, int incy, const double* c, const double* s);
  // CHECK: blasStatus = hipblasZdrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &ds);
  // CHECK-NEXT: blasStatus = hipblasZdrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &ds);
  blasStatus = hipblasZdrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &ds);
  blasStatus = hipblasZdrot(blasHandle, n, &dcomplexx, incx, &dcomplexy, incy, &dc, &ds);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSrotg(hipblasHandle_t handle, float* a, float* b, float* c, float* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSrotg(hipblasHandle_t handle, float* a, float* b, float* c, float* s);
  // CHECK: blasStatus = hipblasSrotg(blasHandle, &fa, &fb, &fc, &fs);
  // CHECK-NEXT: blasStatus = hipblasSrotg(blasHandle, &fa, &fb, &fc, &fs);
  blasStatus = hipblasSrotg(blasHandle, &fa, &fb, &fc, &fs);
  blasStatus = hipblasSrotg(blasHandle, &fa, &fb, &fc, &fs);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDrotg(hipblasHandle_t handle, double* a, double* b, double* c, double* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDrotg(hipblasHandle_t handle, double* a, double* b, double* c, double* s);
  // CHECK: blasStatus = hipblasDrotg(blasHandle, &da, &db, &dc, &ds);
  // CHECK-NEXT: blasStatus = hipblasDrotg(blasHandle, &da, &db, &dc, &ds);
  blasStatus = hipblasDrotg(blasHandle, &da, &db, &dc, &ds);
  blasStatus = hipblasDrotg(blasHandle, &da, &db, &dc, &ds);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCrotg(hipblasHandle_t handle, hipComplex* a, hipComplex* b, float* c, hipComplex* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCrotg(hipblasHandle_t handle, hipblasComplex* a, hipblasComplex* b, float* c, hipblasComplex* s);
  // CHECK: blasStatus = hipblasCrotg(blasHandle, &complexa, &complexb, &fc, &complexs);
  // CHECK-NEXT: blasStatus = hipblasCrotg(blasHandle, &complexa, &complexb, &fc, &complexs);
  blasStatus = hipblasCrotg(blasHandle, &complexa, &complexb, &fc, &complexs);
  blasStatus = hipblasCrotg(blasHandle, &complexa, &complexb, &fc, &complexs);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZrotg(hipblasHandle_t handle, hipDoubleComplex* a, hipDoubleComplex* b, double* c, hipDoubleComplex* s);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZrotg(hipblasHandle_t handle, hipblasDoubleComplex* a, hipblasDoubleComplex* b, double* c, hipblasDoubleComplex* s);
  // CHECK: blasStatus = hipblasZrotg(blasHandle, &dcomplexa, &dcomplexb, &dc, &dcomplexs);
  // CHECK-NEXT: blasStatus = hipblasZrotg(blasHandle, &dcomplexa, &dcomplexb, &dc, &dcomplexs);
  blasStatus = hipblasZrotg(blasHandle, &dcomplexa, &dcomplexb, &dc, &dcomplexs);
  blasStatus = hipblasZrotg(blasHandle, &dcomplexa, &dcomplexb, &dc, &dcomplexs);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSrotm(hipblasHandle_t handle, int n, float* x, int incx, float* y, int incy, const float* param);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSrotm(hipblasHandle_t handle, int n, float* x, int incx, float* y, int incy, const float* param);
  // CHECK: blasStatus = hipblasSrotm(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  // CHECK-NEXT: blasStatus = hipblasSrotm(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  blasStatus = hipblasSrotm(blasHandle, n, &fx, incx, &fy, incy, &fresult);
  blasStatus = hipblasSrotm(blasHandle, n, &fx, incx, &fy, incy, &fresult);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDrotm(hipblasHandle_t handle, int n, double* x, int incx, double* y, int incy, const double* param);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDrotm(hipblasHandle_t handle, int n, double* x, int incx, double* y, int incy, const double* param);
  // CHECK: blasStatus = hipblasDrotm(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  // CHECK-NEXT: blasStatus = hipblasDrotm(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  blasStatus = hipblasDrotm(blasHandle, n, &dx, incx, &dy, incy, &dresult);
  blasStatus = hipblasDrotm(blasHandle, n, &dx, incx, &dy, incy, &dresult);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSrotmg(hipblasHandle_t handle, float* d1, float* d2, float* x1, const float* y1, float* param);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSrotmg(hipblasHandle_t handle, float* d1, float* d2, float* x1, const float* y1, float* param);
  // CHECK: blasStatus = hipblasSrotmg(blasHandle, &fd1, &fd2, &fx1, &fy1, &fresult);
  // CHECK-NEXT: blasStatus = hipblasSrotmg(blasHandle, &fd1, &fd2, &fx1, &fy1, &fresult);
  blasStatus = hipblasSrotmg(blasHandle, &fd1, &fd2, &fx1, &fy1, &fresult);
  blasStatus = hipblasSrotmg(blasHandle, &fd1, &fd2, &fx1, &fy1, &fresult);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDrotmg(hipblasHandle_t handle, double* d1, double* d2, double* x1, const double* y1, double* param);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDrotmg(hipblasHandle_t handle, double* d1, double* d2, double* x1, const double* y1, double* param);
  // CHECK: blasStatus = hipblasDrotmg(blasHandle, &dd1, &dd2, &dx1, &dy1, &dresult);
  // CHECK-NEXT: blasStatus = hipblasDrotmg(blasHandle, &dd1, &dd2, &dx1, &dy1, &dresult);
  blasStatus = hipblasDrotmg(blasHandle, &dd1, &dd2, &dx1, &dy1, &dresult);
  blasStatus = hipblasDrotmg(blasHandle, &dd1, &dd2, &dx1, &dy1, &dresult);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSgemv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, const float* alpha, const float* A, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgemv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, const float* alpha, const float* AP, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // CHECK: blasStatus = hipblasSgemv(blasHandle, blasOperation, m, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  // CHECK-NEXT: blasStatus = hipblasSgemv(blasHandle, blasOperation, m, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSgemv(blasHandle, blasOperation, m, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSgemv(blasHandle, blasOperation, m, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDgemv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, const double* alpha, const double* A, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgemv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, const double* alpha, const double* AP, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // CHECK: blasStatus = hipblasDgemv(blasHandle, blasOperation, m, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  // CHECK-NEXT: blasStatus = hipblasDgemv(blasHandle, blasOperation, m, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDgemv(blasHandle, blasOperation, m, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDgemv(blasHandle, blasOperation, m, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCgemv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, const hipComplex* alpha, const hipComplex* A, int lda, const hipComplex* x, int incx, const hipComplex* beta, hipComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgemv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, const hipblasComplex* alpha, const hipblasComplex* AP, int lda, const hipblasComplex* x, int incx, const hipblasComplex* beta, hipblasComplex* y, int incy);
  // CHECK: blasStatus = hipblasCgemv(blasHandle, blasOperation, m, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  // CHECK-NEXT: blasStatus = hipblasCgemv(blasHandle, blasOperation, m, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCgemv(blasHandle, blasOperation, m, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCgemv(blasHandle, blasOperation, m, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZgemv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* A, int lda, const hipDoubleComplex* x, int incx, const hipDoubleComplex* beta, hipDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgemv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* AP, int lda, const hipblasDoubleComplex* x, int incx, const hipblasDoubleComplex* beta, hipblasDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZgemv(blasHandle, blasOperation, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = hipblasZgemv(blasHandle, blasOperation, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZgemv(blasHandle, blasOperation, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZgemv(blasHandle, blasOperation, m, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSgbmv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int kl, int ku, const float* alpha, const float* A, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSgbmv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int kl, int ku, const float* alpha, const float* AP, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // CHECK: blasStatus = hipblasSgbmv(blasHandle, blasOperation, m, n, kl, ku, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  // CHECK-NEXT: blasStatus = hipblasSgbmv(blasHandle, blasOperation, m, n, kl, ku, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSgbmv(blasHandle, blasOperation, m, n, kl, ku, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSgbmv(blasHandle, blasOperation, m, n, kl, ku, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDgbmv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int kl, int ku, const double* alpha, const double* A, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDgbmv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int kl, int ku, const double* alpha, const double* AP, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // CHECK: blasStatus = hipblasDgbmv(blasHandle, blasOperation, m, n, kl, ku, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  // CHECK-NEXT: blasStatus = hipblasDgbmv(blasHandle, blasOperation, m, n, kl, ku, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDgbmv(blasHandle, blasOperation, m, n, kl, ku, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDgbmv(blasHandle, blasOperation, m, n, kl, ku, &da, &dA, lda, &dx, incx, &db, &dy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCgbmv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int kl, int ku, const hipComplex* alpha, const hipComplex* A, int lda, const hipComplex* x, int incx, const hipComplex* beta, hipComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCgbmv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int kl, int ku, const hipblasComplex* alpha, const hipblasComplex* AP, int lda, const hipblasComplex* x, int incx, const hipblasComplex* beta, hipblasComplex* y, int incy);
  // CHECK: blasStatus = hipblasCgbmv(blasHandle, blasOperation, m, n, kl, ku, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  // CHECK-NEXT: blasStatus = hipblasCgbmv(blasHandle, blasOperation, m, n, kl, ku, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCgbmv(blasHandle, blasOperation, m, n, kl, ku, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCgbmv(blasHandle, blasOperation, m, n, kl, ku, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZgbmv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int kl, int ku, const hipDoubleComplex* alpha, const hipDoubleComplex* A, int lda, const hipDoubleComplex* x, int incx, const hipDoubleComplex* beta, hipDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZgbmv(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int kl, int ku, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* AP, int lda, const hipblasDoubleComplex* x, int incx, const hipblasDoubleComplex* beta, hipblasDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZgbmv(blasHandle, blasOperation, m, n, kl, ku, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = hipblasZgbmv(blasHandle, blasOperation, m, n, kl, ku, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZgbmv(blasHandle, blasOperation, m, n, kl, ku, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZgbmv(blasHandle, blasOperation, m, n, kl, ku, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasStrmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n, const float* A, int lda, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStrmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const float* AP, int lda, float* x, int incx);
  // CHECK: blasStatus = hipblasStrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);
  // CHECK-NEXT: blasStatus = hipblasStrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);
  blasStatus = hipblasStrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);
  blasStatus = hipblasStrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDtrmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n, const double* A, int lda, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtrmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const double* AP, int lda, double* x, int incx);
  // CHECK: blasStatus = hipblasDtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);
  // CHECK-NEXT: blasStatus = hipblasDtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);
  blasStatus = hipblasDtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);
  blasStatus = hipblasDtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCtrmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n, const hipComplex* A, int lda, hipComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtrmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipblasComplex* AP, int lda, hipblasComplex* x, int incx);
  // CHECK: blasStatus = hipblasCtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);
  // CHECK-NEXT: blasStatus = hipblasCtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZtrmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n, const hipDoubleComplex* A, int lda, hipDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtrmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipblasDoubleComplex* AP, int lda, hipblasDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = hipblasZtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtrmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasStbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n, int k, const float* A, int lda, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int k, const float* AP, int lda, float* x, int incx);
  // CHECK: blasStatus = hipblasStbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);
  // CHECK-NEXT: blasStatus = hipblasStbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);
  blasStatus = hipblasStbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);
  blasStatus = hipblasStbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDtbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n, int k, const double* A, int lda, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int k, const double* AP, int lda, double* x, int incx);
  // CHECK: blasStatus = hipblasDtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);
  // CHECK-NEXT: blasStatus = hipblasDtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);
  blasStatus = hipblasDtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);
  blasStatus = hipblasDtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCtbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n, int k, const hipComplex* A, int lda, hipComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int k, const hipblasComplex* AP, int lda, hipblasComplex* x, int incx);
  // CHECK: blasStatus = hipblasCtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);
  // CHECK-NEXT: blasStatus = hipblasCtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZtbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n, int k, const hipDoubleComplex* A, int lda, hipDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, int k, const hipblasDoubleComplex* AP, int lda, hipblasDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = hipblasZtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtbmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasStpmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n, const float* AP, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStpmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const float* AP, float* x, int incx);
  // CHECK: blasStatus = hipblasStpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);
  // CHECK-NEXT: blasStatus = hipblasStpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);
  blasStatus = hipblasStpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);
  blasStatus = hipblasStpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDtpmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n, const double* AP, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtpmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const double* AP, double* x, int incx);
  // CHECK: blasStatus = hipblasDtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);
  // CHECK-NEXT: blasStatus = hipblasDtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);
  blasStatus = hipblasDtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);
  blasStatus = hipblasDtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCtpmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n, const hipComplex* AP, hipComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtpmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipblasComplex* AP, hipblasComplex* x, int incx);
  // CHECK: blasStatus = hipblasCtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);
  // CHECK-NEXT: blasStatus = hipblasCtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);
  blasStatus = hipblasCtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);
  blasStatus = hipblasCtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZtpmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n, const hipDoubleComplex* AP, hipDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtpmv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipblasDoubleComplex* AP, hipblasDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = hipblasZtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);
  blasStatus = hipblasZtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);
  blasStatus = hipblasZtpmv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasStrsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n, const float* A, int lda, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStrsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const float* AP, int lda, float* x, int incx);
  // CHECK: blasStatus = hipblasStrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);
  // CHECK-NEXT: blasStatus = hipblasStrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);
  blasStatus = hipblasStrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);
  blasStatus = hipblasStrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, lda, &fx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDtrsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n, const double* A, int lda, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtrsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const double* AP, int lda, double* x, int incx);
  // CHECK: blasStatus = hipblasDtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);
  // CHECK-NEXT: blasStatus = hipblasDtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);
  blasStatus = hipblasDtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);
  blasStatus = hipblasDtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, lda, &dx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCtrsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n, const hipComplex* A, int lda, hipComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtrsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipblasComplex* AP, int lda, hipblasComplex* x, int incx);
  // CHECK: blasStatus = hipblasCtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);
  // CHECK-NEXT: blasStatus = hipblasCtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, lda, &complexx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZtrsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n, const hipDoubleComplex* A, int lda, hipDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtrsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipblasDoubleComplex* AP, int lda, hipblasDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = hipblasZtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtrsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, lda, &dcomplexx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasStpsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n, const float* AP, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStpsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const float* AP, float* x, int incx);
  // CHECK: blasStatus = hipblasStpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);
  // CHECK-NEXT: blasStatus = hipblasStpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);
  blasStatus = hipblasStpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);
  blasStatus = hipblasStpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &fA, &fx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDtpsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n, const double* AP, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtpsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const double* AP, double* x, int incx);
  // CHECK: blasStatus = hipblasDtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);
  // CHECK-NEXT: blasStatus = hipblasDtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);
  blasStatus = hipblasDtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);
  blasStatus = hipblasDtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dA, &dx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCtpsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n, const hipComplex* AP, hipComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtpsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipblasComplex* AP, hipblasComplex* x, int incx);
  // CHECK: blasStatus = hipblasCtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);
  // CHECK-NEXT: blasStatus = hipblasCtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);
  blasStatus = hipblasCtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);
  blasStatus = hipblasCtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &complexA, &complexx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZtpsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n, const hipDoubleComplex* AP, hipDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtpsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int m, const hipblasDoubleComplex* AP, hipblasDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = hipblasZtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);
  blasStatus = hipblasZtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);
  blasStatus = hipblasZtpsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, &dcomplexA, &dcomplexx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasStbsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n, int k, const float* A, int lda, float* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasStbsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int n, int k, const float* AP, int lda, float* x, int incx);
  // CHECK: blasStatus = hipblasStbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);
  // CHECK-NEXT: blasStatus = hipblasStbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);
  blasStatus = hipblasStbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);
  blasStatus = hipblasStbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &fA, lda, &fx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDtbsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n, int k, const double* A, int lda, double* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDtbsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int n, int k, const double* AP, int lda, double* x, int incx);
  // CHECK: blasStatus = hipblasDtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);
  // CHECK-NEXT: blasStatus = hipblasDtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);
  blasStatus = hipblasDtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);
  blasStatus = hipblasDtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dA, lda, &dx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCtbsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n, int k, const hipComplex* A, int lda, hipComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCtbsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int n, int k, const hipblasComplex* AP, int lda, hipblasComplex* x, int incx);
  // CHECK: blasStatus = hipblasCtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);
  // CHECK-NEXT: blasStatus = hipblasCtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);
  blasStatus = hipblasCtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &complexA, lda, &complexx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZtbsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t trans, hipblasDiagType_t diag, int n, int k, const hipDoubleComplex* A, int lda, hipDoubleComplex* x, int incx);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZtbsv(hipblasHandle_t handle, hipblasFillMode_t uplo, hipblasOperation_t transA, hipblasDiagType_t diag, int n, int k, const hipblasDoubleComplex* AP, int lda, hipblasDoubleComplex* x, int incx);
  // CHECK: blasStatus = hipblasZtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);
  // CHECK-NEXT: blasStatus = hipblasZtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);
  blasStatus = hipblasZtbsv(blasHandle, blasFillMode, blasOperation, blasDiagType, n, k, &dcomplexA, lda, &dcomplexx, incx);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSsymv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const float* A, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsymv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const float* AP, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // CHECK: blasStatus = hipblasSsymv(blasHandle, blasFillMode, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  // CHECK-NEXT: blasStatus = hipblasSsymv(blasHandle, blasFillMode, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSsymv(blasHandle, blasFillMode, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSsymv(blasHandle, blasFillMode, n, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDsymv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const double* alpha, const double* A, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsymv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const double* alpha, const double* AP, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // CHECK: blasStatus = hipblasDsymv(blasHandle, blasFillMode, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  // CHECK-NEXT: blasStatus = hipblasDsymv(blasHandle, blasFillMode, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDsymv(blasHandle, blasFillMode, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDsymv(blasHandle, blasFillMode, n, &da, &dA, lda, &dx, incx, &db, &dy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCsymv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipComplex* alpha, const hipComplex* A, int lda, const hipComplex* x, int incx, const hipComplex* beta, hipComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasCsymv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipblasComplex* alpha, const hipblasComplex* AP, int lda, const hipblasComplex* x, int incx, const hipblasComplex* beta, hipblasComplex* y, int incy);
  // CHECK: blasStatus = hipblasCsymv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  // CHECK-NEXT: blasStatus = hipblasCsymv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCsymv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasCsymv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZsymv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* A, int lda, const hipDoubleComplex* x, int incx, const hipDoubleComplex* beta, hipDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZsymv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* AP, int lda, const hipblasDoubleComplex* x, int incx, const hipblasDoubleComplex* beta, hipblasDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZsymv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = hipblasZsymv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZsymv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZsymv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasChemv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipComplex* alpha, const hipComplex* A, int lda, const hipComplex* x, int incx, const hipComplex* beta, hipComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasChemv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipblasComplex* alpha, const hipblasComplex* AP, int lda, const hipblasComplex* x, int incx, const hipblasComplex* beta, hipblasComplex* y, int incy);
  // CHECK: blasStatus = hipblasChemv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  // CHECK-NEXT: blasStatus = hipblasChemv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChemv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChemv(blasHandle, blasFillMode, n, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZhemv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* A, int lda, const hipDoubleComplex* x, int incx, const hipDoubleComplex* beta, hipDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZhemv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* AP, int lda, const hipblasDoubleComplex* x, int incx, const hipblasDoubleComplex* beta, hipblasDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZhemv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = hipblasZhemv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhemv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhemv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSsbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, int k, const float* alpha, const float* A, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSsbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, int k, const float* alpha, const float* AP, int lda, const float* x, int incx, const float* beta, float* y, int incy);
  // CHECK: blasStatus = hipblasSsbmv(blasHandle, blasFillMode, n, k, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  // CHECK-NEXT: blasStatus = hipblasSsbmv(blasHandle, blasFillMode, n, k, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSsbmv(blasHandle, blasFillMode, n, k, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSsbmv(blasHandle, blasFillMode, n, k, &fa, &fA, lda, &fx, incx, &fb, &fy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDsbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, int k, const double* alpha, const double* A, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDsbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, int k, const double* alpha, const double* AP, int lda, const double* x, int incx, const double* beta, double* y, int incy);
  // CHECK: blasStatus = hipblasDsbmv(blasHandle, blasFillMode, n, k, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  // CHECK-NEXT: blasStatus = hipblasDsbmv(blasHandle, blasFillMode, n, k, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDsbmv(blasHandle, blasFillMode, n, k, &da, &dA, lda, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDsbmv(blasHandle, blasFillMode, n, k, &da, &dA, lda, &dx, incx, &db, &dy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasChbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, int k, const hipComplex* alpha, const hipComplex* A, int lda, const hipComplex* x, int incx, const hipComplex* beta, hipComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasChbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, int k, const hipblasComplex* alpha, const hipblasComplex* AP, int lda, const hipblasComplex* x, int incx, const hipblasComplex* beta, hipblasComplex* y, int incy);
  // CHECK: blasStatus = hipblasChbmv(blasHandle, blasFillMode, n, k, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  // CHECK-NEXT: blasStatus = hipblasChbmv(blasHandle, blasFillMode, n, k, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChbmv(blasHandle, blasFillMode, n, k, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChbmv(blasHandle, blasFillMode, n, k, &complexa, &complexA, lda, &complexx, incx, &complexb, &complexy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZhbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, int k, const hipDoubleComplex* alpha, const hipDoubleComplex* A, int lda, const hipDoubleComplex* x, int incx, const hipDoubleComplex* beta, hipDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZhbmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, int k, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* AP, int lda, const hipblasDoubleComplex* x, int incx, const hipblasDoubleComplex* beta, hipblasDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZhbmv(blasHandle, blasFillMode, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = hipblasZhbmv(blasHandle, blasFillMode, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhbmv(blasHandle, blasFillMode, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhbmv(blasHandle, blasFillMode, n, k, &dcomplexa, &dcomplexA, lda, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasSspmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const float* AP, const float* x, int incx, const float* beta, float* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasSspmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const float* alpha, const float* AP, const float* x, int incx, const float* beta, float* y, int incy);
  // CHECK: blasStatus = hipblasSspmv(blasHandle, blasFillMode, n, &fa, &fA, &fx, incx, &fb, &fy, incy);
  // CHECK-NEXT: blasStatus = hipblasSspmv(blasHandle, blasFillMode, n, &fa, &fA, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSspmv(blasHandle, blasFillMode, n, &fa, &fA, &fx, incx, &fb, &fy, incy);
  blasStatus = hipblasSspmv(blasHandle, blasFillMode, n, &fa, &fA, &fx, incx, &fb, &fy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDspmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const double* alpha, const double* AP, const double* x, int incx, const double* beta, double* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasDspmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const double* alpha, const double* AP, const double* x, int incx, const double* beta, double* y, int incy);
  // CHECK: blasStatus = hipblasDspmv(blasHandle, blasFillMode, n, &da, &dA, &dx, incx, &db, &dy, incy);
  // CHECK-NEXT: blasStatus = hipblasDspmv(blasHandle, blasFillMode, n, &da, &dA, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDspmv(blasHandle, blasFillMode, n, &da, &dA, &dx, incx, &db, &dy, incy);
  blasStatus = hipblasDspmv(blasHandle, blasFillMode, n, &da, &dA, &dx, incx, &db, &dy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasChpmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipComplex* alpha, const hipComplex* AP, const hipComplex* x, int incx, const hipComplex* beta, hipComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasChpmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipblasComplex* alpha, const hipblasComplex* AP, const hipblasComplex* x, int incx, const hipblasComplex* beta, hipblasComplex* y, int incy);
  // CHECK: blasStatus = hipblasChpmv(blasHandle, blasFillMode, n, &complexa, &complexA, &complexx, incx, &complexb, &complexy, incy);
  // CHECK-NEXT: blasStatus = hipblasChpmv(blasHandle, blasFillMode, n, &complexa, &complexA, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChpmv(blasHandle, blasFillMode, n, &complexa, &complexA, &complexx, incx, &complexb, &complexy, incy);
  blasStatus = hipblasChpmv(blasHandle, blasFillMode, n, &complexa, &complexA, &complexx, incx, &complexb, &complexy, incy);

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZhpmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipDoubleComplex* alpha, const hipDoubleComplex* AP, const hipDoubleComplex* x, int incx, const hipDoubleComplex* beta, hipDoubleComplex* y, int incy);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasZhpmv(hipblasHandle_t handle, hipblasFillMode_t uplo, int n, const hipblasDoubleComplex* alpha, const hipblasDoubleComplex* AP, const hipblasDoubleComplex* x, int incx, const hipblasDoubleComplex* beta, hipblasDoubleComplex* y, int incy);
  // CHECK: blasStatus = hipblasZhpmv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  // CHECK-NEXT: blasStatus = hipblasZhpmv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhpmv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);
  blasStatus = hipblasZhpmv(blasHandle, blasFillMode, n, &dcomplexa, &dcomplexA, &dcomplexx, incx, &dcomplexb, &dcomplexy, incy);

#if CUDA_VERSION >= 8000
  // CHECK: hipblasDatatype_t DataType;
  // CHECK-NEXT: hipblasDatatype_t DataType_t;
  // CHECK-NEXT: hipblasDatatype_t blasDataType;
  // CHECK-NEXT: hipblasDatatype_t R_16F = HIPBLAS_R_16F;
  // CHECK-NEXT: hipblasDatatype_t C_16F = HIPBLAS_C_16F;
  // CHECK-NEXT: hipblasDatatype_t R_32F = HIPBLAS_R_32F;
  // CHECK-NEXT: hipblasDatatype_t C_32F = HIPBLAS_C_32F;
  // CHECK-NEXT: hipblasDatatype_t R_64F = HIPBLAS_R_64F;
  // CHECK-NEXT: hipblasDatatype_t C_64F = HIPBLAS_C_64F;
  // CHECK-NEXT: hipblasDatatype_t R_8I = HIPBLAS_R_8I;
  // CHECK-NEXT: hipblasDatatype_t C_8I = HIPBLAS_C_8I;
  // CHECK-NEXT: hipblasDatatype_t R_8U = HIPBLAS_R_8U;
  // CHECK-NEXT: hipblasDatatype_t C_8U = HIPBLAS_C_8U;
  // CHECK-NEXT: hipblasDatatype_t R_32I = HIPBLAS_R_32I;
  // CHECK-NEXT: hipblasDatatype_t C_32I = HIPBLAS_C_32I;
  // CHECK-NEXT: hipblasDatatype_t R_32U = HIPBLAS_R_32U;
  // CHECK-NEXT: hipblasDatatype_t C_32U = HIPBLAS_C_32U;
  hipDataType DataType;
  hipDataType DataType_t;
  hipDataType blasDataType;
  hipDataType R_16F = HIP_R_16F;
  hipDataType C_16F = HIP_C_16F;
  hipDataType R_32F = HIP_R_32F;
  hipDataType C_32F = HIP_C_32F;
  hipDataType R_64F = HIP_R_64F;
  hipDataType C_64F = HIP_C_64F;
  hipDataType R_8I = HIP_R_8I;
  hipDataType C_8I = HIP_C_8I;
  hipDataType R_8U = HIP_R_8U;
  hipDataType C_8U = HIP_C_8U;
  hipDataType R_32I = HIP_R_32I;
  hipDataType C_32I = HIP_C_32I;
  hipDataType R_32U = HIP_R_32U;
  hipDataType C_32U = HIP_C_32U;

  // CHECK: hipblasDatatype_t DataType_2, DataType_3;
  hipDataType DataType_2, DataType_3;

  // CHECK: hipblasGemmAlgo_t blasGemmAlgo;
  // CHECK-NEXT: hipblasGemmAlgo_t BLAS_GEMM_DFALT = HIPBLAS_GEMM_DEFAULT;
  hipblasGemmAlgo_t blasGemmAlgo;
  hipblasGemmAlgo_t BLAS_GEMM_DFALT = HIPBLAS_GEMM_DEFAULT;

  // CUDA: CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasNrm2Ex(hipblasHandle_t handle, int n, const void* x, hipDataType xType, int incx, void* result, hipDataType resultType, hipDataType executionType);
  // HIP: HIPBLAS_EXPORT hipblasStatus_t hipblasNrm2Ex(hipblasHandle_t handle, int n, const void* x, hipblasDatatype_t xType, int incx, void* result, hipblasDatatype_t resultType, hipblasDatatype_t executionType);
  // CHECK: blasStatus = hipblasNrm2Ex(blasHandle, n, image, DataType, incx, image_2, DataType_2, DataType_3);
  blasStatus = hipblasNrm2Ex(blasHandle, n, image, DataType, incx, image_2, DataType_2, DataType_3);
#endif

#if CUDA_VERSION >= 9000
  // CHECK: hipblasGemmAlgo_t BLAS_GEMM_DEFAULT = HIPBLAS_GEMM_DEFAULT;
  hipblasGemmAlgo_t BLAS_GEMM_DEFAULT = HIPBLAS_GEMM_DEFAULT;
#endif

#if CUDA_VERSION >= 10010
  // CHECK: hipblasOperation_t BLAS_OP_HERMITAN = HIPBLAS_OP_C;
  hipblasOperation_t BLAS_OP_HERMITAN = HIPBLAS_OP_C;

  // CHECK: hipblasFillMode_t BLAS_FILL_MODE_FULL = HIPBLAS_FILL_MODE_FULL;
  hipblasFillMode_t BLAS_FILL_MODE_FULL = HIPBLAS_FILL_MODE_FULL;
#endif

#if CUDA_VERSION >= 11000
  // CHECK: hipblasDatatype_t R_16BF = HIPBLAS_R_16B;
  // CHECK-NEXT: hipblasDatatype_t C_16BF = HIPBLAS_C_16B;
  hipDataType R_16BF = HIP_R_16BF;
  hipDataType C_16BF = HIP_C_16BF;
#endif

  return 0;
}
