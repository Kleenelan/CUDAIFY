// RUN: %run_test hipify "%s" "%t" %hipify_args 2 --skip-excluded-preprocessor-conditional-blocks --experimental %clang_args -D__CUDA_API_VERSION_INTERNAL

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// CHECK: #include "hipblas.h"
#include "hipblas.h"

int main() {
  printf("14. cuBLAS API to hipBLAS API synthetic test\n");

  // CHECK: hipblasOperation_t blasOperation;
  // CHECK-NEXT: hipblasOperation_t BLAS_OP_N = HIPBLAS_OP_N;
  // CHECK-NEXT: hipblasOperation_t BLAS_OP_T = HIPBLAS_OP_T;
  // CHECK-NEXT: hipblasOperation_t BLAS_OP_C = HIPBLAS_OP_C;
  hipblasOperation_t blasOperation;
  hipblasOperation_t BLAS_OP_N = HIPBLAS_OP_N;
  hipblasOperation_t BLAS_OP_T = HIPBLAS_OP_T;
  hipblasOperation_t BLAS_OP_C = HIPBLAS_OP_C;

#if CUDA_VERSION >= 10010
  // CHECK: hipblasOperation_t BLAS_OP_HERMITAN = HIPBLAS_OP_C;
  hipblasOperation_t BLAS_OP_HERMITAN = HIPBLAS_OP_C;
#endif

  // CHECK: hipblasStatus_t blasStatus;
  // CHECK-NEXT: hipblasStatus_t blasStatus_t;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_SUCCESS = HIPBLAS_STATUS_SUCCESS;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_NOT_INITIALIZED = HIPBLAS_STATUS_NOT_INITIALIZED;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_ALLOC_FAILED = HIPBLAS_STATUS_ALLOC_FAILED;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_INVALID_VALUE = HIPBLAS_STATUS_INVALID_VALUE;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_MAPPING_ERROR = HIPBLAS_STATUS_MAPPING_ERROR;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_EXECUTION_FAILED = HIPBLAS_STATUS_EXECUTION_FAILED;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_INTERNAL_ERROR = HIPBLAS_STATUS_INTERNAL_ERROR;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_NOT_SUPPORTED = HIPBLAS_STATUS_NOT_SUPPORTED;
  // CHECK-NEXT: hipblasStatus_t BLAS_STATUS_ARCH_MISMATCH = HIPBLAS_STATUS_ARCH_MISMATCH;
  hipblasStatus_t blasStatus;
  hipblasStatus_t blasStatus_t;
  hipblasStatus_t BLAS_STATUS_SUCCESS = HIPBLAS_STATUS_SUCCESS;
  hipblasStatus_t BLAS_STATUS_NOT_INITIALIZED = HIPBLAS_STATUS_NOT_INITIALIZED;
  hipblasStatus_t BLAS_STATUS_ALLOC_FAILED = HIPBLAS_STATUS_ALLOC_FAILED;
  hipblasStatus_t BLAS_STATUS_INVALID_VALUE = HIPBLAS_STATUS_INVALID_VALUE;
  hipblasStatus_t BLAS_STATUS_MAPPING_ERROR = HIPBLAS_STATUS_MAPPING_ERROR;
  hipblasStatus_t BLAS_STATUS_EXECUTION_FAILED = HIPBLAS_STATUS_EXECUTION_FAILED;
  hipblasStatus_t BLAS_STATUS_INTERNAL_ERROR = HIPBLAS_STATUS_INTERNAL_ERROR;
  hipblasStatus_t BLAS_STATUS_NOT_SUPPORTED = HIPBLAS_STATUS_NOT_SUPPORTED;
  hipblasStatus_t BLAS_STATUS_ARCH_MISMATCH = HIPBLAS_STATUS_ARCH_MISMATCH;

  // CHECK: hipblasFillMode_t blasFillMode;
  // CHECK-NEXT: hipblasFillMode_t BLAS_FILL_MODE_LOWER = HIPBLAS_FILL_MODE_LOWER;
  // CHECK-NEXT: hipblasFillMode_t BLAS_FILL_MODE_UPPER = HIPBLAS_FILL_MODE_UPPER;
  hipblasFillMode_t blasFillMode;
  hipblasFillMode_t BLAS_FILL_MODE_LOWER = HIPBLAS_FILL_MODE_LOWER;
  hipblasFillMode_t BLAS_FILL_MODE_UPPER = HIPBLAS_FILL_MODE_UPPER;

#if CUDA_VERSION >= 10010
  // CHECK: hipblasFillMode_t BLAS_FILL_MODE_FULL = HIPBLAS_FILL_MODE_FULL;
  hipblasFillMode_t BLAS_FILL_MODE_FULL = HIPBLAS_FILL_MODE_FULL;
#endif

  // CHECK: hipblasDiagType_t blasDiagType;
  // CHECK-NEXT: hipblasDiagType_t BLAS_DIAG_NON_UNIT = HIPBLAS_DIAG_NON_UNIT;
  // CHECK-NEXT: hipblasDiagType_t BLAS_DIAG_UNIT = HIPBLAS_DIAG_UNIT;
  hipblasDiagType_t blasDiagType;
  hipblasDiagType_t BLAS_DIAG_NON_UNIT = HIPBLAS_DIAG_NON_UNIT;
  hipblasDiagType_t BLAS_DIAG_UNIT = HIPBLAS_DIAG_UNIT;

  // CHECK: hipblasSideMode_t blasSideMode;
  // CHECK-NEXT: hipblasSideMode_t BLAS_SIDE_LEFT = HIPBLAS_SIDE_LEFT;
  // CHECK-NEXT: hipblasSideMode_t BLAS_SIDE_RIGHT = HIPBLAS_SIDE_RIGHT;
  hipblasSideMode_t blasSideMode;
  hipblasSideMode_t BLAS_SIDE_LEFT = HIPBLAS_SIDE_LEFT;
  hipblasSideMode_t BLAS_SIDE_RIGHT = HIPBLAS_SIDE_RIGHT;

  // CHECK: hipblasPointerMode_t blasPointerMode;
  // CHECK-NEXT: hipblasPointerMode_t BLAS_POINTER_MODE_HOST = HIPBLAS_POINTER_MODE_HOST;
  // CHECK-NEXT: hipblasPointerMode_t BLAS_POINTER_MODE_DEVICE = HIPBLAS_POINTER_MODE_DEVICE;
  hipblasPointerMode_t blasPointerMode;
  hipblasPointerMode_t BLAS_POINTER_MODE_HOST = HIPBLAS_POINTER_MODE_HOST;
  hipblasPointerMode_t BLAS_POINTER_MODE_DEVICE = HIPBLAS_POINTER_MODE_DEVICE;

  // CHECK: hipblasAtomicsMode_t blasAtomicsMode;
  // CHECK-NEXT: hipblasAtomicsMode_t BLAS_ATOMICS_NOT_ALLOWED = HIPBLAS_ATOMICS_NOT_ALLOWED;
  // CHECK-NEXT: hipblasAtomicsMode_t BLAS_ATOMICS_ALLOWED = HIPBLAS_ATOMICS_ALLOWED;
  hipblasAtomicsMode_t blasAtomicsMode;
  hipblasAtomicsMode_t BLAS_ATOMICS_NOT_ALLOWED = HIPBLAS_ATOMICS_NOT_ALLOWED;
  hipblasAtomicsMode_t BLAS_ATOMICS_ALLOWED = HIPBLAS_ATOMICS_ALLOWED;

#if CUDA_VERSION >= 8000
  // CHECK: hipblasDatatype_t DataType;
  // CHECK-NEXT: hipblasDatatype_t DataType_t;
  // CHECK-NEXT: hipblasDatatype_t blasDataType;
  // CHECK-NEXT: hipblasDatatype_t R_16F = HIPBLAS_R_16F;
  // CHECK-NEXT: hipblasDatatype_t C_16F = HIPBLAS_C_16F;
  // CHECK-NEXT: hipblasDatatype_t R_32F = HIPBLAS_R_32F;
  // CHECK-NEXT: hipblasDatatype_t C_32F = HIPBLAS_C_32F;
  // CHECK-NEXT: hipblasDatatype_t R_64F = HIPBLAS_R_64F;
  // CHECK-NEXT: hipblasDatatype_t C_64F = HIPBLAS_C_64F;
  // CHECK-NEXT: hipblasDatatype_t R_8I = HIPBLAS_R_8I;
  // CHECK-NEXT: hipblasDatatype_t C_8I = HIPBLAS_C_8I;
  // CHECK-NEXT: hipblasDatatype_t R_8U = HIPBLAS_R_8U;
  // CHECK-NEXT: hipblasDatatype_t C_8U = HIPBLAS_C_8U;
  // CHECK-NEXT: hipblasDatatype_t R_32I = HIPBLAS_R_32I;
  // CHECK-NEXT: hipblasDatatype_t C_32I = HIPBLAS_C_32I;
  // CHECK-NEXT: hipblasDatatype_t R_32U = HIPBLAS_R_32U;
  // CHECK-NEXT: hipblasDatatype_t C_32U = HIPBLAS_C_32U;
  hipDataType DataType;
  hipDataType DataType_t;
  hipDataType blasDataType;
  hipDataType R_16F = HIP_R_16F;
  hipDataType C_16F = HIP_C_16F;
  hipDataType R_32F = HIP_R_32F;
  hipDataType C_32F = HIP_C_32F;
  hipDataType R_64F = HIP_R_64F;
  hipDataType C_64F = HIP_C_64F;
  hipDataType R_8I = HIP_R_8I;
  hipDataType C_8I = HIP_C_8I;
  hipDataType R_8U = HIP_R_8U;
  hipDataType C_8U = HIP_C_8U;
  hipDataType R_32I = HIP_R_32I;
  hipDataType C_32I = HIP_C_32I;
  hipDataType R_32U = HIP_R_32U;
  hipDataType C_32U = HIP_C_32U;
#endif

#if CUDA_VERSION >= 11000
  // CHECK: hipblasDatatype_t R_16BF = HIPBLAS_R_16B;
  // CHECK-NEXT: hipblasDatatype_t C_16BF = HIPBLAS_C_16B;
  hipDataType R_16BF = HIP_R_16BF;
  hipDataType C_16BF = HIP_C_16BF;
#endif

#if CUDA_VERSION >= 8000
  // CHECK: hipblasGemmAlgo_t blasGemmAlgo;
  // CHECK-NEXT: hipblasGemmAlgo_t BLAS_GEMM_DFALT = HIPBLAS_GEMM_DEFAULT;
  hipblasGemmAlgo_t blasGemmAlgo;
  hipblasGemmAlgo_t BLAS_GEMM_DFALT = HIPBLAS_GEMM_DEFAULT;
#endif

#if CUDA_VERSION >= 9000
  // CHECK: hipblasGemmAlgo_t BLAS_GEMM_DEFAULT = HIPBLAS_GEMM_DEFAULT;
  hipblasGemmAlgo_t BLAS_GEMM_DEFAULT = HIPBLAS_GEMM_DEFAULT;
#endif

  // CHECK: hipblasHandle_t blasHandle;
  hipblasHandle_t blasHandle;

  return 0;
}
