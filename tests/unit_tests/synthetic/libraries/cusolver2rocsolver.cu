// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --skip-excluded-preprocessor-conditional-blocks --experimental --roc %clang_args -D__CUDA_API_VERSION_INTERNAL -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "rocsolver.h"
#include "hipsolver.h"

int main() {
  printf("20. cuSOLVER API to rocSOLVER API synthetic test\n");

  // CHECK: rocblas_handle handle;
  hipsolverHandle_t handle;

  // CHECK: rocblas_status status;
  // CHECK-NEXT: rocblas_status STATUS_SUCCESS = rocblas_status_success;
  // CHECK-NEXT: rocblas_status STATUS_NOT_INITIALIZED = rocblas_status_invalid_handle;
  // CHECK-NEXT: rocblas_status STATUS_ALLOC_FAILED = rocblas_status_memory_error;
  // CHECK-NEXT: rocblas_status STATUS_INVALID_VALUE = rocblas_status_invalid_value;
  // CHECK-NEXT: rocblas_status STATUS_ARCH_MISMATCH = rocblas_status_arch_mismatch;
  // CHECK-NEXT: rocblas_status STATUS_MAPPING_ERROR = rocblas_status_not_implemented;
  // CHECK-NEXT: rocblas_status STATUS_EXECUTION_FAILED = rocblas_status_not_implemented;
  // CHECK-NEXT: rocblas_status STATUS_INTERNAL_ERROR = rocblas_status_internal_error;
  // CHECK-NEXT: rocblas_status STATUS_NOT_SUPPORTED = rocblas_status_not_implemented;
  // CHECK-NEXT: rocblas_status STATUS_ZERO_PIVOT = rocblas_status_not_implemented;
  hipsolverStatus_t status;
  hipsolverStatus_t STATUS_SUCCESS = HIPSOLVER_STATUS_SUCCESS;
  hipsolverStatus_t STATUS_NOT_INITIALIZED = HIPSOLVER_STATUS_NOT_INITIALIZED;
  hipsolverStatus_t STATUS_ALLOC_FAILED = HIPSOLVER_STATUS_ALLOC_FAILED;
  hipsolverStatus_t STATUS_INVALID_VALUE = HIPSOLVER_STATUS_INVALID_VALUE;
  hipsolverStatus_t STATUS_ARCH_MISMATCH = HIPSOLVER_STATUS_ARCH_MISMATCH;
  hipsolverStatus_t STATUS_MAPPING_ERROR = HIPSOLVER_STATUS_MAPPING_ERROR;
  hipsolverStatus_t STATUS_EXECUTION_FAILED = HIPSOLVER_STATUS_EXECUTION_FAILED;
  hipsolverStatus_t STATUS_INTERNAL_ERROR = HIPSOLVER_STATUS_INTERNAL_ERROR;
  hipsolverStatus_t STATUS_NOT_SUPPORTED = HIPSOLVER_STATUS_NOT_SUPPORTED;
  hipsolverStatus_t STATUS_ZERO_PIVOT = HIPSOLVER_STATUS_ZERO_PIVOT;

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCreate(cusolverDnHandle_t *handle);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_create_handle(rocblas_handle* handle);
  // CHECK: status = rocblas_create_handle(&handle);
  status = hipsolverDnCreate(&handle);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDestroy(cusolverDnHandle_t handle);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_destroy_handle(rocblas_handle handle);
  // CHECK: status = rocblas_destroy_handle(handle);
  status = hipsolverDnDestroy(handle);

  return 0;
}
