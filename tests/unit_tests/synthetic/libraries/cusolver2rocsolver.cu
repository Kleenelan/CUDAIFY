// RUN: %run_test hipify "%s" "%t" %hipify_args 3 --skip-excluded-preprocessor-conditional-blocks --experimental --roc %clang_args -D__CUDA_API_VERSION_INTERNAL -ferror-limit=500

// CHECK: #include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// CHECK: #include "rocsolver.h"
#include "hipsolver.h"

int main() {
  printf("20. cuSOLVER API to rocSOLVER API synthetic test\n");

  // CHECK: rocblas_handle handle;
  hipsolverHandle_t handle;

  // CHECK: rocblas_status status;
  hipsolverStatus_t status;

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnCreate(cusolverDnHandle_t *handle);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_create_handle(rocblas_handle* handle);
  // CHECK: status = rocblas_create_handle(&handle);
  status = hipsolverDnCreate(&handle);

  // CUDA: cusolverStatus_t CUSOLVERAPI cusolverDnDestroy(cusolverDnHandle_t handle);
  // ROC: ROCBLAS_EXPORT rocblas_status rocblas_destroy_handle(rocblas_handle handle);
  // CHECK: status = rocblas_destroy_handle(handle);
  status = hipsolverDnDestroy(handle);

  return 0;
}
